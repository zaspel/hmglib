#include "hip/hip_runtime.h"
// Copyright (C) 2016 Peter Zaspel
//
// This file is part of hmglib.
//
// hmglib is free software: you can redistribute it and/or modify it under the
// terms of the GNU Lesser General Public License as published by the Free
// Software Foundation, either version 3 of the License, or (at your option) any
// later version.
//
// hmglib is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
// A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
// details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with hmglib.  If not, see <http://www.gnu.org/licenses/>.

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <thrust/scan.h>
#include <thrust/unique.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <thrust/inner_product.h>
#include <thrust/logical.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/partition.h>
#include <thrust/extrema.h>
#include <thrust/gather.h>

#include "system_assembler.h"

#include "linear_algebra.h"

#ifndef CHECK_CUDA_ERROR
#define CHECK_CUDA_ERROR
void checkCUDAError(const char* msg) {
hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
#endif

hipEvent_t ssstart, ssstop;
float mmmilliseconds;
hipEvent_t sssstart, sssstop;
float mmmmilliseconds;



#define TIME_ssstart  // {hipEventCreate(&ssstart); hipEventCreate(&ssstop); hipEventRecord(ssstart);}
#define TIME_ssstop(a)  // {hipEventRecord(ssstop); hipEventSynchronize(ssstop); hipEventElapsedTime(&mmmilliseconds, ssstart, ssstop); printf("%s: Elapsed time: %lf ms\n", a, mmmilliseconds); }

#define TIME_sssstart  // {hipEventCreate(&sssstart); hipEventCreate(&sssstop); hipEventRecord(sssstart);}
#define TIME_sssstop(a)  // {hipEventRecord(sssstop); hipEventSynchronize(sssstop); hipEventElapsedTime(&mmmmilliseconds, sssstart, sssstop); printf("%s: Elapsed time: %lf ms\n", a, mmmmilliseconds); }


struct mat_vec_type_smaller
{
  typedef struct work_item first_argument_type;

  typedef struct work_item second_argument_type;

  typedef bool result_type;

  __host__ __device__ bool operator()(const struct work_item &lhs, const struct work_item &rhs) const
  {
//	  return lhs.work_type==WT_DENSE;  <----- caused bug
	  return lhs.work_type>rhs.work_type;
  }
};

void sort_mat_vec_data(struct work_item* mat_vec_data, int mat_vec_data_count)
{
	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);

	struct mat_vec_type_smaller smaller;
	thrust::stable_sort(mat_vec_data_ptr, mat_vec_data_ptr+mat_vec_data_count, smaller);


}

__global__ void fill_batched_matrix(double* matrix, struct work_item* mat_vec_data, struct point_set* input_set1, struct point_set* input_set2, int* m1, int* m2, int m1_total, int* point_map1, int* work_item_map1, int* point_map_offset1, int* point_map_offset2, int m2_max, struct system_assembler* assem )
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= m1_total*m2_max)
                return;

	int row_in_batched_matrix = idx % m1_total;
	int column_in_batched_matrix = idx / m1_total;


        int work_item_index = work_item_map1[row_in_batched_matrix];

        if (column_in_batched_matrix>=m2[work_item_index])
	{
//		matrix[idx] = 0.0;
                return;
	}

        int i,j;

        j = column_in_batched_matrix;
        i = row_in_batched_matrix - point_map_offset1[work_item_index];
	
	// computing index for output in block_diagonal_matrix
	int full_idx = (point_map_offset2[work_item_index]+column_in_batched_matrix)*m1_total + row_in_batched_matrix;

	int i_global = mat_vec_data[work_item_index].set1_l+i;
	int j_global = mat_vec_data[work_item_index].set2_l+j;
	matrix[full_idx] = assem->get_matrix_entry(i_global, j_global, input_set1, input_set2);

}

__global__ void fill_batched_matrix_magma(double* matrix, struct work_item* mat_vec_data, struct point_set* input_set1, struct point_set* input_set2, int* m1, int* m2, int total_size, int* matrix_offsets, int mat_vec_data_count, struct system_assembler* assem)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= total_size)
                return;

//	int* idx_position_in_matrix_offset = thrust::upper_bound(thrust::device, matrix_offsets, matrix_offsets+mat_vec_data_count, idx);
//	int offset = idx_position_in_matrix_offset - matrix_offsets;
//	idx_position_in_matrix_offset--;
	int work_item_index=mat_vec_data_count-1;
	while ((work_item_index>0)&&(idx<matrix_offsets[work_item_index]))
		work_item_index--;

//	if (idx<800)
//		printf("idx: %d work_item_index: %d\n",idx, work_item_index);

	int idx_local = idx - matrix_offsets[work_item_index];

	int i,j;

	i = idx_local % m1[work_item_index];
	j = idx_local / m1[work_item_index]; 
	
	int i_global = mat_vec_data[work_item_index].set1_l+i;
	int j_global = mat_vec_data[work_item_index].set2_l+j;
	matrix[idx] = assem->get_matrix_entry(i_global, j_global, input_set1, input_set2);


}



/*
__global__ void fill_batched_matrix(double* matrix, struct work_item* mat_vec_data, struct point_set* input_set1, struct point_set* input_set2, int* m1, int* m2, int m1_total, int m2_total, int* point_map1, int* work_item_map1, int* point_map_offset1, int* point_map_offset2, int m2_max, int kernel_type )
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= m1_total*m2_total)
                return;

	int row_in_batched_matrix = idx % m1_total;
	int column_in_batched_matrix = idx / m1_total;


        int dim = input_set1->dim;
        double point1[MAX_DIM];
        double point2[MAX_DIM];

        int work_item_index = work_item_map1[row_in_batched_matrix];

        if ((column_in_batched_matrix<point_map_offset2[work_item_index])||(column_in_batched_matrix>=point_map_offset2[work_item_index]+m2[work_item_index]))
	{
		matrix[idx] = 0.0;
                return;
	}

        int i,j;

        j = column_in_batched_matrix - point_map_offset2[work_item_index];
        i = row_in_batched_matrix - point_map_offset1[work_item_index];
	
        for (int d=0; d<dim; d++)
        {
                point1[d] = input_set1->coords[d][mat_vec_data[work_item_index].set1_l+i];
                point2[d] = input_set2->coords[d][mat_vec_data[work_item_index].set2_l+j];
        }


//        double val = 0;
//        for (int d=0; d<dim; d++)
//        {
//                val += (point1[d]-point2[d])*(point1[d]-point2[d]);
//        }
//        val = sqrt(val);



	// computing index for output in block_diagonal_matrix
//	int full_idx = (point_map_offset2[work_item_index]+column_in_batched_matrix)*m1_total + row_in_batched_matrix;

        matrix[idx] = kernel(point1, point2, dim, kernel_type);

}
*/


__global__ void fill_matrix(double* matrix, struct work_item current_mat_vec_data, struct point_set* input_set1, struct point_set* input_set2, int m1, int m2, struct system_assembler* assem )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= m1*m2)
		return;

	int i,j;

	j = idx / m1;
	i = idx % m1;

	int i_global = current_mat_vec_data.set1_l+i;
	int j_global = current_mat_vec_data.set2_l+j;

	matrix[idx] = assem->get_matrix_entry(i_global, j_global, input_set1, input_set2);


}

void fill_matrix_fun(double* matrix, struct work_item current_mat_vec_data, struct point_set* input_set1, struct point_set* input_set2, int m1, int m2, int grid_size, int block_size, struct system_assembler* assem)
{
	fill_matrix<<<grid_size, block_size>>>(matrix, current_mat_vec_data, input_set1, input_set2, m1, m2, assem);
}

__global__ void fill_kernel_vector(double* vec, int lA, int uA, int iB, struct point_set* input_setA, struct point_set* input_setB, struct system_assembler* assem)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= uA-lA+1)
		return;

	int i_global = lA+idx;
	int j_global = iB;

	vec[idx] = assem->get_matrix_entry(i_global, j_global, input_setA, input_setB);

}

struct scaled_minus
{
	double scaling;
    scaled_minus(double _scaling)
	{
    	scaling = _scaling;
	}

    __host__ __device__ double operator()(const double &lhs, const double &rhs) const {return lhs - scaling*rhs;}
};

struct divide_by
{
	double val;

	__host__ __device__ double operator()(const double &x) const {return x/val;}
};

struct scale_by
{
	double val;

	__host__ __device__ double operator()(const double &x) const {return x*val;}
};

struct compare_absolute
{
	__host__ __device__	bool operator()(double lhs, double rhs)
	{
		return fabs(lhs) < fabs(rhs);
	}
};

void apply_dense_matrix_for_current_work_item(double* x, double* y, struct work_item current_mat_vec_data, struct point_set* input_set1, struct point_set* input_set2, hipblasStatus_t stat, hipblasHandle_t handle, struct system_assembler* assem)
{
	int block_size = MATRIX_ENTRY_BLOCK_SIZE;

	// getting matrix size
	int m1 = current_mat_vec_data.set1_u-current_mat_vec_data.set1_l+1; // number of rows
	int m2 = current_mat_vec_data.set2_u-current_mat_vec_data.set2_l+1; // number of columns

	// allocating local matrix
	double* matrix;
	hipMalloc((void**)&matrix, m1*m2*sizeof(double));

	checkCUDAError("before fill_matrix");
	// setup of local matrix
	fill_matrix<<<(m1*m2 + (block_size - 1)) / block_size, block_size>>>(matrix, current_mat_vec_data, input_set1, input_set2, m1, m2, assem);
	hipDeviceSynchronize();
	checkCUDAError("after fill_matrix");

	// allocation and extraction of local operand
	double* local_x;
	hipMalloc((void**)&local_x, m2*sizeof(double));
	hipMemcpy(local_x, &x[current_mat_vec_data.set2_l], m2*sizeof(double), hipMemcpyDeviceToDevice);

	// allocation of local result
	double* local_y;
	hipMalloc((void**)&local_y, m1*sizeof(double));

	// matrix-vector-product
	double one;
	double zero;
	one = 1.0;
	zero = 0.0;
	stat = hipblasDgemv(handle, HIPBLAS_OP_N, m1, m2, &one, matrix, m1, local_x, 1, &zero, local_y, 1);
	if (stat!=HIPBLAS_STATUS_SUCCESS)
	{
		printf("dgemv did not succeed...\n");
		exit(1);
	}

	thrust::device_ptr<double> local_y_ptr(local_y);
	thrust::device_ptr<double> y_ptr(y);

	// adding local result to full vector
	thrust::transform(y_ptr+current_mat_vec_data.set1_l, y_ptr+current_mat_vec_data.set1_l+m1, local_y_ptr, y_ptr+current_mat_vec_data.set1_l, thrust::plus<double>());

	// cleanup
	hipFree(local_y);
	hipFree(local_x);
	hipFree(matrix);

}




__global__ void fill_kernel_vector_and_substract_previous_vectors(double* vec, int lA, int uA, int iB, struct point_set* input_setA, struct point_set* input_setB, int m1, int m2, double* U, double* V, int r, int i_r, struct system_assembler* assem)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= m2)
		return;

	int i_global = lA+idx;
	int j_global = iB;

	double val = assem->get_matrix_entry(i_global, j_global, input_setA, input_setB);

	for (int l=0; l<r; l++)
	{
		double scaling = U[l*m1+i_r];
		val -= scaling*V[l*m2+idx];
	}

	vec[idx] = val;
}

double compute_frobenius_norm_of_low_rank_matrix(double* U, double* V, int m1, int m2, int k, hipblasStatus_t stat, hipblasHandle_t handle)
{
	// frobenius(U*V') = sqrt(sum(sum((V'*V).*(U'*U))))

	// C = U'*U
	double* C;
	hipMalloc((void**)&C, k*k*sizeof(double));

	double one;
	double zero;
	one = 1.0;
	zero = 0.0;
	stat = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, k, k, m1, &one, U, m1, U, m1, &zero, C, k);
	if (stat!=HIPBLAS_STATUS_SUCCESS)
	{
		printf("dgemm did not succeed...\n");
		exit(1);
	}

	// D = V'*V
	double* D;
	hipMalloc((void**)&D, k*k*sizeof(double));

	stat = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, k, k, m2, &one, V, m2, U, m2, &zero, D, k);
	if (stat!=HIPBLAS_STATUS_SUCCESS)
	{
		printf("dgemm did not succeed...\n");
		exit(1);
	}

	// res = C(:)'*D(:)
	double res;
	stat = hipblasDdot(handle, k*k, C, 1, D, 1, &res);

	hipFree(C);
	hipFree(D);

	// res = sqrt(res)
	return sqrt(res);
}


void apply_aca_for_current_work_item(double* x, double* y, struct work_item current_mat_vec_data, struct point_set* input_set1, struct point_set* input_set2,  hipblasStatus_t stat, hipblasHandle_t handle, double eta, double epsilon, int k, struct system_assembler* assem)
{
	int block_size = MATRIX_ENTRY_BLOCK_SIZE;

	// getting matrix size
	int m1 = current_mat_vec_data.set1_u-current_mat_vec_data.set1_l+1; // number of rows
	int m2 = current_mat_vec_data.set2_u-current_mat_vec_data.set2_l+1; // number of columns

	// if (k>min(m,n))
	//     k= min(m,n);
	// end
	if (k>min(m1, m2))
		k = min(m1, m2);

	double* U;
	hipMalloc((void**)&U, m1*k*sizeof(double));
	checkCUDAError("hipMalloc");
	double* V;
	hipMalloc((void**)&V, m2*k*sizeof(double));
	checkCUDAError("hipMalloc");

	thrust::device_ptr<double> U_ptr(U);
	thrust::device_ptr<double> V_ptr(V);

	thrust::fill(U_ptr, U_ptr+m1*k, 0.0);
	thrust::fill(V_ptr, V_ptr+m2*k, 0.0);

	double* v_r;
	double* u_r;

	// i_r = 0;
	int i_r = -1;

	struct divide_by div;

////	TIME_ssstart;

	// for r=1:k
	for (int r=0; r<k; r++)
	{
		// while (norm(v_tilde_r,Inf)==0.0)
	    //    i_r = i_r+1;
	    //    v_tilde_r = kernel(input_set1(i_r,:), input_set2);
	    //    for l=1:r-1
	    //        v_tilde_r = v_tilde_r - U(i_r,l) * V(l,:);
	    //    end
	    // end

        // U = [U u_r];
        // V = [V; v_r];
		v_r = &V[r*m2];
		u_r = &U[r*m1];
		thrust::device_ptr<double> u_r_ptr(u_r);
		thrust::device_ptr<double> v_r_ptr(v_r);

		do
		{
//		//	TIME_ssstart;
			i_r++;

			fill_kernel_vector_and_substract_previous_vectors<<<(m2 + (block_size - 1)) / block_size, block_size>>>(v_r, current_mat_vec_data.set2_l, current_mat_vec_data.set2_u, current_mat_vec_data.set1_l+i_r, input_set2, input_set1, m1, m2, U, V, r, i_r, assem);
			hipDeviceSynchronize();
			checkCUDAError("fill_kernel_vector_and_substract_previous_vectors");

			double norm = sqrt(thrust::inner_product(v_r_ptr, v_r_ptr+m2, v_r_ptr, 0.0));

//		//	TIME_ssstop("ACA do-loop");

			if (norm>=1.0e-13) break;
		} while (true);


//	//	TIME_ssstart;

	    // [m,j_r] = max(abs(v_tilde_r));
		thrust::device_ptr<double> max_pos = thrust::max_element(v_r_ptr, v_r_ptr+m2, compare_absolute());
		int j_r = max_pos - v_r_ptr;

//	//	TIME_ssstop("ACA intermediate 1");
//	//	TIME_ssstart;

		// v_r = (1.0./(v_tilde_k(j_r)))*v_tilde_r;
	    hipMemcpy(&div.val, &v_r[j_r], sizeof(double), hipMemcpyDeviceToHost);
		checkCUDAError("cudaMemcpy2");
		thrust::transform(v_r_ptr, v_r_ptr+m2, v_r_ptr, div);

//	//	TIME_ssstop("ACA intermediate 2");
//	//	TIME_ssstart;

//		// u_r = kernel(input_set1(:,:),input_set2(j_r,:));
//	    // for l=1:r-1
//	    //     u_r = u_r - V(l,j_r) * U(:,l);
//	    // end

		fill_kernel_vector_and_substract_previous_vectors<<<(m1 + (block_size - 1)) / block_size, block_size>>>(u_r, current_mat_vec_data.set1_l, current_mat_vec_data.set1_u, current_mat_vec_data.set2_l+j_r, input_set1, input_set2, m2, m1, V, U, r, j_r, assem);
		hipDeviceSynchronize();
		checkCUDAError("fill_kernel_vector_and_substract_previous_vectors");

//	//	TIME_ssstop("ACA middle");

		bool check_frobenius = false;

		if ((check_frobenius) && (r%5==0)) // apply stopping criterion only in every fifth iteration since it is very expensive
		{
//		//	TIME_ssstart;

			// frobenius(U*V') = sqrt(sum(sum((V'*V).*(U'*U))))
			double res = compute_frobenius_norm_of_low_rank_matrix(U, V, m1, m2, k, stat, handle);

//		//	TIME_ssstop("ACA frobenius norm");

//		//	TIME_ssstart;

			double u_r_2norm;
			double v_r_2norm;

			// factor of 2 in performance (better) when cuBLAS is not used and thrust is used !!!
			u_r_2norm = sqrt(thrust::inner_product(u_r_ptr, u_r_ptr+m1, u_r_ptr, 0.0));
			v_r_2norm = sqrt(thrust::inner_product(v_r_ptr, v_r_ptr+m2, v_r_ptr, 0.0));


//		//	TIME_ssstop("ACA norm rest");

			//		printf("u v f %le %le %le\n", u_r_2norm, v_r_2norm, res);
			if (u_r_2norm*v_r_2norm <= ((epsilon*(1.0-eta))/(1.0+epsilon))*res)
			{
				//			printf("AAAAUUUUFFFHÖÖÖÖÖRRREEENNNN!!!!!! Schluss jetzt!\n");
				//			printf("r=%d\n", r);
				break;
			}
		}
	}

////	TIME_ssstop("aca approx");

////	TIME_ssstart;

	// allocation and extraction of local operand
	double* local_x;
	hipMalloc((void**)&local_x, m2*sizeof(double));
	checkCUDAError("hipMalloc");
	hipMemcpy(local_x, &x[current_mat_vec_data.set2_l], m2*sizeof(double), hipMemcpyDeviceToDevice);

	// allocation of local intermediate result
	double* local_tmp;
	hipMalloc((void**)&local_tmp, k*sizeof(double));
	checkCUDAError("hipMalloc");

	// allocation of local result
	double* local_y;
	hipMalloc((void**)&local_y, m1*sizeof(double));
	checkCUDAError("hipMalloc");

	// matrix-vector-product
	double one;
	double zero;
	one = 1.0;
	zero = 0.0;
	stat = hipblasDgemv(handle, HIPBLAS_OP_T, m2, k, &one, V, m2, local_x, 1, &zero, local_tmp, 1);
	if (stat!=HIPBLAS_STATUS_SUCCESS)
	{
		printf("dgemv did not succeed...\n");
		exit(1);
	}

	stat = hipblasDgemv(handle, HIPBLAS_OP_N, m1, k, &one, U, m1, local_tmp, 1, &zero, local_y, 1);
	if (stat!=HIPBLAS_STATUS_SUCCESS)
	{
		printf("dgemv did not succeed...\n");
		exit(1);
	}

	thrust::device_ptr<double> local_y_ptr(local_y);
	thrust::device_ptr<double> y_ptr(y);

	// adding local result to full vector
	thrust::transform(y_ptr+current_mat_vec_data.set1_l, y_ptr+current_mat_vec_data.set1_l+m1, local_y_ptr, y_ptr+current_mat_vec_data.set1_l, thrust::plus<double>());

////	TIME_ssstop("aca apply");

	hipFree(local_x);
	hipFree(local_y);
	hipFree(local_tmp);
	hipFree(U);
	hipFree(V);
//	hipFree(v_r);
//	hipFree(u_r);

}

__global__ void set_bounds_for_point_maps(int* point_map1, int* point_map2, int* point_map_offsets1, int* point_map_offsets2, int* m1, int* m2, int work_item_type, struct work_item* current_level_data, int mat_vec_data_count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= mat_vec_data_count)
		return;

	struct work_item* work = &current_level_data[idx];

	if (work->work_type!=work_item_type)
		return;

	if (work->set1_l!=work->set1_u)
	{
		point_map1[point_map_offsets1[idx]] = work->set1_l;
		point_map1[point_map_offsets1[idx]+m1[idx]-1] = -(work->set1_u-1);
	}

	if (work->set2_l!=work->set2_u)
	{
		point_map2[point_map_offsets2[idx]] = work->set2_l;
		point_map2[point_map_offsets2[idx]+m2[idx]-1] = -(work->set2_u-1);
	}
}

__global__ void set_bounds_for_point_maps_with_padding(int* point_map2, int* m2, int padding2, int work_item_type, struct work_item* current_level_data, int mat_vec_data_count)
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= mat_vec_data_count)
                return;

        struct work_item* work = &current_level_data[idx];

        if (work->work_type!=work_item_type)
                return;

	if (work->set2_l!=work->set2_u)
	{
        	point_map2[padding2*idx] = work->set2_l;
        	point_map2[padding2*idx+m2[idx]-1] = -(work->set2_u-1);
		if (m2[idx]<padding2)
			point_map2[padding2*(idx+1)-1] = -(padding2-(work->set2_u-work->set2_l+1)-1);
	}
}

__global__ void set_bounds_for_pattern_with_padding(int* pattern2, int* m2, int padding2, int work_item_type, struct work_item* current_level_data, int mat_vec_data_count)
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= mat_vec_data_count)
                return;

        struct work_item* work = &current_level_data[idx];

        if (work->work_type!=work_item_type)
                return;

	if (m2[idx]>0)  // DEBUG: Is this enough????
	{
	        pattern2[padding2*idx] = 1;
	        pattern2[padding2*idx+m2[idx]-1] = -1;
	}
}


__global__ void correct_bounds_for_point_maps(int* point_map1, int* point_map2, int* point_map_offsets1, int* point_map_offsets2, int* m1, int* m2, int work_item_type, struct work_item* current_level_data, int mat_vec_data_count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= mat_vec_data_count)
		return;

	struct work_item* work = &current_level_data[idx];

	if (work->work_type!=work_item_type)
		return;

//	if (work->set1_u!=work->set1_l)
		point_map1[point_map_offsets1[idx]+m1[idx]-1] = work->set1_u;

//	if (work->set2_u!=work->set2_l)
		point_map2[point_map_offsets2[idx]+m2[idx]-1] = work->set2_u;
}

__global__ void correct_bounds_for_point_maps_with_padding(int* point_map2, int* m2, int padding2, int work_item_type, struct work_item* current_level_data, int mat_vec_data_count)
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= mat_vec_data_count)
                return;

        struct work_item* work = &current_level_data[idx];

        if (work->work_type!=work_item_type)
                return;

//	if (work->set2_l!=work->set2_u)
	        point_map2[padding2*idx+m2[idx]-1] = work->set2_u;
}

__global__ void correct_bounds_for_pattern_with_padding(int* pattern2, int* m2, int padding2, int work_item_type, struct work_item* current_level_data, int mat_vec_data_count)
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= mat_vec_data_count)
                return;

        struct work_item* work = &current_level_data[idx];

        if (work->work_type!=work_item_type)
                return;

//	if (m2[idx]>0)
	        pattern2[padding2*idx+m2[idx]-1] = 1;
}


__global__ void set_bounds_for_work_item_maps(int* work_item_map1, int* work_item_map2, int* point_map_offsets1, int* point_map_offsets2, int* m1, int* m2, int work_item_type, struct work_item* current_level_data, int mat_vec_data_count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= mat_vec_data_count)
		return;

	struct work_item* work = &current_level_data[idx];

	if (work->work_type!=work_item_type)
		return;

	if (m1[idx]>0)
	{
		work_item_map1[point_map_offsets1[idx]] = idx;
		work_item_map1[point_map_offsets1[idx]+m1[idx]-1] = -idx;
	}

	if (m2[idx]>0)
	{
		work_item_map2[point_map_offsets2[idx]] = idx;
		work_item_map2[point_map_offsets2[idx]+m2[idx]-1] = -idx;
	}
}

__global__ void correct_bounds_for_work_item_maps(int* work_item_map1, int* work_item_map2, int* point_map_offsets1, int* point_map_offsets2, int* m1, int* m2, int work_item_type, struct work_item* current_level_data, int mat_vec_data_count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= mat_vec_data_count)
		return;

	struct work_item* work = &current_level_data[idx];

	if (work->work_type!=work_item_type)
		return;

//	if (m1[idx]>0)
		work_item_map1[point_map_offsets1[idx]+m1[idx]-1] = idx;

//	if (m2[idx]>0)
		work_item_map2[point_map_offsets2[idx]+m2[idx]-1] = idx;
}

//__global__ void set_bounds_for_point_maps_valid_entries(int* point_map_valid_entries1, int* point_map_valid_entries2, int* point_map_offsets1, int* point_map_offsets2, int work_item_type, struct work_item* current_level_data, int mat_vec_data_count)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (idx >= mat_vec_data_count)
//		return;
//
//	struct work_item* work = &current_level_data[idx];
//
//	if (work->work_type!=work_item_type)
//		return;
//
//	point_map_valid_entries1[point_map_offsets1[idx]] = 1;
//	point_map_valid_entries1[point_map_offsets1[idx+1]-1] = -1;
//
//	point_map_valid_entries2[point_map_offsets2[idx]] = 1;
//	point_map_valid_entries2[point_map_offsets2[idx+1]-1] = -1;
//}
//
//__global__ void correct_bounds_for_point_maps_valid_entries(int* point_map1, int* point_map2, int* point_map_offsets1, int* point_map_offsets2, int work_item_type, struct work_item* current_level_data, int mat_vec_data_count)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (idx >= mat_vec_data_count)
//		return;
//
//	struct work_item* work = &current_level_data[idx];
//
//	if (work->work_type!=work_item_type)
//		return;
//
//	point_map1[point_map_offsets1[idx+1]-1] = 1;
//
//	point_map2[point_map_offsets2[idx+1]-1] = 1;
//}



struct minus_plus_1
{
    __host__ __device__ double operator()(const double &lhs, const double &rhs) const {return lhs - rhs + 1;}
};

__global__ void set_k_per_item(int* k_per_item, int k, int mat_vec_data_count, int* m1, int* m2)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= mat_vec_data_count)
		return;

	int _m1, _m2;
	_m1 = m1[idx];
	_m2 = m2[idx];

	if (k>min(_m1, _m2))
	{
		k_per_item[idx] = min(_m1, _m2);
	}
	else
		k_per_item[idx] = k;

}

__global__ void batched_fill_kernel_vector_v_r(double* v_r, int* point_map2, int* point_map1, int* point_map_offsets1, int* work_item_map2, int* i_r, int* compute_v_r, struct point_set* input_set2, struct point_set* input_set1, int m2_total, struct system_assembler* assem)
{
	//    v_tilde_r = kernel(input_set1(i_r,:), input_set2);

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= m2_total)
		return;

	int global_point_index2 = point_map2[idx];

// not necessary since all entries are valid
//	if (global_point_index2==-1)
//		return;

	int work_item_index = work_item_map2[idx];

	// this work item has already been successfully computed
	if (compute_v_r[work_item_index]==0)
		return;

	int i_global = point_map1[point_map_offsets1[work_item_index]]+i_r[work_item_index];
	int j_global = global_point_index2;

	v_r[idx] = assem->get_matrix_entry(i_global, j_global, input_set1, input_set2);
}

struct is_zero
{
	__host__ __device__ bool operator()(int x) { return x==0; }
};

struct is_one
{
	__host__ __device__ bool operator()(int x) { return x==1; }
};

struct is_minus_one
{
	__host__ __device__ bool operator()(int x) { return x==-1; }
};

struct is_not_minus_one
{
	__host__ __device__ bool operator()(int x) { return x!=-1; }
};

struct square
{
	__host__ __device__ double operator()(double x) {return x*x;}
};

struct square_root
{
	__host__ __device__ double operator()(double x) {return sqrt(x);}
};

struct bigger_than_eps
{
	__host__ __device__ bool operator()(double x) {return fabs(x)>=1.0e-13;}
};

struct add_one
{
	__host__ __device__ int operator()(int a) { return a+1; }
};

struct is_not_WT_ACA
{
	__host__ __device__ bool operator()(struct work_item a) { return a.work_type!=WT_ACA; }
};

struct is_not_WT_DENSE
{
	__host__ __device__ bool operator()(struct work_item a) { return a.work_type!=WT_DENSE; }
};

struct is_smaller_or_equal_r
{
	int r;

	is_smaller_or_equal_r(int _r)
	{
		r = _r;
	}

	__host__ __device__ bool operator()(int a) { return a<=r; }
};

struct is_smaller
{
	double val;

	is_smaller(double _val)
	{
		val = _val;
	}

	__host__ __device__ bool operator()(double a) { return a<val; }
};


__global__ void batched_scaled_substraction_for_v_r(double* v_r, int* point_map2, int* point_map_offsets1, int* work_item_map2, int* i_r, int* compute_v_r, double* V, double* U, struct point_set* input_set2, struct point_set* input_set1, int* k_per_item, int r, int m2_total, int m1_total)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= m2_total)
		return;

	int work_item_index = work_item_map2[idx];

	// nothing to do left for this element
	if (compute_v_r[work_item_index]==0)
		return;

	for (int l=0; (l<r)&&(l<k_per_item[work_item_index]); l++)
	{
		//		hipMemcpy(&scaling, &U[l*m1+i_r], sizeof(double), hipMemcpyDeviceToHost);
		//		checkCUDAError("cudaMemcpy1");
		//
		//		struct scaled_minus p(scaling);
		//
		//		thrust::transform(v_r_ptr, v_r_ptr+m2, V_ptr+(l*m2), v_r_ptr, p);

		double scaling = U[l*m1_total + point_map_offsets1[work_item_index] + i_r[work_item_index]];
		v_r[idx] -= scaling * V[l*m2_total+idx];
	}
}

__global__ void batched_fill_kernel_vector_and_scaled_substraction_for_v_r(double* v_r, int* point_map2, int* point_map1, int* point_map_offsets1, int* work_item_map2, int* i_r, bool* search_for_new_v_r, struct point_set* input_set2, struct point_set* input_set1, int m2_total, int m1_total, double* V, double* U, int r, int* k_per_item, struct system_assembler* assem)
{
	//    v_tilde_r = kernel(input_set1(i_r,:), input_set2);

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= m2_total)
		return;

	int global_point_index2 = point_map2[idx];

// not necessary since all entries are valid
//	if (global_point_index2==-1)
//		return;

	int work_item_index = work_item_map2[idx];

	// stopping if I shall not search for a new v_r
	if (!search_for_new_v_r[work_item_index])
		return;

	int i_global = point_map1[point_map_offsets1[work_item_index]]+i_r[work_item_index];
	int j_global = global_point_index2;

	double val = assem->get_matrix_entry(i_global, j_global, input_set1, input_set2);


	for (int l=0; (l<r)/*&&(l<k_per_item[work_item_index])*/; l++)  // commented out stuff that should not be necessary after checking for if (compute_v_r[work_item_index]==0)
	{
		//		hipMemcpy(&scaling, &U[l*m1+i_r], sizeof(double), hipMemcpyDeviceToHost);
		//		checkCUDAError("cudaMemcpy1");
		//
		//		struct scaled_minus p(scaling);
		//
		//		thrust::transform(v_r_ptr, v_r_ptr+m2, V_ptr+(l*m2), v_r_ptr, p);

		double scaling = U[l*m1_total + point_map_offsets1[work_item_index] + i_r[work_item_index]];
		val -= scaling * V[l*m2_total+idx];
	}

	v_r[idx] = val;
}



__global__ void batched_scaling_of_v_r(double* v_r, int* work_item_to_batch_map, int* work_item_map2, int* k_per_item, int r, int* j_r, int m2_total, bool* stop_full_aca_for_batch)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= m2_total)
		return;

	int work_item_index = work_item_map2[idx];

	if (stop_full_aca_for_batch[work_item_index])
		return;

//	if (r>=k_per_item[work_item_index])
//		return;

	// not necessary => is always valid by construction
	// if (work_item_index==-1)
	//	return;

	int batch_number = work_item_to_batch_map[work_item_index];

    // v_r = (1.0./(v_tilde_k(j_r)))*v_tilde_r;
	v_r[idx] = v_r[idx] / v_r[j_r[batch_number]];
}


typedef thrust::tuple<double,int> DoubleIntTuple;

struct tuple_absolute_maximum
{
	__host__ __device__ DoubleIntTuple operator()(const DoubleIntTuple lhs, const DoubleIntTuple rhs) const
	{
		double x1 = thrust::get<0>(lhs);
		double x2 = thrust::get<0>(rhs);
		double i1 = thrust::get<1>(lhs);
		double i2 = thrust::get<1>(rhs);
		double abs_max_x;
		double abs_max_i;

		if (fabs(x1)>fabs(x2))
		{
			abs_max_x = x1;
			abs_max_i = i1;
		}
		else
		{
			abs_max_x = x2;
			abs_max_i = i2;
		}

		DoubleIntTuple result;
		thrust::get<0>(result) = abs_max_x;
		thrust::get<1>(result) = abs_max_i;
		return result;
	}
};


__global__ void batched_fill_kernel_vector_u_r(double* u_r, int* point_map1, int* point_map2, int* work_item_to_batch_map, int* work_item_map1, int* k_per_item, int r, int* j_r_global, struct point_set* input_set1, struct point_set* input_set2, int m1_total, struct system_assembler* assem)
{
	//    // u_r = kernel(input_set1(:,:),input_set2(j_r,:));
	//	fill_kernel_vector<<<(m1 + (block_size - 1)) / block_size, block_size>>>(u_r, current_mat_vec_data.set1_l, current_mat_vec_data.set1_u, current_mat_vec_data.set2_l+j_r, input_set1, input_set2);
	//	hipDeviceSynchronize();
	//	checkCUDAError("fill_kernel_vector2");

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= m1_total)
		return;

	int global_point_index1 = point_map1[idx];

// is always valid by construction
//	if (global_point_index1==-1)
//		return;

	int work_item_index = work_item_map1[idx];

	if (r>=k_per_item[work_item_index])
		return;

	// oh my gosh, the following is extremely expensive!
	int batch_number = work_item_to_batch_map[work_item_index];
	int global_point_index2 = point_map2[j_r_global[batch_number]];

	int i_global = global_point_index1;
	int j_global = global_point_index2;

	u_r[idx] = assem->get_matrix_entry(i_global, j_global, input_set1, input_set2);
}

__global__ void batched_scaled_substraction_for_u_r(double* u_r, int* point_map1, int* work_item_to_batch_map, int* work_item_map1, int* j_r_global, double* U, double* V, struct point_set* input_set1, struct point_set* input_set2, int* k_per_item, int r, int m1_total, int m2_total)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= m1_total)
		return;

	int work_item_index = work_item_map1[idx];

	if (r>=k_per_item[work_item_index])
		return;

	int batch_number = work_item_to_batch_map[work_item_index];

	for (int l=0; l<r; l++)
	{
		//	    // for l=1:r-1
		//	    //     u_r = u_r - V(l,j_r) * U(:,l);
		//	    // end
		//		for (int l=0; l<r; l++)
		//		{
		//			double scaling;
		//			hipMemcpy(&scaling, &V[l*m2+j_r], sizeof(double), hipMemcpyDeviceToHost);
		//			checkCUDAError("cudaMemcpy3");
		//
		//			struct scaled_minus p(scaling);
		//
		//			thrust::transform(u_r_ptr, u_r_ptr+m1, U_ptr+(l*m1), u_r_ptr, p);
		//		}

		double scaling = V[l*m2_total + j_r_global[batch_number]];
		u_r[idx] -= scaling * U[l*m1_total+idx];
	}
}



__global__ void batched_fill_kernel_vector_and_scaled_substraction_for_u_r(double* u_r, int* point_map1, int* point_map2, int* work_item_to_batch_map, int* work_item_map1, int* k_per_item, int r, int* j_r_global, struct point_set* input_set1, struct point_set* input_set2, int m1_total, int m2_total, double* U, double* V, struct system_assembler* assem, bool* stop_full_aca_for_batch)
{
	//    // u_r = kernel(input_set1(:,:),input_set2(j_r,:));
	//	fill_kernel_vector<<<(m1 + (block_size - 1)) / block_size, block_size>>>(u_r, current_mat_vec_data.set1_l, current_mat_vec_data.set1_u, current_mat_vec_data.set2_l+j_r, input_set1, input_set2);
	//	hipDeviceSynchronize();
	//	checkCUDAError("fill_kernel_vector2");

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= m1_total)
		return;

	int global_point_index1 = point_map1[idx];

// is always valid by construction
//	if (global_point_index1==-1)
//		return;

	int work_item_index = work_item_map1[idx];

	if (stop_full_aca_for_batch[work_item_index])
		return;

	// oh my gosh, the following is extremely expensive!
	int batch_number = work_item_to_batch_map[work_item_index];

	int j_r_global_evaluated = j_r_global[batch_number];

	int global_point_index2 = point_map2[j_r_global_evaluated];

	int i_global = global_point_index1;
	int j_global = global_point_index2;

	double val = assem->get_matrix_entry(i_global, j_global, input_set1, input_set2);

	for (int l=0; l<r; l++)
	{
		//	    // for l=1:r-1
		//	    //     u_r = u_r - V(l,j_r) * U(:,l);
		//	    // end
		//		for (int l=0; l<r; l++)
		//		{
		//			double scaling;
		//			hipMemcpy(&scaling, &V[l*m2+j_r], sizeof(double), hipMemcpyDeviceToHost);
		//			checkCUDAError("cudaMemcpy3");
		//
		//			struct scaled_minus p(scaling);
		//
		//			thrust::transform(u_r_ptr, u_r_ptr+m1, U_ptr+(l*m1), u_r_ptr, p);
		//		}

		double scaling = V[l*m2_total + j_r_global_evaluated];
		val -= scaling * U[l*m1_total + idx];
	}

	u_r[idx] = val;
}




__device__ double myAtomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void add_batched_local_results_to_full_vector(double* y, double* y_local, int* point_map1, int* work_item_map1, int m1_total)
{
	// adding batched local results to full vector
	//thrust::transform(y_ptr+current_mat_vec_data.set1_l, y_ptr+current_mat_vec_data.set1_l+m1, local_y_ptr, y_ptr+current_mat_vec_data.set1_l, thrust::plus<double>());

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= m1_total)
		return;

	double val = y_local[idx];

	myAtomicAdd(&y[point_map1[idx]], val);
//	atomicAdd(&y[point_map1[idx]], val);

}

__global__ void get_work_item_point_set_limits_for_given_type(int* l, int* u, int mat_vec_data_type, struct work_item* mat_vec_data, int mat_vec_data_count, int point_set_nr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= mat_vec_data_count)
		return;


	if (mat_vec_data[idx].work_type==mat_vec_data_type)
	{
		if (point_set_nr==1)
		{
			l[idx] = mat_vec_data[idx].set1_l;
			u[idx] = mat_vec_data[idx].set1_u;
		}
		else
		{
			l[idx] = mat_vec_data[idx].set2_l;
			u[idx] = mat_vec_data[idx].set2_u;
		}
	}
	else
	{
		l[idx] = -1;
		u[idx] = -2;
	}

}

// if a work_item has finished, set respective compute_v_r entry to 0
__global__ void update_ir(int* i_r, bool* search_for_new_v_r, int mat_vec_data_count, int* keys_output, double* values_output, int output_set_counts, int* m1)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= output_set_counts)
		return;

	int work_item_index = keys_output[idx];

	if (!search_for_new_v_r[work_item_index])
		return;

	if (fabs(values_output[idx])>=1.0e-13)
	{
		search_for_new_v_r[work_item_index] = false;
		return;
	}
}

__global__ void remove_rubbish_from_maxima(bool* search_for_new_v_r, int mat_vec_data_count, int* keys_output, double* values_output, int output_set_counts)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= output_set_counts)
		return;

	int work_item_index = keys_output[idx];

	if (!search_for_new_v_r[work_item_index])
	{
		values_output[idx] = 0.0;
		return;
	}
}

__global__ void finalize_norm_computation(double* values_output,int output_set_counts)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= output_set_counts)
		return;

	values_output[idx] = sqrt(values_output[idx]);
}


//double* values_output;
//hipMalloc((void**)&values_output, m_total*sizeof(double));

void compute_batched_norms(double* batched_norms, int* norm_count, double* x, int m_total, thrust::device_ptr<int> work_item_map_ptr, int block_size)
{
		double* x_tmp;
		int* keys_output;
		hipMalloc((void**)&x_tmp, m_total*sizeof(double));
		hipMalloc((void**)&keys_output, m_total*sizeof(int));
		thrust::device_ptr<double> x_tmp_ptr(x_tmp);
		thrust::device_ptr<int> keys_output_ptr(keys_output);
		thrust::device_ptr<double> batched_norms_ptr(batched_norms);

		// computing norms of batched vectors
		hipMemcpy(x_tmp, x, m_total*sizeof(double), hipMemcpyDeviceToDevice);
		thrust::transform(x_tmp_ptr, x_tmp_ptr+m_total, x_tmp_ptr, square());
		thrust::pair<thrust::device_ptr<int>, thrust::device_ptr<double> > new_end;
		new_end = thrust::reduce_by_key(work_item_map_ptr, work_item_map_ptr+m_total, x_tmp_ptr, keys_output_ptr, batched_norms_ptr, thrust::equal_to<int>(), thrust::plus<double>());

		// output_set_count is NOT equal to mat_mat_vec_data_count, since invalid entries are discarded
		*norm_count = new_end.first - keys_output_ptr;

		finalize_norm_computation<<<(*norm_count + (block_size-1)) / block_size, block_size>>>(batched_norms, *norm_count);
		checkCUDAError("finalize_norm_computation");

		hipFree(keys_output);
		hipFree(x_tmp);
}

void compute_batched_norms_with_keys_output(double* batched_norms, int* keys_output, int* norm_count, double* x, int m_total, thrust::device_ptr<int> work_item_map_ptr, int block_size)
{
		double* x_tmp;
		hipMalloc((void**)&x_tmp, m_total*sizeof(double));
		checkCUDAError("hipMalloc");
		thrust::device_ptr<double> x_tmp_ptr(x_tmp);
		thrust::device_ptr<int> keys_output_ptr(keys_output);
		thrust::device_ptr<double> batched_norms_ptr(batched_norms);

		// computing norms of batched vectors
		hipMemcpy(x_tmp, x, m_total*sizeof(double), hipMemcpyDeviceToDevice);
		checkCUDAError("hipMemcpy");
		thrust::transform(x_tmp_ptr, x_tmp_ptr+m_total, x_tmp_ptr, square());
		thrust::pair<thrust::device_ptr<int>, thrust::device_ptr<double> > new_end;
		new_end = thrust::reduce_by_key(work_item_map_ptr, work_item_map_ptr+m_total, x_tmp_ptr, keys_output_ptr, batched_norms_ptr, thrust::equal_to<int>(), thrust::plus<double>());

		// output_set_count is NOT equal to mat_mat_vec_data_count, since invalid entries are discarded
		*norm_count = new_end.first - keys_output_ptr;

		finalize_norm_computation<<<(*norm_count + (block_size-1)) / block_size, block_size>>>(batched_norms, *norm_count);
		checkCUDAError("finalize_norm_computation");

		hipFree(x_tmp);
}


void compute_batched_products_for_kxk_matrices(double* batched_products, int* products_count, double* C, double* D, int m_total, thrust::device_ptr<int> work_item_map_ptr, int block_size, bool* stop_aca_as_soon_as_possible)
{
		double* x_tmp;
		int* keys_output;
		hipMalloc((void**)&x_tmp, m_total*sizeof(double));
		hipMalloc((void**)&keys_output, m_total*sizeof(int));
		thrust::device_ptr<double> x_tmp_ptr(x_tmp);
		thrust::device_ptr<int> keys_output_ptr(keys_output);
		thrust::device_ptr<double> batched_products_ptr(batched_products);
		thrust::device_ptr<double> C_ptr(C);
		thrust::device_ptr<double> D_ptr(D);
		thrust::device_ptr<bool> stop_aca_as_soon_as_possible_ptr(stop_aca_as_soon_as_possible);

//		double* batched_products_old;
//		hipMalloc((void**)&batched_products_old, batch_count*sizeof(bool));
//		hipMemcpy(batched_products_old, batched_products, batch_count*sizeof(bool), hipMemcpyDeviceToDevice);
//
		// do pointwise product between C and D matrices
		thrust::transform(C_ptr,C_ptr+m_total,D_ptr,x_tmp_ptr, thrust::multiplies<double>());

		// computing batched products
		thrust::pair<thrust::device_ptr<int>, thrust::device_ptr<double> > new_end;
		new_end = thrust::reduce_by_key(work_item_map_ptr, work_item_map_ptr+m_total, x_tmp_ptr, keys_output_ptr, batched_products_ptr, thrust::equal_to<int>(), thrust::plus<double>());

		// output_set_count is NOT equal to mat_mat_vec_data_count, since invalid entries are discarded
		*products_count = new_end.first - keys_output_ptr;

		thrust::replace_if(batched_products_ptr, batched_products_ptr+*products_count, stop_aca_as_soon_as_possible_ptr, thrust::identity<bool>(), 1.0/0.0);

		finalize_norm_computation<<<(*products_count + (block_size-1)) / block_size, block_size>>>(batched_products, *products_count);
		checkCUDAError("finalize_norm_computation");

		hipFree(keys_output);
		hipFree(x_tmp);
}


void batched_low_rank_mvp(double* x, double* y, double* U, double* V, int m1_total, int m2_total, int* m1_h, int* m2_h, int mat_vec_data_count, int batch_count, int k, int* k_per_item, hipblasStatus_t stat, hipblasHandle_t handle , int* point_map_offsets1_h, int* point_map_offsets2_h, int* point_map1, int* point_map2, int* work_item_map1 )
{
	int block_size = MATRIX_ENTRY_BLOCK_SIZE;

	thrust::device_ptr<int> point_map1_ptr(point_map1);
	thrust::device_ptr<int> point_map2_ptr(point_map2);

	// allocation and extraction of batched local operands
	double* local_x;
	hipMalloc((void**)&local_x, m2_total*sizeof(double));
	checkCUDAError("hipMalloc");
	thrust::device_ptr<double> local_x_ptr(local_x);
	thrust::device_ptr<double> x_ptr(x);
	thrust::gather(point_map2_ptr, point_map2_ptr+m2_total, x_ptr, local_x_ptr);


	// allocation of batched local intermediate results
	double* local_tmp;
	hipMalloc((void**)&local_tmp, batch_count*k*sizeof(double));
	checkCUDAError("hipMalloc");

	// allocation of batched local results
	double* local_y;
	hipMalloc((void**)&local_y, m1_total*sizeof(double));
	checkCUDAError("hipMalloc");



	// low-rank matrix-vector-product
	double one;
	double zero;
	one = 1.0;
	zero = 0.0;


	int* k_per_item_h;
	k_per_item_h = new int[mat_vec_data_count];
	hipMemcpy(k_per_item_h, k_per_item, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);

	int current_batch = 0;

	for (int s=0; s<mat_vec_data_count; s++)
	{
		if ((m2_h[s]>0)&&(m1_h[s]>0)) // check whether current work item is valid
		{
//			hipblasSetStream(handle, streams[current_batch]);

			hipblasDgemv(handle, HIPBLAS_OP_T, m2_h[s], k_per_item_h[s], &one, &V[point_map_offsets2_h[s]], m2_total, &local_x[point_map_offsets2_h[s]], 1, &zero, &local_tmp[current_batch*k], 1);
			hipblasDgemv(handle, HIPBLAS_OP_N, m1_h[s], k_per_item_h[s], &one, &U[point_map_offsets1_h[s]], m1_total, &local_tmp[k*current_batch], 1, &zero, &local_y[point_map_offsets1_h[s]], 1);
//			hipblasDgemv(handle, HIPBLAS_OP_T, m2_h[s], k, &one, &V[point_map_offsets2_h[s]], m2_total, &local_x[point_map_offsets2_h[s]], 1, &zero, &local_tmp[current_batch*k], 1);
//			hipblasDgemv(handle, HIPBLAS_OP_N, m1_h[s], k, &one, &U[point_map_offsets1_h[s]], m1_total, &local_tmp[k*current_batch], 1, &zero, &local_y[point_map_offsets1_h[s]], 1);

			current_batch++;
		}
	}

	delete[] k_per_item_h;


	thrust::device_ptr<double> local_y_ptr(local_y);
	thrust::device_ptr<double> y_ptr(y);

	// adding batched local results to full vector
	//thrust::transform(y_ptr+current_mat_vec_data.set1_l, y_ptr+current_mat_vec_data.set1_l+m1, local_y_ptr, y_ptr+current_mat_vec_data.set1_l, thrust::plus<double>());
	add_batched_local_results_to_full_vector<<<(m1_total + (block_size - 1)) / block_size, block_size>>>(y, local_y, point_map1, work_item_map1, m1_total);

////	TIME_ssstop("ACA apply");

	hipFree(local_x);
	hipFree(local_y);
	hipFree(local_tmp);



}



void batched_low_rank_mvp_magma(double* x, double* y, double* U, double* V, int m1_total, int m2_total, int* m1, int* m2, int mat_vec_data_count, int batch_count, int k, int* k_per_item, hipblasStatus_t stat, hipblasHandle_t handle, magma_queue_t* queue, int* point_map_offsets1_h, int* point_map_offsets2_h, int* point_map1, int* point_map2, int* work_item_map1 )
{
	int block_size = MATRIX_ENTRY_BLOCK_SIZE;

	thrust::device_ptr<int> point_map1_ptr(point_map1);
	thrust::device_ptr<int> point_map2_ptr(point_map2);

	// allocation and extraction of batched local operands
	double* local_x;
	hipMalloc((void**)&local_x, m2_total*sizeof(double));
	checkCUDAError("hipMalloc");
	thrust::device_ptr<double> local_x_ptr(local_x);
	thrust::device_ptr<double> x_ptr(x);
	thrust::gather(point_map2_ptr, point_map2_ptr+m2_total, x_ptr, local_x_ptr);


	// allocation of batched local intermediate results
	double* local_tmp;
	hipMalloc((void**)&local_tmp, batch_count*k*sizeof(double));
	checkCUDAError("hipMalloc");
	thrust::device_ptr<double> local_tmp_ptr(local_tmp);
	thrust::fill(local_tmp_ptr, local_tmp_ptr+(batch_count*k), 0.0);

	// allocation of batched local results
	double* local_y;
	hipMalloc((void**)&local_y, m1_total*sizeof(double));
	checkCUDAError("hipMalloc");
	thrust::device_ptr<double> local_y_ptr(local_y);
	thrust::fill(local_y_ptr, local_y_ptr+m1_total, 0.0);



//      TIME_ssstop("Preparing y_local");
//      TIME_ssstart;

        // batched matrix-vector-product
        double one;
        double zero;
        one = 1.0;
        zero = 0.0;

        int* incx;
        int* incy;
        int* ldda;
        hipMalloc((void**)&incx,sizeof(int)*(mat_vec_data_count+1));
        hipMalloc((void**)&incy,sizeof(int)*(mat_vec_data_count+1));
        hipMalloc((void**)&ldda,sizeof(int)*(mat_vec_data_count+1));

	checkCUDAError("cudaMalloc1");

        thrust::device_ptr<int> incx_ptr(incx);
        thrust::device_ptr<int> incy_ptr(incy);
	thrust::device_ptr<int> ldda_ptr(ldda);
        thrust::fill(incx_ptr, incx_ptr+(mat_vec_data_count+1), 1);
        thrust::fill(incy_ptr, incy_ptr+(mat_vec_data_count+1), 1);
	thrust::fill(ldda_ptr, ldda_ptr+(mat_vec_data_count+1), m2_total);

	double** dU_array_h = new double*[mat_vec_data_count+1];
	double** dV_array_h = new double*[mat_vec_data_count+1];
	double** dx_array_h = new double*[mat_vec_data_count+1];
	double** dtmp_array_h = new double*[mat_vec_data_count+1];
	double** dy_array_h = new double*[mat_vec_data_count+1];

	double** dU_array;
	double** dV_array;
	double** dx_array;
	double** dtmp_array;
	double** dy_array;

	hipMalloc((void**)&dU_array, (mat_vec_data_count+1)*sizeof(double*));
	hipMalloc((void**)&dV_array, (mat_vec_data_count+1)*sizeof(double*));
	hipMalloc((void**)&dx_array, (mat_vec_data_count+1)*sizeof(double*));
	hipMalloc((void**)&dtmp_array, (mat_vec_data_count+1)*sizeof(double*));
	hipMalloc((void**)&dy_array, (mat_vec_data_count+1)*sizeof(double*));
	checkCUDAError("cudaMalloc2");

	int current_batch = 0;

	int* m1_h = new int[mat_vec_data_count];
	int* m2_h = new int[mat_vec_data_count];
	hipMemcpy(m1_h, m1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(m2_h, m2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);



	for (int s=0; s<mat_vec_data_count; s++)
        {
                if ((m2_h[s]>0)&&(m1_h[s]>0)) // check whether current work item is valid
                {
			dV_array_h[current_batch] = &V[point_map_offsets2_h[s]];
			dU_array_h[current_batch] = &U[point_map_offsets1_h[s]];
			dx_array_h[current_batch] = &local_x[point_map_offsets2_h[s]];
			dtmp_array_h[current_batch] = &local_tmp[current_batch*k];
			dy_array_h[current_batch] = &local_y[point_map_offsets1_h[s]];

			current_batch++;
                }
		else
		{
			printf("\n\n\n KKKKKOOOOOMMMMIIIISSSSCCCCHHHHH \n\n\n"); fflush(stdout);
		}
        }

	delete [] m1_h;
	delete [] m2_h;

	hipMemcpy(dV_array, dV_array_h, (mat_vec_data_count)*sizeof(double*), hipMemcpyHostToDevice);
	hipMemcpy(dU_array, dU_array_h, (mat_vec_data_count)*sizeof(double*), hipMemcpyHostToDevice);
	hipMemcpy(dx_array, dx_array_h, (mat_vec_data_count)*sizeof(double*), hipMemcpyHostToDevice);
	hipMemcpy(dtmp_array, dtmp_array_h, (mat_vec_data_count)*sizeof(double*), hipMemcpyHostToDevice);
	hipMemcpy(dy_array, dy_array_h, (mat_vec_data_count)*sizeof(double*), hipMemcpyHostToDevice);

	checkCUDAError("hipMemcpy");

	delete [] dU_array_h;
	delete [] dV_array_h;
	delete [] dx_array_h;
	delete [] dtmp_array_h;
	delete [] dy_array_h;

//	printf("dgemv m2 %p , k_per_item %p, dV_array %p, ldda %p, dx_array%p , incx %p , dtmp_array, %p incy %p, current_batch %d\n", m2, k_per_item, dV_array, ldda, dx_array, incx, dtmp_array, incy, current_batch);

        magmablas_dgemv_vbatched( MagmaTrans, m2, k_per_item, one, dV_array, ldda, dx_array, incx, zero, dtmp_array, incy, current_batch, *queue);

	checkCUDAError("magmablas_dgemv_vbatched11");

	thrust::fill(ldda_ptr, ldda_ptr+(mat_vec_data_count+1), m1_total);

        magmablas_dgemv_vbatched( MagmaNoTrans, m1, k_per_item, one, dU_array, ldda, dtmp_array, incx, zero, dy_array, incy, current_batch, *queue);

	checkCUDAError("magmablas_dgemv_vbatched12");

	hipFree(dU_array);
	hipFree(dV_array);
	hipFree(dx_array);
	hipFree(dtmp_array);
	hipFree(dy_array);

	checkCUDAError("cudaFree1");

	hipFree(incx);
	hipFree(incy);
	hipFree(ldda);

	checkCUDAError("cudaFree2");

//	thrust::device_ptr<double> local_y_ptr(local_y);
	thrust::device_ptr<double> y_ptr(y);

	// adding batched local results to full vector
	//thrust::transform(y_ptr+current_mat_vec_data.set1_l, y_ptr+current_mat_vec_data.set1_l+m1, local_y_ptr, y_ptr+current_mat_vec_data.set1_l, thrust::plus<double>());
	add_batched_local_results_to_full_vector<<<(m1_total + (block_size - 1)) / block_size, block_size>>>(y, local_y, point_map1, work_item_map1, m1_total);
	hipDeviceSynchronize();
	checkCUDAError("add_batched_local_results_to_full_vector");

////	TIME_ssstop("ACA apply");

	hipFree(local_x);
	hipFree(local_y);
	hipFree(local_tmp);

	checkCUDAError("cudaFree3");

}



bool do_stop_based_on_batched_frobenius_norm(double* U, double* V, double* u_r, double* v_r, int m1_total, int m2_total, int* point_map_offsets1_h, int* point_map_offsets2_h, bool* stop_aca_as_soon_as_possible, bool* stop_aca_as_soon_as_possible_h, int* work_item_map1, int* work_item_map2, int batch_count, int r, int mat_vec_data_count, int* m1_h, int* m2_h, double eta, double epsilon, hipStream_t *streams, hipblasStatus_t stat, hipblasHandle_t handle )
{
			int block_size = 512;

	
			thrust::device_ptr<int> work_item_map1_ptr(work_item_map1);
			thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);
			thrust::device_ptr<bool> stop_aca_as_soon_as_possible_ptr(stop_aca_as_soon_as_possible);


			int current_batch = 0;

			// ATTENTION: C, D will have a different memory layout than U,V
			// U,V  [batch1_column1, batch2_column1, ... , batch1_column2, batch2_column2, ...]
			// C,D  [batch1_column1, batch1_column2, ... , batch2_column1, batch2_column2, ...]
			double* C;
			hipMalloc((void**)&C, batch_count*(r+1)*(r+1)*sizeof(double));

			double* D;
			hipMalloc((void**)&D, batch_count*(r+1)*(r+1)*sizeof(double));

			double one;
			double zero;
			one = 1.0;
			zero = 0.0;


			for (int s=0; s<mat_vec_data_count; s++)
			{
				if ((m2_h[s]>0)&&(m1_h[s]>0)) // check whether current work item is valid
				{
					if (!stop_aca_as_soon_as_possible_h[current_batch])
					{
						hipblasSetStream(handle, streams[current_batch]);

						// C = U'*U
						hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, (r+1), (r+1), m1_h[s], &one, &U[point_map_offsets1_h[s]], m1_total, &U[point_map_offsets1_h[s]], m1_total, &zero, &C[current_batch*(r+1)*(r+1)], (r+1));

						// D = V'*V
						hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, (r+1), (r+1), m2_h[s], &one, &V[point_map_offsets2_h[s]], m2_total, &V[point_map_offsets2_h[s]], m2_total, &zero, &D[current_batch*(r+1)*(r+1)], (r+1));
					}

					current_batch++;
				}
			}		// res = C(:)'*D(:)

			hipblasSetStream(handle, 0);

//		//	TIME_ssstop("ACA Frobenius C,D computation");

//		//	TIME_ssstart;

			double* res;
			hipMalloc((void**)&res, sizeof(double)*batch_count);
			thrust::device_ptr<double> res_ptr(res);
			double* res_h = new double[batch_count];

			double* u_r_2norm;
			double* v_r_2norm;
			hipMalloc((void**)&u_r_2norm, sizeof(double)*batch_count);
			hipMalloc((void**)&v_r_2norm, sizeof(double)*batch_count);
			thrust::device_ptr<double> u_r_2norm_ptr(u_r_2norm);
			thrust::device_ptr<double> v_r_2norm_ptr(v_r_2norm);
			double* u_r_2norm_h = new double[batch_count];
			double* v_r_2norm_h = new double[batch_count];


			// ------------------------------------------------------------
			// construct work map for batched dot product of k x k matrices
			// ------------------------------------------------------------
			int* offsets_of_kxk_matrices;
			int* ones_to_scatter;
			int* work_item_map_for_kxk_products;
			hipMalloc((void**)&offsets_of_kxk_matrices, sizeof(int)*batch_count);
			hipMalloc((void**)&ones_to_scatter, sizeof(int)*batch_count);
			hipMalloc((void**)&work_item_map_for_kxk_products, sizeof(int)*batch_count*(r+1)*(r+1));
			thrust::device_ptr<int> offsets_of_kxk_matrices_ptr(offsets_of_kxk_matrices);
			thrust::device_ptr<int> ones_to_scatter_ptr(ones_to_scatter);
			thrust::device_ptr<int> work_item_map_for_kxk_products_ptr(work_item_map_for_kxk_products);

			// compute offsets of the kxk matrix blocks
			thrust::sequence(offsets_of_kxk_matrices_ptr, offsets_of_kxk_matrices_ptr+batch_count, 0, (r+1)*(r+1));

			// fill array with ones to scatter
			thrust::fill(ones_to_scatter_ptr, ones_to_scatter_ptr+batch_count, 1);

			// fill work_item_map with zeros
			thrust::fill(work_item_map_for_kxk_products_ptr, work_item_map_for_kxk_products_ptr+batch_count*(r+1)*(r+1), 0);

			// scatter ones to beginnings of kxk matrix blocks
			thrust::scatter(ones_to_scatter_ptr, ones_to_scatter_ptr+batch_count, offsets_of_kxk_matrices_ptr, work_item_map_for_kxk_products_ptr);

			// create pattern of the form 11112222333344445555666677778888...
			thrust::inclusive_scan(work_item_map_for_kxk_products_ptr, work_item_map_for_kxk_products_ptr+batch_count*(r+1)*(r+1), work_item_map_for_kxk_products_ptr);

			hipFree(offsets_of_kxk_matrices);
			hipFree(ones_to_scatter);
			// ------------------------------------------------------------

			// compute Frobenius norms of kxk matrices
			int kxk_products_count;
			compute_batched_products_for_kxk_matrices(res, &kxk_products_count, C, D, batch_count*(r+1)*(r+1), work_item_map_for_kxk_products_ptr, block_size, stop_aca_as_soon_as_possible);

			hipFree(work_item_map_for_kxk_products);

//		//	TIME_ssstop("ACA Frobenius norm_eval (part 1)");
//		//	TIME_ssstart;

			int norm_count;
			compute_batched_norms(u_r_2norm, &norm_count, u_r, m1_total, work_item_map1_ptr, block_size);

			if (norm_count!=batch_count)
			{
				printf("Exiting: norm_count=%d, batch_count=%d\n", norm_count, batch_count);
				exit(1);
			}

			compute_batched_norms(v_r_2norm, &norm_count, v_r, m2_total, work_item_map2_ptr, block_size);

//		//	TIME_ssstop("ACA Frobenius norm_eval (part 2)");
//		//	TIME_ssstart;

			hipFree(C);
			hipFree(D);

			//		if (u_r_2norm*v_r_2norm <= ((1.0e-8*(1.0-eta))/(1.0+1.0e-8))*res)
			//		{
			//			//			printf("AAAAUUUUFFFHÖÖÖÖÖRRREEENNNN!!!!!! Schluss jetzt!\n");
			//			printf("r=%d\n", r);
			//			break;
			//		}
			thrust::transform(u_r_2norm_ptr, u_r_2norm_ptr+batch_count, v_r_2norm_ptr, u_r_2norm_ptr, thrust::multiplies<double>());
			thrust::transform(u_r_2norm_ptr, u_r_2norm_ptr+batch_count, res_ptr, u_r_2norm_ptr, thrust::divides<double>());
			thrust::replace_if(stop_aca_as_soon_as_possible_ptr, stop_aca_as_soon_as_possible_ptr+batch_count, u_r_2norm_ptr, is_smaller(((epsilon*(1.0-eta))/(1.0+epsilon))), true);

			hipMemcpy(stop_aca_as_soon_as_possible_h, stop_aca_as_soon_as_possible, batch_count*sizeof(bool), hipMemcpyDeviceToHost);

			hipFree(u_r_2norm);
			hipFree(v_r_2norm);
			hipFree(res);


			//				hipblasDgemv(handle, HIPBLAS_OP_T, m2_h[s], k_per_item_h[s], &one, &V[point_map_offsets2_h[s]], m2_total, &local_x[point_map_offsets2_h[s]], 1, &zero, &local_tmp[current_batch*k], 1);
			//				hipblasDgemv(handle, HIPBLAS_OP_N, m1_h[s], k_per_item_h[s], &one, &U[point_map_offsets1_h[s]], m1_total, &local_tmp[k*current_batch], 1, &zero, &local_y[point_map_offsets1_h[s]], 1);
			//			hipblasDgemv(handle, HIPBLAS_OP_T, m2_h[s], k, &one, &V[point_map_offsets2_h[s]], m2_total, &local_x[point_map_offsets2_h[s]], 1, &zero, &local_tmp[current_batch*k], 1);
			//			hipblasDgemv(handle, HIPBLAS_OP_N, m1_h[s], k, &one, &U[point_map_offsets1_h[s]], m1_total, &local_tmp[k*current_batch], 1, &zero, &local_y[point_map_offsets1_h[s]], 1);


//		//	TIME_ssstop("ACA Frobenius rest");

			bool stop = thrust::all_of(stop_aca_as_soon_as_possible_ptr, stop_aca_as_soon_as_possible_ptr+batch_count, thrust::identity<bool>());

			if (stop)
			{
				//			printf("r %d\n", r);
				return true;
			}
			else
				return false;

}

//--------------------------------------------------------------
// compute mapping of batch data entries to global point indices
//--------------------------------------------------------------
void compute_point_map(int* point_map1, int* point_map2, int m1_total, int m2_total, int* m1, int* m2, int* point_map_offsets1, int* point_map_offsets2, struct work_item* mat_vec_data, int mat_vec_data_count, int work_type)
{
	int block_size = 512;

	thrust::device_ptr<int> point_map1_ptr(point_map1);
	thrust::device_ptr<int> point_map2_ptr(point_map2);

	// start with one's
	thrust::fill(point_map1_ptr, point_map1_ptr+m1_total, 1);
	thrust::fill(point_map2_ptr, point_map2_ptr+m2_total, 1);

	// set index bounds to l, -(u-1)
	// 2  1 -3  5  1  1  1 -8  2  1  1 -4
	set_bounds_for_point_maps<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(point_map1, point_map2, point_map_offsets1, point_map_offsets2, m1, m2, work_type, mat_vec_data, mat_vec_data_count);

	// use inclusive scan to generate index map
	// 2  3  0  5  6  7  8  0  2  3  4  0  
	thrust::inclusive_scan(point_map1_ptr, point_map1_ptr+m1_total, point_map1_ptr);
	thrust::inclusive_scan(point_map2_ptr, point_map2_ptr+m2_total, point_map2_ptr);

	// correct upper bounds
	// 2  3  4  5  6  7  8  9  2  3  4  5 
	correct_bounds_for_point_maps<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(point_map1, point_map2, point_map_offsets1, point_map_offsets2, m1, m2, work_type, mat_vec_data, mat_vec_data_count);



}

//--------------------------------------------------------------
// compute mapping of batch data entries to global point indices
//--------------------------------------------------------------
void compute_point_map_and_pattern_with_padding(int* point_map2, int* pattern2, int m2_total, int* m2, int* point_map_offsets2, int padding2, struct work_item* mat_vec_data, int mat_vec_data_count)
{
	int block_size = 512;

	thrust::device_ptr<int> point_map2_ptr(point_map2);

	// input point sets: [2,5], [8,10]

	// start with one's
	// | 1  1  1  1  1  1 | 1  1  1  1  1  1 |
	thrust::fill(point_map2_ptr, point_map2_ptr+padding2*mat_vec_data_count, 1);
//	printf("point_map_with_padding\n");
//	print_int(point_map2, padding2*mat_vec_data_count);

	// set index bounds to l, -(u-1), and upper correction term to -((padding-(u-l+1)-1))
	// | 2  1  1 -4  1 -1 | 8  1 -9  1  1 -2 | 
	set_bounds_for_point_maps_with_padding<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(point_map2, m2, padding2, WT_DENSE, mat_vec_data, mat_vec_data_count);
//	printf("point_map_with_padding\n");
//	print_int(point_map2, padding2*mat_vec_data_count);

	// use inclusive scan to generate index map
	// | 2  3  4  0  1  0 | 8  9  0  1  2  0 |  
	thrust::inclusive_scan(point_map2_ptr, point_map2_ptr+padding2*mat_vec_data_count, point_map2_ptr);
//	printf("point_map_with_padding\n");
//	print_int(point_map2, padding2*mat_vec_data_count);

	// correct upper bounds
	// | 2  3  4  5  1  0 | 8  9 10  1  2  0 | 
	correct_bounds_for_point_maps_with_padding<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(point_map2, m2, padding2, WT_DENSE, mat_vec_data, mat_vec_data_count);
//	printf("point_map_with_padding\n");
//	print_int(point_map2, padding2*mat_vec_data_count);

	// padding part will be ignored based on a pattern, thus we get
	// | 2  3  4  5  *  * | 8  9 10  *  *  * |


	// compute pattern to ignore padding

	thrust::device_ptr<int> pattern2_ptr(pattern2);

	// start with zeros
	// | 0  0  0  0  0  0 | 0  0  0  0  0  0 |
	thrust::fill(pattern2_ptr, pattern2_ptr+padding2*mat_vec_data_count, 0);

	// set bounds
	// | 1  0  0 -1  0  0 | 1  0 -1  0  0  0 |
        set_bounds_for_pattern_with_padding<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(pattern2, m2, padding2, WT_DENSE, mat_vec_data, mat_vec_data_count);	
	
	// do inclusive scan to create main part of pattern
	// | 1  1  1  0  0  0 | 1  1  0  0  0  0 |
	thrust::inclusive_scan(pattern2_ptr, pattern2_ptr+padding2*mat_vec_data_count, pattern2_ptr);

	// correct upper bounds
	// | 1  1  1  1  0  0 | 1  1  1  0  0  0 |
        correct_bounds_for_pattern_with_padding<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(pattern2, m2, padding2, WT_DENSE, mat_vec_data, mat_vec_data_count);

	// done!

//	printf("pattern\n");
//	print_int(pattern2, padding2*mat_vec_data_count);

}


// --------------------------------------------------------------
// compute mapping of rows in batched data to index in work_queue	
// --------------------------------------------------------------
void compute_work_item_maps(int* work_item_map1, int* work_item_map2, int m1_total, int m2_total, int* point_map_offsets1, int* point_map_offsets2, int* m1, int* m2, struct work_item* mat_vec_data, int mat_vec_data_count, int work_type)
{
	int block_size = 512;

	thrust::device_ptr<int> work_item_map1_ptr(work_item_map1);
	thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);

	// set maps to zero
	// 0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0  0
	thrust::fill(work_item_map1_ptr, work_item_map1_ptr+m1_total, 0);
	thrust::fill(work_item_map2_ptr, work_item_map2_ptr+m2_total, 0);

	// set bounds for the back mapping of rows to work_items
	// 0  0  2  0 -2  0  0  3  0  0  0 -3  1  0  0 -1  0
	set_bounds_for_work_item_maps<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(work_item_map1, work_item_map2, point_map_offsets1, point_map_offsets2, m1, m2, work_type, mat_vec_data, mat_vec_data_count); // TODO: Do I need m1, m2 here? Should be implicitely available by point_map_offsets

	// fill gaps
	// 0  0  2  2  0  0  0  3  3  3  3  0  1  1  1  0  0
	thrust::inclusive_scan(work_item_map1_ptr, work_item_map1_ptr+m1_total, work_item_map1_ptr);
	thrust::inclusive_scan(work_item_map2_ptr, work_item_map2_ptr+m2_total, work_item_map2_ptr);

	// correct upper bounds
	// 0  0  2  2  2  0  0  3  3  3  3  3  1  1  1  1  0
	correct_bounds_for_work_item_maps<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(work_item_map1, work_item_map2, point_map_offsets1, point_map_offsets2, m1, m2, work_type, mat_vec_data, mat_vec_data_count);

}


// ------------------------------------------------------------------------------------------------------------
// creating map between work item list (including invalid entries) and batch set list (without invalid entries)
// ------------------------------------------------------------------------------------------------------------
void compute_work_item_to_batch_map(int* work_item_to_batch_map, struct work_item* mat_vec_data, int mat_vec_data_count, int* batch_count, int work_type)
{
	thrust::device_ptr<int> work_item_to_batch_map_ptr(work_item_to_batch_map);
	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);

	int* tmp_field;
	hipMalloc((void**)&tmp_field, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> tmp_field_ptr(tmp_field);

	// fill tmp_field with sequence
	//              tmp_field =  0  1  2  3  4  5
	thrust::sequence(tmp_field_ptr, tmp_field_ptr+mat_vec_data_count);

	// remove invalid entries
	//              tmp_field =  1  2  4
	thrust::device_ptr<int> end_after_removal;
	if (work_type==WT_ACA)
		end_after_removal = thrust::remove_if(tmp_field_ptr, tmp_field_ptr+mat_vec_data_count, mat_vec_data_ptr, is_not_WT_ACA());
	else
		end_after_removal = thrust::remove_if(tmp_field_ptr, tmp_field_ptr+mat_vec_data_count, mat_vec_data_ptr, is_not_WT_DENSE());
	
	*batch_count = end_after_removal-tmp_field_ptr;

	// set map to -1
	// work_item_to_batch_map = -1 -1 -1 -1 -1 -1
	thrust::fill(work_item_to_batch_map_ptr, work_item_to_batch_map_ptr+mat_vec_data_count, -1);

	// scatter sequence to appropriate positions (indicated by tmp_field)
	// work_item_to_batch_map = -1  0  1 -1  2 -1
	thrust::scatter(thrust::make_counting_iterator(0), thrust::make_counting_iterator(*batch_count), tmp_field_ptr, work_item_to_batch_map_ptr);
	hipFree(tmp_field);

}


void compute_m1_m2(int* m1, int* m2, struct work_item* mat_vec_data, int mat_vec_data_count, int work_type)
{
	int block_size = 512;

	int* l1;
	int* u1;
	hipMalloc((void**)&l1, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&u1, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> l1_ptr(l1);
	thrust::device_ptr<int> u1_ptr(u1);

	int* l2;
	int* u2;
	hipMalloc((void**)&l2, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&u2, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> l2_ptr(l2);
	thrust::device_ptr<int> u2_ptr(u2);

	get_work_item_point_set_limits_for_given_type<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(l1, u1, work_type, mat_vec_data, mat_vec_data_count, 1);
	hipDeviceSynchronize();
	checkCUDAError("get_work_item_point_set_limits");

	get_work_item_point_set_limits_for_given_type<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(l2, u2, work_type, mat_vec_data, mat_vec_data_count, 2);
	hipDeviceSynchronize();
	checkCUDAError("get_work_item_point_set_limits");

	thrust::device_ptr<int> m1_ptr(m1);
	thrust::device_ptr<int> m2_ptr(m2);

	// getting matrix sizes
	thrust::transform(u1_ptr, u1_ptr+mat_vec_data_count, l1_ptr, m1_ptr, minus_plus_1()); // numbers of rows
	thrust::transform(u2_ptr, u2_ptr+mat_vec_data_count, l2_ptr, m2_ptr, minus_plus_1()); // numbers of columns

	// l1, u1, l2, u2 are no longer needed
	hipFree(l1);
	hipFree(l2);
	hipFree(u1);
	hipFree(u2);


}

void create_maps_and_indices(int* m1, int* m2, int m1_total, int m2_total, int* point_map_offsets1, int* point_map_offsets2, int* point_map1, int* point_map2, int* work_item_map1, int* work_item_map2, int* work_item_to_batch_map, int* batch_count, struct work_item* mat_vec_data, int mat_vec_data_count, int work_type)
{

	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);
	thrust::device_ptr<int> m1_ptr(m1);
	thrust::device_ptr<int> m2_ptr(m2);


	// ------------------------------------------------------
	// create mapping of work_items to offset in batched data
	// ------------------------------------------------------
	thrust::device_ptr<int> point_map_offsets1_ptr(point_map_offsets1);
	thrust::device_ptr<int> point_map_offsets2_ptr(point_map_offsets2);
	thrust::exclusive_scan(m1_ptr, m1_ptr+mat_vec_data_count, point_map_offsets1_ptr, 0);
	thrust::exclusive_scan(m2_ptr, m2_ptr+mat_vec_data_count, point_map_offsets2_ptr, 0);

	//--------------------------------------------------------------
	// compute mapping of batch data entries to global point indices
	//--------------------------------------------------------------
	thrust::device_ptr<int> point_map1_ptr(point_map1);
	thrust::device_ptr<int> point_map2_ptr(point_map2);

	compute_point_map(point_map1, point_map2, m1_total, m2_total, m1, m2, point_map_offsets1, point_map_offsets2, mat_vec_data, mat_vec_data_count, work_type);

	// --------------------------------------------------------------
	// compute mapping of rows in batched data to index in work_queue	
	// --------------------------------------------------------------
	thrust::device_ptr<int> work_item_map1_ptr(work_item_map1);
	thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);

	compute_work_item_maps(work_item_map1, work_item_map2, m1_total, m2_total, point_map_offsets1, point_map_offsets2, m1, m2, mat_vec_data, mat_vec_data_count, work_type);

	// ------------------------------------------------------------------------------------------------------------
	// creating map between work item list (including invalid entries) and batch set list (without invalid entries)
	// ------------------------------------------------------------------------------------------------------------
	thrust::device_ptr<int> work_item_to_batch_map_ptr(work_item_to_batch_map);

	compute_work_item_to_batch_map(work_item_to_batch_map, mat_vec_data, mat_vec_data_count, batch_count, work_type);



}

__global__ void check_for_maximum_i_r(bool* search_for_new_v_r, bool* stop_full_aca_for_batch, int* i_r, int* m1, int mat_vec_data_count)
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx>=mat_vec_data_count)
		return;

	if ((!search_for_new_v_r[idx])||(stop_full_aca_for_batch[idx]))
		return;

	if (i_r[idx]>=m1[idx]-1)
	{
		search_for_new_v_r[idx] = false;
		stop_full_aca_for_batch[idx] = true;
	}
}

struct is_true
{
	__host__ __device__ bool operator()(bool b)
	{
		return b == true;
	}
};



void compute_current_batched_v_r(double* v_r, double* U, double* V, int m1_total, int m2_total, struct work_item* mat_vec_data, int mat_vec_data_count, bool* search_for_new_v_r, int* i_r, int* point_map1, int* point_map2, int* point_map_offsets1, int* point_map_offsets2, int* work_item_map2, struct point_set* input_set1, struct point_set* input_set2, int* k_per_item, int* m1, int r, struct system_assembler* assem, int k_max, bool* stop_full_aca_for_batch)
{

	int block_size = MATRIX_ENTRY_BLOCK_SIZE;

	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);
	thrust::device_ptr<bool> search_for_new_v_r_ptr(search_for_new_v_r);
	thrust::device_ptr<int> i_r_ptr(i_r);
	thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);
	thrust::device_ptr<int> k_per_item_ptr(k_per_item);
	thrust::device_ptr<bool> stop_full_aca_for_batch_ptr(stop_full_aca_for_batch);

	int* keys_output;
	double* v_r_norms;
	hipMalloc((void**)&keys_output, m2_total*sizeof(int));
	hipMalloc((void**)&v_r_norms, m2_total*sizeof(double));
	thrust::device_ptr<int> keys_output_ptr(keys_output);
	thrust::device_ptr<double> v_r_norms_ptr(v_r_norms);

	checkCUDAError("hipMalloc");

	// assuming that I start computing in all batches
	thrust::fill(search_for_new_v_r_ptr, search_for_new_v_r_ptr+mat_vec_data_count, true);
	// ... except if I shall not compute on a batch
	is_true it;
	thrust::replace_if(search_for_new_v_r_ptr, search_for_new_v_r_ptr+mat_vec_data_count, stop_full_aca_for_batch_ptr, it, false);


////	TIME_ssstop("ACA beginning of r loop");

////	TIME_ssstart;

	while (true)
	{
		// sanity check whether batch-wise i_r would hits upper bound of m1 (number of rows) after next increase; 
		// if this is the case, stop calculation for corresponding batch
		check_for_maximum_i_r<<<(mat_vec_data_count+(block_size-1))/block_size, block_size>>>(search_for_new_v_r, stop_full_aca_for_batch, i_r, m1, mat_vec_data_count);	
		hipDeviceSynchronize();
		checkCUDAError("check_for_maximum_i_r");

		// increase i_r entry for all elements on which we shall still compute
		thrust::transform_if(i_r_ptr, i_r_ptr+mat_vec_data_count, search_for_new_v_r_ptr, i_r_ptr, add_one(), is_true());
		checkCUDAError("__transform_if");

//		printf("i_r:\n");
//		print_int(i_r, mat_vec_data_count);

//		//    v_tilde_r = kernel(input_set1(i_r,:), input_set2);
//		batched_fill_kernel_vector_v_r<<<(m2_total + (block_size - 1)) / block_size, block_size>>>(v_r, point_map2, point_map1, point_map_offsets1, work_item_map2, i_r, compute_v_r, input_set2, input_set1, m2_total);
//		hipDeviceSynchronize();
//		checkCUDAError("fill_kernel_vector1");
//
//		//    for l=1:r-1
//	    //        v_tilde_r = v_tilde_r - U(i_r,l) * V(l,:);
//	    //    end
//		batched_scaled_substraction_for_v_r<<<(m2_total + (block_size - 1)) / block_size, block_size>>>(v_r, point_map2, point_map_offsets1, work_item_map2, i_r, compute_v_r, V, U, input_set2, input_set1, k_per_item, r, m2_total, m1_total);
//		hipDeviceSynchronize();
//		checkCUDAError("batched_scaled_substraction_of_vectors");
//
		batched_fill_kernel_vector_and_scaled_substraction_for_v_r<<<(m2_total + (block_size - 1)) / block_size, block_size>>>(v_r, point_map2, point_map1, point_map_offsets1, work_item_map2, i_r, search_for_new_v_r, input_set2, input_set1, m2_total, m1_total, V, U, r, k_per_item, assem);
		hipDeviceSynchronize();
		checkCUDAError("__batched_fill_kernel_vector_v_r");

//		// computing norms of batched vectors (to check whether to increase i_r)
		int v_r_norms_count;
		compute_batched_norms_with_keys_output(v_r_norms, keys_output, &v_r_norms_count, v_r, m2_total, work_item_map2_ptr, block_size);


// there are no invalid entries by construction
//		// remove potential rubbish in invalid entries
//		thrust::replace_if(v_r_norms_ptr, v_r_norms_ptr+output_set_counts, keys_output_ptr, is_minus_one(), 0.0);
			
		remove_rubbish_from_maxima<<<(v_r_norms_count + (block_size-1)) / block_size, block_size>>>(search_for_new_v_r, mat_vec_data_count, keys_output, v_r_norms, v_r_norms_count);
		hipDeviceSynchronize();
		checkCUDAError("remove_rubbish_from_maxima");


		// if a work_item has finished, set respective compute_v_r entry to 0
		update_ir<<<(v_r_norms_count + (block_size-1)) / block_size, block_size>>>(i_r, search_for_new_v_r, mat_vec_data_count, keys_output, v_r_norms, v_r_norms_count, m1);
		hipDeviceSynchronize();
		checkCUDAError("update_ir");


		bool continue_search_for_new_v_r;
		continue_search_for_new_v_r = thrust::reduce(search_for_new_v_r_ptr, search_for_new_v_r_ptr+mat_vec_data_count, false, thrust::logical_or<bool>());

//		print_int(compute_v_r, mat_vec_data_count);
//		printf("max: %d\n", max_of_compute_v_r);


		if (!continue_search_for_new_v_r)
			break;
//		// stop iteration when no v_r is (almost) zero
//		if (thrust::all_of(v_r_norms_ptr, v_r_norms_ptr+output_set_counts, bigger_than_eps()))
//			break;

	} //while (sqrt(thrust::inner_product(v_r_ptr, v_r_ptr+m2, v_r_ptr, 0.0))<1.0e-14);


		

////	TIME_ssstop("ACA v_r computation loop");

////	TIME_ssstart;

	hipFree(keys_output);
	checkCUDAError("hipFree");
	hipFree(v_r_norms);
	checkCUDAError("hipFree");

}

void compute_current_batched_u_r(double* u_r, double* v_r, double* U, double* V, int m1_total, int m2_total, struct work_item* mat_vec_data, int mat_vec_data_count, int* point_map1, int* point_map2, int* work_item_map1, int* work_item_map2, struct point_set* input_set1, struct point_set* input_set2, int* k_per_item, int* j_r_global, int* work_item_to_batch_map, int r, struct system_assembler* assem, bool* stop_full_aca_for_batch)
{

		int block_size = MATRIX_ENTRY_BLOCK_SIZE;

		thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);
		thrust::device_ptr<double> v_r_ptr(v_r);
		thrust::device_ptr<int> j_r_global_ptr(j_r_global);

		double* maximum_values;
		hipMalloc((void**)&maximum_values, mat_vec_data_count*sizeof(double));
		checkCUDAError("hipMalloc");
		thrust::device_ptr<double> maximum_values_ptr(maximum_values);

		int* batch_to_work_item_map; // maps batch set to work item number
		hipMalloc((void**)&batch_to_work_item_map, mat_vec_data_count*sizeof(double));  // mat_vec_data_count is just an upper bound
		checkCUDAError("hipMalloc");
		thrust::device_ptr<int> batch_to_work_item_map_ptr(batch_to_work_item_map);

//	//	TIME_ssstop("ACA u_r computation 2");
//	//	TIME_ssstart;

		// allocate and fill "indices" with 0, 1, 2, ...
		int* indices;
		hipMalloc((void**)&indices, m2_total*sizeof(double));
		checkCUDAError("hipMalloc");
		thrust::device_ptr<int> indices_ptr(indices);
		thrust::sequence(indices_ptr, indices_ptr+m2_total);

		// compute block-wise maximum and maximum positions at the same time
		thrust::pair<thrust::device_ptr<int>, thrust::zip_iterator<thrust::tuple<thrust::device_ptr<double>,thrust::device_ptr<int> > > > new_end2;
		new_end2 = thrust::reduce_by_key(work_item_map2_ptr, work_item_map2_ptr+m2_total, thrust::make_zip_iterator(thrust::make_tuple(v_r_ptr, indices_ptr)), batch_to_work_item_map_ptr, thrust::make_zip_iterator(thrust::make_tuple(maximum_values_ptr, j_r_global_ptr)), thrust::equal_to<int>(), tuple_absolute_maximum());

		// ATTENTION: In the following, I assume that the output size is identical to mat_vec_data_count, which is mandatory!!!

		hipFree(maximum_values);
		hipFree(batch_to_work_item_map);

//	//	TIME_ssstop("ACA u_r computation 3");
//	//	TIME_ssstart;

// not necessary since they all are valid by construction
//		// invalidate j_r_global entries in case they belong to invalid work items
//		thrust::replace_if(j_r_global_ptr, j_r_global_ptr+mat_vec_data_count, out_keys_ptr, is_minus_one(), -1);


	    // v_r = (1.0./(v_tilde_k(j_r)))*v_tilde_r;
		batched_scaling_of_v_r<<<(m2_total + (block_size-1)) / block_size, block_size>>>(v_r, work_item_to_batch_map, work_item_map2, k_per_item, r, j_r_global, m2_total, stop_full_aca_for_batch);
		hipDeviceSynchronize();
		checkCUDAError("batched_scaling_of_v_r");

	    // u_r = kernel(input_set1(:,:),input_set2(j_r,:));
	    // for l=1:r-1
	    //     u_r = u_r - V(l,j_r) * U(:,l);
	    // end


//	//	TIME_ssstop("ACA u_r computation 4");
//	//	TIME_ssstart;

//		batched_fill_kernel_vector_u_r<<<(m1_total + (block_size-1)) / block_size, block_size>>>(u_r, point_map1, point_map2, work_item_to_batch_map, work_item_map1, k_per_item, r, j_r_global, input_set1, input_set2, m1_total);
//		hipDeviceSynchronize();
//		checkCUDAError("batched_fill_kernel_vector_u_r");
//		batched_scaled_substraction_for_u_r<<<(m1_total + (block_size-1)) / block_size, block_size>>>(u_r, point_map1, work_item_to_batch_map, work_item_map1, j_r_global, U, V, input_set1, input_set2, k_per_item, r, m1_total, m2_total);
//		hipDeviceSynchronize();
//		checkCUDAError("batched_scaled_substraction_for_u_r");

		batched_fill_kernel_vector_and_scaled_substraction_for_u_r<<<(m1_total + (block_size-1)) / block_size, block_size>>>(u_r, point_map1, point_map2, work_item_to_batch_map, work_item_map1, k_per_item, r, j_r_global, input_set1, input_set2, m1_total, m2_total, U, V, assem, stop_full_aca_for_batch);
		hipDeviceSynchronize();
		checkCUDAError("batched_fill_kernel_vector_and_scaled_substraction_for_u_r");


	hipFree(indices);

}


void apply_batched_aca(double* x, double* y, struct work_item* mat_vec_data, int mat_vec_data_count, struct point_set* input_set1, struct point_set* input_set2, hipblasStatus_t stat, hipblasHandle_t handle, double eta, double epsilon, int k, struct system_assembler* assem)
{
	int block_size = 512;

	
	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);

//	{
//	size_t free_mem, total_mem;
//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("1:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0));
//	}



	// ------------------------
	// compute sizes of batches
	// ------------------------
	int* m1;
	int* m2;
	hipMalloc((void**)&m1, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&m2, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> m1_ptr(m1);
	thrust::device_ptr<int> m2_ptr(m2);
	compute_m1_m2(m1, m2, mat_vec_data, mat_vec_data_count, WT_ACA);
	int m1_total;
	int m2_total;
	m1_total = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count);
	m2_total = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count);


	// -------------------------------
	// generate all the necessary maps
	// -------------------------------
	int batch_count;

	int* point_map_offsets1;  // mapping of work_items to offset in batched data
	int* point_map_offsets2;
	int* point_map1; // map of rows of U to point indices in point_set1
	int* point_map2; // map of rows of V to point indices in point_set2
	int* work_item_map1; // map of rows of U to work item indices in mat_vec_data
	int* work_item_map2; // map of rows of V to work item indices in mat_vec_data
	int* work_item_to_batch_map;  // map between work item list (including invalid entries) and batch set list (without invalid entries)

	hipMalloc((void**)&point_map_offsets1, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map_offsets2, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map1, m1_total*sizeof(int));
	hipMalloc((void**)&point_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_map1, m1_total*sizeof(int));
	hipMalloc((void**)&work_item_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_to_batch_map, mat_vec_data_count*sizeof(int));

	thrust::device_ptr<int> point_map_offsets1_ptr(point_map_offsets1);
	thrust::device_ptr<int> point_map_offsets2_ptr(point_map_offsets2);
	thrust::device_ptr<int> point_map1_ptr(point_map1);
	thrust::device_ptr<int> point_map2_ptr(point_map2);
	thrust::device_ptr<int> work_item_map1_ptr(work_item_map1);
	thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);
	thrust::device_ptr<int> work_item_to_batch_map_ptr(work_item_to_batch_map);

	create_maps_and_indices(m1, m2, m1_total, m2_total, point_map_offsets1, point_map_offsets2, point_map1, point_map2, work_item_map1, work_item_map2, work_item_to_batch_map, &batch_count, mat_vec_data, mat_vec_data_count, WT_ACA);


	// -----------------------------------------------------
	// create local copies of some of the index / map fields
	// -----------------------------------------------------
	int* m1_h;
	int* m2_h;
	int* point_map_offsets2_h;
	int* point_map_offsets1_h;
	
	m1_h = new int[mat_vec_data_count];
	m2_h = new int[mat_vec_data_count];
	point_map_offsets2_h = new int[mat_vec_data_count];
	point_map_offsets1_h = new int[mat_vec_data_count];
	
	hipMemcpy(m1_h, m1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(m2_h, m2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets1_h, point_map_offsets1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets2_h, point_map_offsets2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);



	//--------------------------
	// compute the "k" per batch
	//--------------------------
	int* k_per_item;
	hipMalloc((void**)&k_per_item, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> k_per_item_ptr(k_per_item);
	// if (k>min(m,n))
	//     k= min(m,n);
	// end
	set_k_per_item<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(k_per_item, k, mat_vec_data_count, m1, m2);
	hipDeviceSynchronize();
	checkCUDAError("set_k_per_item");


	//-----------------------------
	// set upper bound for global k
	//-----------------------------
	int m1_max = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
	int m2_max = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
	if (k>min(m1_max, m2_max))
	{
		k = min(m1_max, m2_max);
	}

//	printf("MAX: m1 %d  m2 %d\n", m1_max, m2_max);


	//-------------------------------
	// allocate and init batched U, V
	//-------------------------------
	double* U;
	hipMalloc((void**)&U, m1_total*k*sizeof(double));  
	checkCUDAError("hipMalloc");
	double* V;
	hipMalloc((void**)&V, m2_total*k*sizeof(double));
	checkCUDAError("hipMalloc");

	thrust::device_ptr<double> U_ptr(U);
	thrust::device_ptr<double> V_ptr(V);

	// TODO: Fill this with nan -> algo should still work
	thrust::fill(U_ptr, U_ptr+m1_total*k, 0.0);
	thrust::fill(V_ptr, V_ptr+m2_total*k, 0.0);


	double* v_r;
	double* u_r;





	// i_r = 0;
//	int i_r = -1;

	int* i_r;
	hipMalloc((void**)&i_r, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> i_r_ptr(i_r);

	thrust::fill(i_r_ptr, i_r_ptr+mat_vec_data_count, -1);
	// on invalid entries (in mat_vec_data) we shall never compute
	thrust::replace_if(i_r_ptr, i_r_ptr+mat_vec_data_count, mat_vec_data_ptr, is_not_WT_ACA(), -1);

	bool* search_for_new_v_r;
	hipMalloc((void**)&search_for_new_v_r, mat_vec_data_count*sizeof(bool));
//	thrust::device_ptr<int> compute_v_r_ptr(compute_v_r);

  	
//	hipStream_t *streams = new hipStream_t[batch_count];
//	for(int b=0; b<batch_count; b++)
//        	hipStreamCreate(&streams[b]);

/*
	bool* stop_aca_as_soon_as_possible;
	hipMalloc((void**)&stop_aca_as_soon_as_possible, batch_count*sizeof(bool));
	thrust::device_ptr<bool> stop_aca_as_soon_as_possible_ptr(stop_aca_as_soon_as_possible);

	thrust::fill(stop_aca_as_soon_as_possible_ptr, stop_aca_as_soon_as_possible_ptr+batch_count, false);

	bool* stop_aca_as_soon_as_possible_h;
	stop_aca_as_soon_as_possible_h = new bool[batch_count];
	hipMemcpy(stop_aca_as_soon_as_possible_h, stop_aca_as_soon_as_possible, batch_count*sizeof(bool), hipMemcpyDeviceToHost);
*/


        bool* stop_full_aca_for_batch;
        hipMalloc((void**)&stop_full_aca_for_batch, mat_vec_data_count*sizeof(bool));
        thrust::device_ptr<bool> stop_full_aca_for_batch_ptr(stop_full_aca_for_batch);
        thrust::fill(stop_full_aca_for_batch_ptr, stop_full_aca_for_batch_ptr+mat_vec_data_count, false);

        // if it is not a valid batch, I will certainly not start computing
        thrust::replace_if(stop_full_aca_for_batch_ptr, stop_full_aca_for_batch_ptr+mat_vec_data_count, mat_vec_data_ptr, is_not_WT_ACA(), true);



        // for r=1:k
        for (int r=0; r<k; r++)
        {
                // while (norm(v_tilde_r,Inf)==0.0)
            //    i_r = i_r+1;
            //    v_tilde_r = kernel(input_set1(i_r,:), input_set2);
            //    for l=1:r-1
            //        v_tilde_r = v_tilde_r - U(i_r,l) * V(l,:);
            //    end
            // end

        // U = [U u_r];
        // V = [V; v_r];
                v_r = &V[r*m2_total];
                u_r = &U[r*m1_total];
                thrust::device_ptr<double> u_r_ptr(u_r);
                thrust::device_ptr<double> v_r_ptr(v_r);


                // if r>=k_per_item[batch_item] on the batch_item'th batch, don't do anything on this batch
                is_smaller_or_equal_r ser(r);
                thrust::replace_if(stop_full_aca_for_batch_ptr, stop_full_aca_for_batch_ptr+mat_vec_data_count, k_per_item_ptr, ser, true);


                compute_current_batched_v_r(v_r, U, V, m1_total, m2_total, mat_vec_data, mat_vec_data_count, search_for_new_v_r, i_r, point_map1, point_map2, point_map_offsets1, point_map_offsets2, work_item_map2, input_set1, input_set2, k_per_item, m1, r, assem, k, stop_full_aca_for_batch);


                //// [m,j_r] = max(abs(v_tilde_r));
                //thrust::device_ptr<double> max_pos = thrust::max_element(v_r_ptr, v_r_ptr+m2, compare_absolute());
                //int j_r = max_pos - v_r_ptr;

                int* j_r_global; // j_r index (maximum positions) as global indices in the batched vector
                hipMalloc((void**)&j_r_global, mat_vec_data_count*sizeof(int));  // mat_vec_data_count is an upper bound to the actual amount of batches
                checkCUDAError("hipMalloc");
                thrust::device_ptr<int> j_r_global_ptr(j_r_global);

                compute_current_batched_u_r(u_r, v_r, U, V, m1_total, m2_total, mat_vec_data, mat_vec_data_count, point_map1, point_map2, work_item_map1, work_item_map2, input_set1, input_set2, k_per_item, j_r_global, work_item_to_batch_map, r, assem, stop_full_aca_for_batch);


                hipFree(j_r_global);


//              bool check_frobenius = false;
//
//              if (check_frobenius && (r%5==0))
//              {
//
//                      bool stop = do_stop_based_on_batched_frobenius_norm(*U, *V, u_r, v_r, m1_total, m2_total, point_map_offsets1_h, point_map_offsets2_h, stop_aca_as_soon_as_possible, stop_aca_as_soon_as_possible_h, work_item_map1, work_item_map2, batch_count, r, mat_vec_data_count, m1_h, m2_h, eta, epsilon, streams, stat, handle );
//
//                      if (stop)
//                      {
//                              break;
//                      }
//
//      
//              }
        }

        hipFree(stop_full_aca_for_batch);



	hipFree(work_item_to_batch_map);
	hipFree(i_r);
	hipFree(search_for_new_v_r);
/*
	hipFree(stop_aca_as_soon_as_possible);
	delete [] stop_aca_as_soon_as_possible_h;
*/



//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("2:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

	hipFree(m1);
	hipFree(m2);
	hipFree(point_map_offsets1);
	hipFree(point_map_offsets2);

//	TIME_sssstart;

	// apply low-rank matrix-vector product
	batched_low_rank_mvp(x, y, U, V, m1_total, m2_total, m1_h, m2_h, mat_vec_data_count, batch_count, k, k_per_item, stat, handle , point_map_offsets1_h, point_map_offsets2_h, point_map1, point_map2, work_item_map1 );
//	TIME_sssstop("batched_low_rank_mvp");


//	hipblasSetStream(handle, 0);
//	for (int b=0; b<batch_count; b++)
//		hipStreamDestroy(streams[b]);
//	delete[] streams;

	delete [] m1_h;
	delete [] m2_h;
	delete [] point_map_offsets1_h;
	delete [] point_map_offsets2_h;


	hipFree(U);
	hipFree(V);
	hipFree(k_per_item);
	hipFree(point_map1);
	hipFree(point_map2);
	hipFree(work_item_map1);
	hipFree(work_item_map2);
	checkCUDAError("cudaFrees a the end of batched ACA");


//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("3:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

}

void apply_batched_dense(double* x, double* y, struct work_item* mat_vec_data, int mat_vec_data_count, struct point_set* input_set1, struct point_set* input_set2, hipblasStatus_t stat, hipblasHandle_t handle, struct system_assembler* assem, double* S)
{
	int block_size = MATRIX_ENTRY_BLOCK_SIZE;

	
	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);

//	{
//	size_t free_mem, total_mem;
//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("1:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0));
//	}

//	TIME_ssstart;

	// ------------------------
	// compute sizes of batches
	// ------------------------
	int* m1;
	int* m2;
	hipMalloc((void**)&m1, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&m2, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> m1_ptr(m1);
	thrust::device_ptr<int> m2_ptr(m2);
	compute_m1_m2(m1, m2, mat_vec_data, mat_vec_data_count, WT_DENSE);
	int m1_total;
	int m2_total;
	m1_total = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count);
	m2_total = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count);

//	TIME_ssstop("Indexing a.1");
//	TIME_ssstart;



	// -------------------------------
	// generate all the necessary maps
	// -------------------------------
	int batch_count;

	int* point_map_offsets1;  // mapping of work_items to offset in batched data
	int* point_map_offsets2;
	int* point_map1; // map of rows of U to point indices in point_set1
	int* point_map2; // map of rows of V to point indices in point_set2
	int* work_item_map1; // map of rows of U to work item indices in mat_vec_data
	int* work_item_map2; // map of rows of V to work item indices in mat_vec_data
	int* work_item_to_batch_map;  // map between work item list (including invalid entries) and batch set list (without invalid entries)
//	TIME_ssstop("Indexing a.1");

//	printf("%d %d %d\n",mat_vec_data_count, m1_total, m2_total);


//	TIME_ssstart;
	hipMalloc((void**)&point_map_offsets1, mat_vec_data_count*sizeof(int));
//	TIME_ssstop("Indexing a.1a");
//	TIME_ssstart;
	hipMalloc((void**)&point_map_offsets2, mat_vec_data_count*sizeof(int));
//	TIME_ssstop("Indexing a.1b");
//	TIME_ssstart;
	hipMalloc((void**)&point_map1, m1_total*sizeof(int));
//	TIME_ssstop("Indexing a.1c");
//	TIME_ssstart;
	hipMalloc((void**)&point_map2, m2_total*sizeof(int));
//	TIME_ssstop("Indexing a.1d");
//	TIME_ssstart;
	hipMalloc((void**)&work_item_map1, m1_total*sizeof(int));
//	TIME_ssstop("Indexing a.1e");
//	TIME_ssstart;
	hipMalloc((void**)&work_item_map2, m2_total*sizeof(int));
//	TIME_ssstop("Indexing a.1f");
//	TIME_ssstart;
	hipMalloc((void**)&work_item_to_batch_map, mat_vec_data_count*sizeof(int));
//	TIME_ssstop("Indexing a.1g");
//	TIME_ssstart;


	thrust::device_ptr<int> point_map_offsets1_ptr(point_map_offsets1);
	thrust::device_ptr<int> point_map_offsets2_ptr(point_map_offsets2);
	thrust::device_ptr<int> point_map1_ptr(point_map1);
	thrust::device_ptr<int> point_map2_ptr(point_map2);
	thrust::device_ptr<int> work_item_map1_ptr(work_item_map1);
	thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);
	thrust::device_ptr<int> work_item_to_batch_map_ptr(work_item_to_batch_map);

//	TIME_ssstop("Indexing a.2");
//	TIME_ssstart;

	create_maps_and_indices(m1, m2, m1_total, m2_total, point_map_offsets1, point_map_offsets2, point_map1, point_map2, work_item_map1, work_item_map2, work_item_to_batch_map, &batch_count, mat_vec_data, mat_vec_data_count, WT_DENSE);

//	TIME_ssstop("Indexing b");
//	TIME_ssstart;

	// -----------------------------------------------------
	// create local copies of some of the index / map fields
	// -----------------------------------------------------
	int* m1_h;
	int* m2_h;
	int* point_map_offsets2_h;
	int* point_map_offsets1_h;
	
	m1_h = new int[mat_vec_data_count];
	m2_h = new int[mat_vec_data_count];
	point_map_offsets2_h = new int[mat_vec_data_count];
	point_map_offsets1_h = new int[mat_vec_data_count];
	
	hipMemcpy(m1_h, m1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(m2_h, m2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets1_h, point_map_offsets1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets2_h, point_map_offsets2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);

//	for (int i=0; i<mat_vec_data_count; i++)
//	{
//		printf("m1_h[%d] %d   m2_h[%d] %d\n", i, m1_h[i], i, m2_h[i]);
//	}


	//------------------------------
	// get maximal m1,m2 for padding
	//------------------------------
	int m1_max = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
	int m2_max = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count, 0, thrust::maximum<int>());

//	TIME_ssstop("Indexing c");	
//	TIME_ssstart;

//	// --------------------------------------
//	// compute lookup table to gether local x
//	// --------------------------------------
//	int* point_map_with_padding;
//	hipMalloc((void**)&point_map_with_padding, m2_max*mat_vec_data_count*sizeof(int));
//
//	int* pattern_with_padding;
//	hipMalloc((void**)&pattern_with_padding, m2_max*mat_vec_data_count*sizeof(int));
//
//	thrust::device_ptr<int> point_map_with_padding_ptr(point_map_with_padding);
//	thrust::device_ptr<int> pattern_with_padding_ptr(pattern_with_padding);
//
//	compute_point_map_and_pattern_with_padding(point_map_with_padding, pattern_with_padding, m2_total, m2, point_map_offsets2, m2_max, mat_vec_data, mat_vec_data_count);

	//------------------------------------------
	// allocate and init batched dense subblocks
	//------------------------------------------

//	double* S;
//	hipMalloc((void**)&S, (m1_total*m2_total)*sizeof(double));  
//	checkCUDAError("hipMalloc");

//	TIME_ssstop("FILLING.1")
//	TIME_ssstart;

	thrust::device_ptr<double> S_ptr(S);

//	thrust::fill(S_ptr, S_ptr+m1_total*m2_total, 0.0);

	hipMemset(S, 0, m1_total*m2_total*sizeof(double));
//	TIME_ssstop("FILLING.2")
//	TIME_ssstart;
	
	//-------------------------------------------
	// fill batched dense subblocks
	//-------------------------------------------

//	printf("m1_total %d  m2_max %d\n", m1_total, m2_max);

//	printf("fill_batched_matrix %d %d\n",(m1_total*m2_max + (block_size - 1)) / block_size, block_size);

	fill_batched_matrix<<<(m1_total*m2_max + (block_size - 1)) / block_size, block_size>>>(S, mat_vec_data, input_set1, input_set2, m1, m2, m1_total, point_map1, work_item_map1, point_map_offsets1, point_map_offsets2, m2_max, assem );
	hipDeviceSynchronize();
	checkCUDAError("fill_batched_matrix");

//	fill_batched_matrix<<<(m1_total*m2_total + (block_size - 1)) / block_size, block_size>>>(S, mat_vec_data, input_set1, input_set2, m1, m2, m1_total, m2_total, point_map1, work_item_map1, point_map_offsets1, point_map_offsets2, m2_max, kernel_type );
//	hipDeviceSynchronize();
//	checkCUDAError("fill_batched_matrix");




//	TIME_ssstop("FILLING.3")
//	TIME_ssstart;
	

//        thrust::device_ptr<int> point_map1_ptr(point_map1);
//        thrust::device_ptr<int> point_map2_ptr(point_map2);

	//-------------------------------------------------
        // allocation and extraction of batched local operands including padding
	//------------------------------------------------

	// allocation of local x (including padding)
        double* local_x;
        hipMalloc((void**)&local_x, m2_total*sizeof(double));
        checkCUDAError("hipMalloc");

	// getting pointers
        thrust::device_ptr<double> local_x_ptr(local_x);
        thrust::device_ptr<double> x_ptr(x);

	thrust::fill(local_x_ptr, local_x_ptr+m2_max*mat_vec_data_count, 0.0);

        thrust::gather(point_map2_ptr, point_map2_ptr+m2_total, x_ptr, local_x_ptr);

//	TIME_ssstop("Getting x_local");
//	TIME_ssstart;

        // allocation of batched local results
        double* local_y;
        hipMalloc((void**)&local_y, m1_total*sizeof(double));
        checkCUDAError("hipMalloc");

        // batched matrix-vector-product
        double one;
        double zero;
        one = 1.0;
        zero = 0.0;

	hipblasDgemv(handle, HIPBLAS_OP_N, m1_total, m2_total, &one, S, m1_total, local_x, 1, &zero, local_y, 1);

//	TIME_ssstop("MVP");
//	TIME_ssstart;

        thrust::device_ptr<double> local_y_ptr(local_y);
        thrust::device_ptr<double> y_ptr(y);

        // adding batched local results to full vector
        //thrust::transform(y_ptr+current_mat_vec_data.set1_l, y_ptr+current_mat_vec_data.set1_l+m1, local_y_ptr, y_ptr+current_mat_vec_data.set1_l, thrust::plus<double>());
        add_batched_local_results_to_full_vector<<<(m1_total + (block_size - 1)) / block_size, block_size>>>(y, local_y, point_map1, work_item_map1, m1_total);

//      TIME_ssstop("ACA apply");


        hipFree(local_x);
        hipFree(local_y);

//	TIME_ssstop("Update");

//	hipFree(point_map_with_padding);
//	hipFree(pattern_with_padding);


	//-------------------------------------------------
	//

	hipFree(work_item_to_batch_map);




//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("2:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

	hipFree(m1);
	hipFree(m2);
	hipFree(point_map_offsets1);
	hipFree(point_map_offsets2);




	delete [] m1_h;
	delete [] m2_h;
	delete [] point_map_offsets1_h;
	delete [] point_map_offsets2_h;


//	hipFree(S);
	hipFree(point_map1);
	hipFree(point_map2);
	hipFree(work_item_map1);
	hipFree(work_item_map2);
	checkCUDAError("cudaFrees a the end of batched ACA");


//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("3:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

}

__global__ void set_dA_array(double** dA_array, double* hA, int mat_vec_data_count, int* matrix_offsets)
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= mat_vec_data_count)
                return;

	dA_array[idx] = hA + matrix_offsets[idx];

	return;
}

void apply_batched_dense_magma(double* x, double* y, struct work_item* mat_vec_data, int mat_vec_data_count, struct point_set* input_set1, struct point_set* input_set2, hipblasStatus_t stat, hipblasHandle_t handle, struct system_assembler* assem, magma_queue_t* queue, double* hA, bool use_precomputed_data)
{
	int block_size = MATRIX_ENTRY_BLOCK_SIZE;

	if (mat_vec_data_count==0)
		return;
	
	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);

//	{
//	size_t free_mem, total_mem;
//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("1:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0));
//	}

//	TIME_ssstart;

	// ------------------------
	// compute sizes of batches
	// ------------------------
	int* m1;
	int* m2;
	hipMalloc((void**)&m1, (mat_vec_data_count+1)*sizeof(int));
	hipMalloc((void**)&m2, (mat_vec_data_count+1)*sizeof(int));
	thrust::device_ptr<int> m1_ptr(m1);
	thrust::device_ptr<int> m2_ptr(m2);
	compute_m1_m2(m1, m2, mat_vec_data, mat_vec_data_count, WT_DENSE);
	int m1_total;
	int m2_total;
	m1_total = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count);
	m2_total = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count);

/*
	int* matrix_sizes;
	hipMalloc((void**)&matrix_sizes, (mat_vec_data_count)*sizeof(int));
	thrust::device_ptr<int> matrix_sizes_ptr(matrix_sizes);
	thrust::transform(m1_ptr, m1_ptr+mat_vec_data_count, m2_ptr, matrix_sizes_ptr, thrust::multiplies<int>());
	int* matrix_offsets;
	hipMalloc((void**)&matrix_offsets, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> matrix_offsets_ptr(matrix_offsets);
	thrust::exclusive_scan(matrix_sizes_ptr, matrix_sizes_ptr+mat_vec_data_count, matrix_offsets_ptr);	
	int total_batched_matrix_size = thrust::reduce(matrix_sizes_ptr, matrix_sizes_ptr+mat_vec_data_count);
*/
	// -------------------------------
	// generate all the necessary maps
	// -------------------------------
	int batch_count;

	int* point_map_offsets1;  // mapping of work_items to offset in batched data
	int* point_map_offsets2;
	int* point_map1; // map of rows of U to point indices in point_set1
	int* point_map2; // map of rows of V to point indices in point_set2
	int* work_item_map1; // map of rows of U to work item indices in mat_vec_data
	int* work_item_map2; // map of rows of V to work item indices in mat_vec_data
	int* work_item_to_batch_map;  // map between work item list (including invalid entries) and batch set list (without invalid entries)

//	printf("%d %d %d\n",mat_vec_data_count, m1_total, m2_total);


	hipMalloc((void**)&point_map_offsets1, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map_offsets2, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map1, m1_total*sizeof(int));
	hipMalloc((void**)&point_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_map1, m1_total*sizeof(int));
	hipMalloc((void**)&work_item_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_to_batch_map, mat_vec_data_count*sizeof(int));


	thrust::device_ptr<int> point_map_offsets1_ptr(point_map_offsets1);
	thrust::device_ptr<int> point_map_offsets2_ptr(point_map_offsets2);
	thrust::device_ptr<int> point_map1_ptr(point_map1);
	thrust::device_ptr<int> point_map2_ptr(point_map2);
	thrust::device_ptr<int> work_item_map1_ptr(work_item_map1);
	thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);
	thrust::device_ptr<int> work_item_to_batch_map_ptr(work_item_to_batch_map);


	create_maps_and_indices(m1, m2, m1_total, m2_total, point_map_offsets1, point_map_offsets2, point_map1, point_map2, work_item_map1, work_item_map2, work_item_to_batch_map, &batch_count, mat_vec_data, mat_vec_data_count, WT_DENSE);


	// -----------------------------------------------------
	// create local copies of some of the index / map fields
	// -----------------------------------------------------
	int* m1_h;
	int* m2_h;
	int* point_map_offsets2_h;
	int* point_map_offsets1_h;
	
	m1_h = new int[mat_vec_data_count];
	m2_h = new int[mat_vec_data_count];
	point_map_offsets2_h = new int[mat_vec_data_count];
	point_map_offsets1_h = new int[mat_vec_data_count];
	
	hipMemcpy(m1_h, m1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(m2_h, m2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets1_h, point_map_offsets1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets2_h, point_map_offsets2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);


//	//------------------------------
//	// get maximal m1,m2 for padding
//	//------------------------------
	int m1_max = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
	int m2_max = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count, 0, thrust::maximum<int>());

//	printf("MAX: m1 %d  m2 %d\n", m1_max, m2_max);

//	TIME_ssstop("Indexing c");	
//	TIME_ssstart;

	
	//-------------------------------------------
	// fill batched dense subblocks
	//-------------------------------------------

//	printf("m1_total %d  m2_max %d\n", m1_total, m2_max);

//	printf("fill_batched_matrix %d %d\n",(m1_total*m2_max + (block_size - 1)) / block_size, block_size);

	double** hx_array = new double*[mat_vec_data_count];
	double** hy_array = new double*[mat_vec_data_count];

//	TIME_ssstop("FILLING.0.1")
//	TIME_ssstart;

//	printf("comparison: %d %d\n", total_batched_matrix_size, m1_total*m2_max);

//	TIME_ssstop("FILLING.0.2")
//	TIME_ssstart;


	int* matrix_offsets;
	hipMalloc((void**)&matrix_offsets, (mat_vec_data_count+1)*sizeof(int));
	thrust::device_ptr<int> matrix_offsets_ptr(matrix_offsets);
	thrust::multiplies<int> op;
	thrust::transform(m1_ptr, m1_ptr+mat_vec_data_count, m2_ptr, matrix_offsets_ptr, op);
	int total_size = reduce(matrix_offsets_ptr, matrix_offsets_ptr+mat_vec_data_count);
	thrust::exclusive_scan(matrix_offsets_ptr, matrix_offsets_ptr+mat_vec_data_count, matrix_offsets_ptr);



	double** dA_array;
	hipMalloc((void**)&dA_array, mat_vec_data_count*sizeof(double*));
	
	set_dA_array<<<(mat_vec_data_count+(block_size-1)) / block_size, block_size>>>(dA_array, hA, mat_vec_data_count, matrix_offsets);
	hipDeviceSynchronize();
	checkCUDAError("set_dA_array");

//	TIME_ssstop("FILLING.0.3")
//	TIME_ssstart;

	double** dx_array;
	double** dy_array;
	hipMalloc((void**)&dx_array, mat_vec_data_count*sizeof(double*));
	hipMalloc((void**)&dy_array, mat_vec_data_count*sizeof(double*));

	checkCUDAError("hipMalloc");
	
	if (!use_precomputed_data)
	{
	//	TIME_ssstop("FILLING.1")
//		TIME_ssstart;
	
		fill_batched_matrix_magma<<<(total_size + (block_size - 1)) / block_size, block_size>>>(hA, mat_vec_data, input_set1, input_set2, m1, m2, total_size, matrix_offsets, mat_vec_data_count, assem);
		hipDeviceSynchronize();
		checkCUDAError("fill_batched_matrix_magma");
	
//		TIME_ssstop("FILLING.2")
	}
//	TIME_ssstart;

	//-------------------------------------------------
        // allocation and extraction of batched local operands including padding
	//------------------------------------------------

	// allocation of local x (including padding)
        double* local_x;
        hipMalloc((void**)&local_x, m2_total*sizeof(double));
        checkCUDAError("hipMalloc");

	// getting pointers
        thrust::device_ptr<double> local_x_ptr(local_x);
        thrust::device_ptr<double> x_ptr(x);

	thrust::fill(local_x_ptr, local_x_ptr+m2_total, 0.0);

        thrust::gather(point_map2_ptr, point_map2_ptr+m2_total, x_ptr, local_x_ptr);

	double* current_pointer = local_x;

	for (int i=0; i<mat_vec_data_count; i++)
	{
		hx_array[i] = current_pointer;
		current_pointer=current_pointer+m2_h[i];
	}
	hipMemcpy(dx_array, hx_array, mat_vec_data_count*sizeof(double*), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy");

//	TIME_ssstop("Getting x_local");
//	TIME_ssstart;


        // allocation of batched local results
        double* local_y;
        hipMalloc((void**)&local_y, m1_total*sizeof(double));
        checkCUDAError("hipMalloc");

	current_pointer = local_y;

	thrust::device_ptr<double> local_y_ptr(local_y);

	// need to fill local_y with zeros since magma seems not to ignore y as input if beta=0 (bug in Magma)
	thrust::fill(local_y_ptr, local_y_ptr+m1_total, 0.0);

	for (int i=0; i<mat_vec_data_count; i++)
	{
		hy_array[i] = current_pointer;
		current_pointer=current_pointer+m1_h[i];
	}
	hipMemcpy(dy_array, hy_array, mat_vec_data_count*sizeof(double*), hipMemcpyHostToDevice);

//	TIME_ssstop("Preparing y_local");
//	TIME_ssstart;

        // batched matrix-vector-product
        double one;
        double zero;
        one = 1.0;
        zero = 0.0;

	int* incx;
	int* incy;
	int* ldda;	
	hipMalloc((void**)&incx,sizeof(int)*(mat_vec_data_count+1));
	hipMalloc((void**)&incy,sizeof(int)*(mat_vec_data_count+1));
	hipMalloc((void**)&ldda,sizeof(int)*(mat_vec_data_count+1));
	hipMemcpy(ldda, m1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToDevice);
	thrust::device_ptr<int> incx_ptr(incx);
	thrust::device_ptr<int> incy_ptr(incy);
	thrust::fill(incx_ptr, incx_ptr+(mat_vec_data_count+1), 1);
	thrust::fill(incy_ptr, incy_ptr+(mat_vec_data_count+1), 1);
	
//	TIME_ssstop("Magma init");
//	TIME_ssstart;

	magmablas_dgemv_vbatched( MagmaNoTrans, m1, m2, one, dA_array, ldda, dx_array, incx, zero, dy_array, incy, mat_vec_data_count, *queue);

//	TIME_ssstop("MVP");
//	TIME_ssstart;


        thrust::device_ptr<double> y_ptr(y);

        // adding batched local results to full vector
        //thrust::transform(y_ptr+current_mat_vec_data.set1_l, y_ptr+current_mat_vec_data.set1_l+m1, local_y_ptr, y_ptr+current_mat_vec_data.set1_l, thrust::plus<double>());
        add_batched_local_results_to_full_vector<<<(m1_total + (block_size - 1)) / block_size, block_size>>>(y, local_y, point_map1, work_item_map1, m1_total);


        hipFree(local_x);
        hipFree(local_y);

//	TIME_ssstop("Update");
//	TIME_ssstart;

	//-------------------------------------------------
	//

//	hipFree(matrix_sizes);
//	hipFree(matrix_offsets);

	hipFree(work_item_to_batch_map);

	hipFree(dx_array);
	hipFree(dy_array);
	hipFree(dA_array);
	hipFree(matrix_offsets);
	hipFree(incx);
	hipFree(incy);
	hipFree(ldda);

	delete [] hx_array;
	delete [] hy_array;


//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("2:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

	hipFree(m1);
	hipFree(m2);
	hipFree(point_map_offsets1);
	hipFree(point_map_offsets2);




	delete [] m1_h;
	delete [] m2_h;
	delete [] point_map_offsets1_h;
	delete [] point_map_offsets2_h;


	hipFree(point_map1);
	hipFree(point_map2);
	hipFree(work_item_map1);
	hipFree(work_item_map2);
	checkCUDAError("cudaFrees a the end of batched ACA");


//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("3:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

//	TIME_ssstop("frees");


}


void precompute_batched_dense_magma(struct work_item* mat_vec_data, int mat_vec_data_count, struct point_set* input_set1, struct point_set* input_set2, hipblasStatus_t stat, hipblasHandle_t handle, struct system_assembler* assem, magma_queue_t* queue, double* hA)
{
	int block_size = MATRIX_ENTRY_BLOCK_SIZE;

	if (mat_vec_data_count == 0)
		return;

	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);

//	{
//	size_t free_mem, total_mem;
//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("1:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0));
//	}

//	TIME_ssstart;

	// ------------------------
	// compute sizes of batches
	// ------------------------
	int* m1;
	int* m2;
	hipMalloc((void**)&m1, (mat_vec_data_count+1)*sizeof(int));
	hipMalloc((void**)&m2, (mat_vec_data_count+1)*sizeof(int));
	thrust::device_ptr<int> m1_ptr(m1);
	thrust::device_ptr<int> m2_ptr(m2);
	compute_m1_m2(m1, m2, mat_vec_data, mat_vec_data_count, WT_DENSE);
	int m1_total;
	int m2_total;
	m1_total = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count);
	m2_total = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count);

/*
	int* matrix_sizes;
	hipMalloc((void**)&matrix_sizes, (mat_vec_data_count)*sizeof(int));
	thrust::device_ptr<int> matrix_sizes_ptr(matrix_sizes);
	thrust::transform(m1_ptr, m1_ptr+mat_vec_data_count, m2_ptr, matrix_sizes_ptr, thrust::multiplies<int>());
	int* matrix_offsets;
	hipMalloc((void**)&matrix_offsets, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> matrix_offsets_ptr(matrix_offsets);
	thrust::exclusive_scan(matrix_sizes_ptr, matrix_sizes_ptr+mat_vec_data_count, matrix_offsets_ptr);	
	int total_batched_matrix_size = thrust::reduce(matrix_sizes_ptr, matrix_sizes_ptr+mat_vec_data_count);
*/
	// -------------------------------
	// generate all the necessary maps
	// -------------------------------
	int batch_count;

	int* point_map_offsets1;  // mapping of work_items to offset in batched data
	int* point_map_offsets2;
	int* point_map1; // map of rows of U to point indices in point_set1
	int* point_map2; // map of rows of V to point indices in point_set2
	int* work_item_map1; // map of rows of U to work item indices in mat_vec_data
	int* work_item_map2; // map of rows of V to work item indices in mat_vec_data
	int* work_item_to_batch_map;  // map between work item list (including invalid entries) and batch set list (without invalid entries)

//	printf("%d %d %d\n",mat_vec_data_count, m1_total, m2_total);


	hipMalloc((void**)&point_map_offsets1, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map_offsets2, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map1, m1_total*sizeof(int));
	hipMalloc((void**)&point_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_map1, m1_total*sizeof(int));
	hipMalloc((void**)&work_item_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_to_batch_map, mat_vec_data_count*sizeof(int));


	thrust::device_ptr<int> point_map_offsets1_ptr(point_map_offsets1);
	thrust::device_ptr<int> point_map_offsets2_ptr(point_map_offsets2);
	thrust::device_ptr<int> point_map1_ptr(point_map1);
	thrust::device_ptr<int> point_map2_ptr(point_map2);
	thrust::device_ptr<int> work_item_map1_ptr(work_item_map1);
	thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);
	thrust::device_ptr<int> work_item_to_batch_map_ptr(work_item_to_batch_map);


	create_maps_and_indices(m1, m2, m1_total, m2_total, point_map_offsets1, point_map_offsets2, point_map1, point_map2, work_item_map1, work_item_map2, work_item_to_batch_map, &batch_count, mat_vec_data, mat_vec_data_count, WT_DENSE);


	// -----------------------------------------------------
	// create local copies of some of the index / map fields
	// -----------------------------------------------------
	int* m1_h;
	int* m2_h;
	int* point_map_offsets2_h;
	int* point_map_offsets1_h;
	
	m1_h = new int[mat_vec_data_count];
	m2_h = new int[mat_vec_data_count];
	point_map_offsets2_h = new int[mat_vec_data_count];
	point_map_offsets1_h = new int[mat_vec_data_count];
	
	hipMemcpy(m1_h, m1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(m2_h, m2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets1_h, point_map_offsets1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets2_h, point_map_offsets2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);


//	//------------------------------
//	// get maximal m1,m2 for padding
//	//------------------------------
//	int m1_max = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
//	int m2_max = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count, 0, thrust::maximum<int>());

//	TIME_ssstop("Indexing c");	
//	TIME_ssstart;

	
	//-------------------------------------------
	// fill batched dense subblocks
	//-------------------------------------------

//	printf("m1_total %d  m2_max %d\n", m1_total, m2_max);

//	printf("fill_batched_matrix %d %d\n",(m1_total*m2_max + (block_size - 1)) / block_size, block_size);

//	TIME_ssstop("FILLING.0.1")
//	TIME_ssstart;

//	printf("comparison: %d %d\n", total_batched_matrix_size, m1_total*m2_max);

//	TIME_ssstop("FILLING.0.2")
//	TIME_ssstart;

        int* matrix_offsets;
        hipMalloc((void**)&matrix_offsets, (mat_vec_data_count+1)*sizeof(int));
        thrust::device_ptr<int> matrix_offsets_ptr(matrix_offsets);
        thrust::multiplies<int> op;
        thrust::transform(m1_ptr, m1_ptr+mat_vec_data_count, m2_ptr, matrix_offsets_ptr, op);
        int total_size = reduce(matrix_offsets_ptr, matrix_offsets_ptr+mat_vec_data_count);
        thrust::exclusive_scan(matrix_offsets_ptr, matrix_offsets_ptr+mat_vec_data_count, matrix_offsets_ptr);


        double** dA_array;
        hipMalloc((void**)&dA_array, mat_vec_data_count*sizeof(double*));

        set_dA_array<<<(mat_vec_data_count+(block_size-1)) / block_size, block_size>>>(dA_array, hA, mat_vec_data_count, matrix_offsets);
        hipDeviceSynchronize();
        checkCUDAError("set_dA_array");

//	TIME_ssstop("FILLING.0.3")
//	TIME_ssstart;

//	TIME_ssstop("FILLING.1")
//	TIME_ssstart;

//        printf("grid_size %d block_size %d\n",(total_size + (block_size - 1)) / block_size, block_size);
//	printf("hA %p mat_vec_data %p input_set1 %p input_set2 %p m1 %p m2 %p total_size %d  matrix_offsets %p mat_vec_data_count %d assem %p\n", hA, mat_vec_data, input_set1, input_set2, m1, m2, total_size, matrix_offsets, mat_vec_data_count, assem);
//
//	print_work_items(mat_vec_data, mat_vec_data_count);
//	print_int(matrix_offsets, mat_vec_data_count);
//
//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("2:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }


	fill_batched_matrix_magma<<<(total_size + (block_size - 1)) / block_size, block_size>>>(hA, mat_vec_data, input_set1, input_set2, m1, m2, total_size, matrix_offsets, mat_vec_data_count, assem);
        hipDeviceSynchronize();
        checkCUDAError("fill_batched_matrix_magma");


//	TIME_ssstop("FILLING.2")
//	TIME_ssstart;


//	hipFree(matrix_sizes);
//	hipFree(matrix_offsets);

	hipFree(work_item_to_batch_map);

	hipFree(matrix_offsets);

	hipFree(dA_array);


//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("2:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

	hipFree(m1);
	hipFree(m2);
	hipFree(point_map_offsets1);
	hipFree(point_map_offsets2);




	delete [] m1_h;
	delete [] m2_h;
	delete [] point_map_offsets1_h;
	delete [] point_map_offsets2_h;


	hipFree(point_map1);
	hipFree(point_map2);
	hipFree(work_item_map1);
	hipFree(work_item_map2);
	checkCUDAError("cudaFrees a the end of batched ACA");


//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("3:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

//	TIME_ssstop("frees");


}


void apply_precomputed_batched_aca_magma(double* x, double* y, struct work_item* mat_vec_data, int mat_vec_data_count, struct point_set* input_set1, struct point_set* input_set2, hipblasStatus_t stat, hipblasHandle_t handle, magma_queue_t* queue, double eta, double epsilon, int k, double* U, double* V)
{
	int block_size = 512;

	if (mat_vec_data_count==0)
		return;

	
	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);

//	{
//	size_t free_mem, total_mem;
//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("1:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0));
//	}



	// ------------------------
	// compute sizes of batches
	// ------------------------
	int* m1;
	int* m2;
	hipMalloc((void**)&m1, (mat_vec_data_count+1)*sizeof(int));
	hipMalloc((void**)&m2, (mat_vec_data_count+1)*sizeof(int));
	thrust::device_ptr<int> m1_ptr(m1);
	thrust::device_ptr<int> m2_ptr(m2);
	compute_m1_m2(m1, m2, mat_vec_data, mat_vec_data_count, WT_ACA);
	int m1_total;
	int m2_total;
	m1_total = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count);
	m2_total = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count);


	// -------------------------------
	// generate all the necessary maps
	// -------------------------------
	int batch_count;

	int* point_map_offsets1;  // mapping of work_items to offset in batched data
	int* point_map_offsets2;
	int* point_map1; // map of rows of U to point indices in point_set1
	int* point_map2; // map of rows of V to point indices in point_set2
	int* work_item_map1; // map of rows of U to work item indices in mat_vec_data
	int* work_item_map2; // map of rows of V to work item indices in mat_vec_data
	int* work_item_to_batch_map;  // map between work item list (including invalid entries) and batch set list (without invalid entries)

	hipMalloc((void**)&point_map_offsets1, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map_offsets2, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map1, m1_total*sizeof(int));
	hipMalloc((void**)&point_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_map1, m1_total*sizeof(int));
	hipMalloc((void**)&work_item_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_to_batch_map, mat_vec_data_count*sizeof(int));

	thrust::device_ptr<int> point_map_offsets1_ptr(point_map_offsets1);
	thrust::device_ptr<int> point_map_offsets2_ptr(point_map_offsets2);
	thrust::device_ptr<int> point_map1_ptr(point_map1);
	thrust::device_ptr<int> point_map2_ptr(point_map2);
	thrust::device_ptr<int> work_item_map1_ptr(work_item_map1);
	thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);
	thrust::device_ptr<int> work_item_to_batch_map_ptr(work_item_to_batch_map);

	create_maps_and_indices(m1, m2, m1_total, m2_total, point_map_offsets1, point_map_offsets2, point_map1, point_map2, work_item_map1, work_item_map2, work_item_to_batch_map, &batch_count, mat_vec_data, mat_vec_data_count, WT_ACA);


	// -----------------------------------------------------
	// create local copies of some of the index / map fields
	// -----------------------------------------------------
	int* m1_h;
	int* m2_h;
	int* point_map_offsets2_h;
	int* point_map_offsets1_h;
	
	m1_h = new int[mat_vec_data_count];
	m2_h = new int[mat_vec_data_count];
	point_map_offsets2_h = new int[mat_vec_data_count];
	point_map_offsets1_h = new int[mat_vec_data_count];
	
	hipMemcpy(m1_h, m1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(m2_h, m2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets1_h, point_map_offsets1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets2_h, point_map_offsets2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);



	//--------------------------
	// compute the "k" per batch
	//--------------------------
	int* k_per_item;
	hipMalloc((void**)&k_per_item, (mat_vec_data_count+1)*sizeof(int));
	thrust::device_ptr<int> k_per_item_ptr(k_per_item);
	// if (k>min(m,n))
	//     k= min(m,n);
	// end
	set_k_per_item<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(k_per_item, k, mat_vec_data_count, m1, m2);
	hipDeviceSynchronize();
	checkCUDAError("set_k_per_item");


	//-----------------------------
	// set upper bound for global k
	//-----------------------------
	int m1_max = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
	int m2_max = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
	if (k>min(m1_max, m2_max))
	{
		k = min(m1_max, m2_max);
	}


	thrust::device_ptr<double> U_ptr(U);
	thrust::device_ptr<double> V_ptr(V);

//	hipStream_t *streams = new hipStream_t[batch_count];
//	for(int b=0; b<batch_count; b++)
  //      	hipStreamCreate(&streams[b]);




//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("2:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

	hipFree(work_item_to_batch_map);
	hipFree(point_map_offsets1);
	hipFree(point_map_offsets2);

//	TIME_sssstart;

	// apply low-rank matrix-vector product
	batched_low_rank_mvp_magma(x, y, U, V, m1_total, m2_total, m1, m2, mat_vec_data_count, batch_count, k, k_per_item, stat, handle, queue, point_map_offsets1_h, point_map_offsets2_h, point_map1, point_map2, work_item_map1 );

//	TIME_sssstop("batched_low_rank_mvp");


//	hipblasSetStream(handle, 0);
//	for (int b=0; b<batch_count; b++)
//		hipStreamDestroy(streams[b]);
//	delete[] streams;

	delete [] m1_h;
	delete [] m2_h;
	delete [] point_map_offsets1_h;
	delete [] point_map_offsets2_h;

	hipFree(m1);
	hipFree(m2);

	hipFree(k_per_item);
	hipFree(point_map1);
	hipFree(point_map2);
	hipFree(work_item_map1);
	hipFree(work_item_map2);
	checkCUDAError("cudaFrees a the end of batched ACA");


//      {
//       size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("3:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

}


void apply_precomputed_batched_aca(double* x, double* y, struct work_item* mat_vec_data, int mat_vec_data_count, struct point_set* input_set1, struct point_set* input_set2, hipblasStatus_t stat, hipblasHandle_t handle, double eta, double epsilon, int k, double* U, double* V)
{
	int block_size = 512;

	if (mat_vec_data_count==0)
		return;
	
	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);

//	{
//	size_t free_mem, total_mem;
//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("1:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0));
//	}



	// ------------------------
	// compute sizes of batches
	// ------------------------
	int* m1;
	int* m2;
	hipMalloc((void**)&m1, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&m2, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> m1_ptr(m1);
	thrust::device_ptr<int> m2_ptr(m2);
	compute_m1_m2(m1, m2, mat_vec_data, mat_vec_data_count, WT_ACA);
	int m1_total;
	int m2_total;
	m1_total = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count);
	m2_total = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count);


	// -------------------------------
	// generate all the necessary maps
	// -------------------------------
	int batch_count;

	int* point_map_offsets1;  // mapping of work_items to offset in batched data
	int* point_map_offsets2;
	int* point_map1; // map of rows of U to point indices in point_set1
	int* point_map2; // map of rows of V to point indices in point_set2
	int* work_item_map1; // map of rows of U to work item indices in mat_vec_data
	int* work_item_map2; // map of rows of V to work item indices in mat_vec_data
	int* work_item_to_batch_map;  // map between work item list (including invalid entries) and batch set list (without invalid entries)

	hipMalloc((void**)&point_map_offsets1, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map_offsets2, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map1, m1_total*sizeof(int));
	hipMalloc((void**)&point_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_map1, m1_total*sizeof(int));
	hipMalloc((void**)&work_item_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_to_batch_map, mat_vec_data_count*sizeof(int));

	thrust::device_ptr<int> point_map_offsets1_ptr(point_map_offsets1);
	thrust::device_ptr<int> point_map_offsets2_ptr(point_map_offsets2);
	thrust::device_ptr<int> point_map1_ptr(point_map1);
	thrust::device_ptr<int> point_map2_ptr(point_map2);
	thrust::device_ptr<int> work_item_map1_ptr(work_item_map1);
	thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);
	thrust::device_ptr<int> work_item_to_batch_map_ptr(work_item_to_batch_map);

	create_maps_and_indices(m1, m2, m1_total, m2_total, point_map_offsets1, point_map_offsets2, point_map1, point_map2, work_item_map1, work_item_map2, work_item_to_batch_map, &batch_count, mat_vec_data, mat_vec_data_count, WT_ACA);


	// -----------------------------------------------------
	// create local copies of some of the index / map fields
	// -----------------------------------------------------
	int* m1_h;
	int* m2_h;
	int* point_map_offsets2_h;
	int* point_map_offsets1_h;
	
	m1_h = new int[mat_vec_data_count];
	m2_h = new int[mat_vec_data_count];
	point_map_offsets2_h = new int[mat_vec_data_count];
	point_map_offsets1_h = new int[mat_vec_data_count];
	
	hipMemcpy(m1_h, m1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(m2_h, m2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets1_h, point_map_offsets1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets2_h, point_map_offsets2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);



	//--------------------------
	// compute the "k" per batch
	//--------------------------
	int* k_per_item;
	hipMalloc((void**)&k_per_item, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> k_per_item_ptr(k_per_item);
	// if (k>min(m,n))
	//     k= min(m,n);
	// end
	set_k_per_item<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(k_per_item, k, mat_vec_data_count, m1, m2);
	hipDeviceSynchronize();
	checkCUDAError("set_k_per_item");


	//-----------------------------
	// set upper bound for global k
	//-----------------------------
	int m1_max = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
	int m2_max = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
	if (k>min(m1_max, m2_max))
	{
		k = min(m1_max, m2_max);
	}


	thrust::device_ptr<double> U_ptr(U);
	thrust::device_ptr<double> V_ptr(V);

//	hipStream_t *streams = new hipStream_t[batch_count];
//	for(int b=0; b<batch_count; b++)
  //      	hipStreamCreate(&streams[b]);




//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("2:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

	hipFree(work_item_to_batch_map);
	hipFree(m1);
	hipFree(m2);
	hipFree(point_map_offsets1);
	hipFree(point_map_offsets2);

	TIME_sssstart;

	// apply low-rank matrix-vector product
	batched_low_rank_mvp(x, y, U, V, m1_total, m2_total, m1_h, m2_h, mat_vec_data_count, batch_count, k, k_per_item, stat, handle , point_map_offsets1_h, point_map_offsets2_h, point_map1, point_map2, work_item_map1 );

	TIME_sssstop("batched_low_rank_mvp");


//	hipblasSetStream(handle, 0);
//	for (int b=0; b<batch_count; b++)
//		hipStreamDestroy(streams[b]);
//	delete[] streams;

	delete [] m1_h;
	delete [] m2_h;
	delete [] point_map_offsets1_h;
	delete [] point_map_offsets2_h;


	hipFree(k_per_item);
	hipFree(point_map1);
	hipFree(point_map2);
	hipFree(work_item_map1);
	hipFree(work_item_map2);
	checkCUDAError("cudaFrees a the end of batched ACA");


//      {
//       size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("3:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

}




void precompute_batched_aca(struct work_item* mat_vec_data, int mat_vec_data_count, struct point_set* input_set1, struct point_set* input_set2, hipblasStatus_t stat, hipblasHandle_t handle, double eta, double epsilon, int k, double** U, double** V, struct system_assembler* assem)
{
	int block_size = MATRIX_ENTRY_BLOCK_SIZE;

	if (mat_vec_data_count==0)
		return;
	
	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);

//	{
//	size_t free_mem, total_mem;
//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("1:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0));
//	}



	// ------------------------
	// compute sizes of batches
	// ------------------------
	int* m1;
	int* m2;
	hipMalloc((void**)&m1, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&m2, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> m1_ptr(m1);
	thrust::device_ptr<int> m2_ptr(m2);
	compute_m1_m2(m1, m2, mat_vec_data, mat_vec_data_count, WT_ACA);
	int m1_total;
	int m2_total;
	m1_total = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count);
	m2_total = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count);


	// -------------------------------
	// generate all the necessary maps
	// -------------------------------
	int batch_count;

	int* point_map_offsets1;  // mapping of work_items to offset in batched data
	int* point_map_offsets2;
	int* point_map1; // map of rows of U to point indices in point_set1
	int* point_map2; // map of rows of V to point indices in point_set2
	int* work_item_map1; // map of rows of U to work item indices in mat_vec_data
	int* work_item_map2; // map of rows of V to work item indices in mat_vec_data
	int* work_item_to_batch_map;  // map between work item list (including invalid entries) and batch set list (without invalid entries)

	hipMalloc((void**)&point_map_offsets1, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map_offsets2, mat_vec_data_count*sizeof(int));
	hipMalloc((void**)&point_map1, m1_total*sizeof(int));
	hipMalloc((void**)&point_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_map1, m1_total*sizeof(int));
	hipMalloc((void**)&work_item_map2, m2_total*sizeof(int));
	hipMalloc((void**)&work_item_to_batch_map, mat_vec_data_count*sizeof(int));

	thrust::device_ptr<int> point_map_offsets1_ptr(point_map_offsets1);
	thrust::device_ptr<int> point_map_offsets2_ptr(point_map_offsets2);
	thrust::device_ptr<int> point_map1_ptr(point_map1);
	thrust::device_ptr<int> point_map2_ptr(point_map2);
	thrust::device_ptr<int> work_item_map1_ptr(work_item_map1);
	thrust::device_ptr<int> work_item_map2_ptr(work_item_map2);
	thrust::device_ptr<int> work_item_to_batch_map_ptr(work_item_to_batch_map);

	create_maps_and_indices(m1, m2, m1_total, m2_total, point_map_offsets1, point_map_offsets2, point_map1, point_map2, work_item_map1, work_item_map2, work_item_to_batch_map, &batch_count, mat_vec_data, mat_vec_data_count, WT_ACA);


	// -----------------------------------------------------
	// create local copies of some of the index / map fields
	// -----------------------------------------------------
	int* m1_h;
	int* m2_h;
	int* point_map_offsets2_h;
	int* point_map_offsets1_h;
	
	m1_h = new int[mat_vec_data_count];
	m2_h = new int[mat_vec_data_count];
	point_map_offsets2_h = new int[mat_vec_data_count];
	point_map_offsets1_h = new int[mat_vec_data_count];
	
	hipMemcpy(m1_h, m1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(m2_h, m2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets1_h, point_map_offsets1, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(point_map_offsets2_h, point_map_offsets2, mat_vec_data_count*sizeof(int), hipMemcpyDeviceToHost);



	//--------------------------
	// compute the "k" per batch
	//--------------------------
	int* k_per_item;
	hipMalloc((void**)&k_per_item, mat_vec_data_count*sizeof(int));
	thrust::device_ptr<int> k_per_item_ptr(k_per_item);
	// if (k>min(m,n))
	//     k= min(m,n);
	// end
	set_k_per_item<<<(mat_vec_data_count + (block_size - 1)) / block_size, block_size>>>(k_per_item, k, mat_vec_data_count, m1, m2);
	hipDeviceSynchronize();
	checkCUDAError("set_k_per_item");


	//-----------------------------
	// set upper bound for global k
	//-----------------------------
//	int m1_max = thrust::reduce(m1_ptr, m1_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
//	int m2_max = thrust::reduce(m2_ptr, m2_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
//	if (k>min(m1_max, m2_max))
//	{
//		k = min(m1_max, m2_max);
//	}

	int k_per_item_max = thrust::reduce(k_per_item_ptr, k_per_item_ptr+mat_vec_data_count, 0, thrust::maximum<int>());
	k = k_per_item_max;


//	size_t free_mem;
//	size_t total_mem;
//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("00 Memory free: %d / %d MB\n", (int)(free_mem/1024/1024), (int)(total_mem/1024/1024)); fflush(stdout);

	
	printf("Allocating %lf MB of memory for k=%d, with m1_total=%d and m2_total=%d\n", (double)((m1_total+m2_total)*k*sizeof(double))/(1024.0*1024.0), k, m1_total, m2_total);

	//-------------------------------
	// allocate and init batched U, V
	//-------------------------------
	hipMalloc((void**)U, m1_total*k*sizeof(double));  
	checkCUDAError("hipMalloc");
	hipMalloc((void**)V, m2_total*k*sizeof(double));
	checkCUDAError("hipMalloc");

	thrust::device_ptr<double> U_ptr(*U);
	thrust::device_ptr<double> V_ptr(*V);


//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("0 Memory free: %d / %d MB\n", (int)(free_mem/1024/1024), (int)(total_mem/1024/1024)); fflush(stdout);



	// TODO: Fill this with nan -> algo should still work
	thrust::fill(U_ptr, U_ptr+m1_total*k, 0.0);
	thrust::fill(V_ptr, V_ptr+m2_total*k, 0.0);


	double* v_r;
	double* u_r;





	// i_r = 0;
//	int i_r = -1;

	int* i_r;
	hipMalloc((void**)&i_r, mat_vec_data_count*sizeof(int));
	checkCUDAError("hipMalloc");
	thrust::device_ptr<int> i_r_ptr(i_r);

// 	hipMemGetInfo(&free_mem, &total_mem);
//	printf("1 Memory free: %d / %d MB\n", (int)(free_mem/1024/1024), (int)(total_mem/1024/1024)); fflush(stdout);
	thrust::fill(i_r_ptr, i_r_ptr+mat_vec_data_count, -1);
	// on invalid entries (in mat_vec_data) we shall never compute
	thrust::replace_if(i_r_ptr, i_r_ptr+mat_vec_data_count, mat_vec_data_ptr, is_not_WT_ACA(), -1);

	bool* search_for_new_v_r;
	hipMalloc((void**)&search_for_new_v_r, mat_vec_data_count*sizeof(bool));
	checkCUDAError("cudaMalloc1");
//	thrust::device_ptr<int> compute_v_r_ptr(compute_v_r);

// 	hipMemGetInfo(&free_mem, &total_mem);
//	printf("2 Memory free: %d / %d MB\n", (int)(free_mem/1024/1024), (int)(total_mem/1024/1024)); fflush(stdout);

//	printf("Batch count %d\n",batch_count); fflush(stdout);

 	
//	hipStream_t *streams = new hipStream_t[batch_count];
//	for(int b=0; b<batch_count; b++)
//      	hipStreamCreate(&streams[b]);

//	hipMemGetInfo(&free_mem, &total_mem);
//        printf("2.5 Memory free: %d / %d MB\n", (int)(free_mem/1024/1024), (int)(total_mem/1024/1024)); fflush(stdout);


/*
	bool* stop_aca_as_soon_as_possible;
	hipMalloc((void**)&stop_aca_as_soon_as_possible, batch_count*sizeof(bool));
	checkCUDAError("cudaMalloc2");
	thrust::device_ptr<bool> stop_aca_as_soon_as_possible_ptr(stop_aca_as_soon_as_possible);
*/
//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("2 Memory free: %d / %d MB\n", (int)(free_mem/1024/1024), (int)(total_mem/1024/1024)); fflush(stdout);

/*
	thrust::fill(stop_aca_as_soon_as_possible_ptr, stop_aca_as_soon_as_possible_ptr+batch_count, false);

	bool* stop_aca_as_soon_as_possible_h;
	stop_aca_as_soon_as_possible_h = new bool[batch_count];
	hipMemcpy(stop_aca_as_soon_as_possible_h, stop_aca_as_soon_as_possible, batch_count*sizeof(bool), hipMemcpyDeviceToHost);
*/
	
//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("3 Memory free: %d / %d MB\n", (int)(free_mem/1024/1024), (int)(total_mem/1024/1024)); fflush(stdout);


	bool* stop_full_aca_for_batch;
	hipMalloc((void**)&stop_full_aca_for_batch, mat_vec_data_count*sizeof(bool));
	thrust::device_ptr<bool> stop_full_aca_for_batch_ptr(stop_full_aca_for_batch);
	thrust::fill(stop_full_aca_for_batch_ptr, stop_full_aca_for_batch_ptr+mat_vec_data_count, false);

	// if it is not a valid batch, I will certainly not start computing
	thrust::replace_if(stop_full_aca_for_batch_ptr, stop_full_aca_for_batch_ptr+mat_vec_data_count, mat_vec_data_ptr, is_not_WT_ACA(), true);



	// for r=1:k
	for (int r=0; r<k; r++)
	{
		// while (norm(v_tilde_r,Inf)==0.0)
	    //    i_r = i_r+1;
	    //    v_tilde_r = kernel(input_set1(i_r,:), input_set2);
	    //    for l=1:r-1
	    //        v_tilde_r = v_tilde_r - U(i_r,l) * V(l,:);
	    //    end
	    // end

        // U = [U u_r];
        // V = [V; v_r];
		v_r = &((*V)[r*m2_total]);
		u_r = &((*U)[r*m1_total]);
		thrust::device_ptr<double> u_r_ptr(u_r);
		thrust::device_ptr<double> v_r_ptr(v_r);


		// if r>=k_per_item[batch_item] on the batch_item'th batch, don't do anything on this batch
		is_smaller_or_equal_r ser(r);
		thrust::replace_if(stop_full_aca_for_batch_ptr, stop_full_aca_for_batch_ptr+mat_vec_data_count, k_per_item_ptr, ser, true);


		compute_current_batched_v_r(v_r, *U, *V, m1_total, m2_total, mat_vec_data, mat_vec_data_count, search_for_new_v_r, i_r, point_map1, point_map2, point_map_offsets1, point_map_offsets2, work_item_map2, input_set1, input_set2, k_per_item, m1, r, assem, k, stop_full_aca_for_batch);


		//// [m,j_r] = max(abs(v_tilde_r));
		//thrust::device_ptr<double> max_pos = thrust::max_element(v_r_ptr, v_r_ptr+m2, compare_absolute());
		//int j_r = max_pos - v_r_ptr;

		int* j_r_global; // j_r index (maximum positions) as global indices in the batched vector
		hipMalloc((void**)&j_r_global, mat_vec_data_count*sizeof(int));  // mat_vec_data_count is an upper bound to the actual amount of batches
		checkCUDAError("hipMalloc");
		thrust::device_ptr<int> j_r_global_ptr(j_r_global);

		compute_current_batched_u_r(u_r, v_r, *U, *V, m1_total, m2_total, mat_vec_data, mat_vec_data_count, point_map1, point_map2, work_item_map1, work_item_map2, input_set1, input_set2, k_per_item, j_r_global, work_item_to_batch_map, r, assem, stop_full_aca_for_batch);


		hipFree(j_r_global);


//		bool check_frobenius = false;
//
//		if (check_frobenius && (r%5==0))
//		{
//
//			bool stop = do_stop_based_on_batched_frobenius_norm(*U, *V, u_r, v_r, m1_total, m2_total, point_map_offsets1_h, point_map_offsets2_h, stop_aca_as_soon_as_possible, stop_aca_as_soon_as_possible_h, work_item_map1, work_item_map2, batch_count, r, mat_vec_data_count, m1_h, m2_h, eta, epsilon, streams, stat, handle );
//
//			if (stop)
//			{
//				break;
//			}
//
//	
//		}
	}

	hipFree(stop_full_aca_for_batch);


//	hipMemGetInfo(&free_mem, &total_mem);
//	printf("4 Memory free: %d / %d MB\n", (int)(free_mem/1024/1024), (int)(total_mem/1024/1024)); fflush(stdout);


	hipFree(work_item_to_batch_map);
	hipFree(i_r);
	hipFree(search_for_new_v_r);
/*
	hipFree(stop_aca_as_soon_as_possible);
	delete [] stop_aca_as_soon_as_possible_h;
*/



//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("2:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

	hipFree(m1);
	hipFree(m2);
	hipFree(point_map_offsets1);
	hipFree(point_map_offsets2);


//	hipblasSetStream(handle, 0);
//	for (int b=0; b<batch_count; b++)
//		hipStreamDestroy(streams[b]);
//	delete[] streams;

	delete [] m1_h;
	delete [] m2_h;
	delete [] point_map_offsets1_h;
	delete [] point_map_offsets2_h;


	hipFree(k_per_item);
	hipFree(point_map1);
	hipFree(point_map2);
	hipFree(work_item_map1);
	hipFree(work_item_map2);
	checkCUDAError("cudaFrees a the end of batched ACA");

	printf("precomputing of ACA done"); fflush(stdout);


//       {
//        size_t free_mem, total_mem;
//        hipMemGetInfo(&free_mem, &total_mem);
//        printf("3:   %lf MB of %lf MB available.\n", (double)free_mem/(1024.0*1024.0), (double)total_mem/(1024.0*1024.0));
//        }

}


void test_precomputation_of_batched_aca(double* x, double* y, struct work_item* mat_vec_data, int mat_vec_data_count, struct point_set* input_set1, struct point_set* input_set2, hipblasStatus_t stat, hipblasHandle_t handle, double eta, double epsilon, int k, struct system_assembler* assem)
{

	double* U;
	double* V;

//	printf("PRECOMPUTE\n");


	precompute_batched_aca(mat_vec_data, mat_vec_data_count, input_set1, input_set2, stat, handle, eta, epsilon, k, &U, &V, assem);

//	printf("APPLY\n");

	apply_precomputed_batched_aca(x, y, mat_vec_data, mat_vec_data_count, input_set1, input_set2, stat, handle, eta, epsilon, k, U, V);

	hipFree(U);
	hipFree(V);

}



void organize_mat_vec_data(struct work_item* mat_vec_data, int mat_vec_data_count, struct mat_vec_data_info* mat_vec_info)
{
	printf("before sort\n");fflush(stdout);
	sort_mat_vec_data(mat_vec_data, mat_vec_data_count);

	printf("before partition_point\n"); fflush(stdout);
	thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);
	thrust::device_ptr<struct work_item> dense_end_ptr = thrust::partition_point(mat_vec_data_ptr, mat_vec_data_ptr+mat_vec_data_count, is_not_WT_ACA());

	printf("after partition point\n"); fflush(stdout);
	mat_vec_info->dense_count = dense_end_ptr - mat_vec_data_ptr;
	printf("dense count: %d\n", mat_vec_info->dense_count);
	mat_vec_info->aca_count = mat_vec_data_count - mat_vec_info->dense_count;
	printf("aca_count: %d\n", mat_vec_info->aca_count);

	mat_vec_info->total_count = mat_vec_data_count;

}

__global__ void linear_algebra_get_point_count_dim(int* point_count, int* dim, struct point_set* input_set1)
{
        *point_count = input_set1->size;
        *dim = input_set1->dim;
}

int compute_current_dense_work_size_magma(struct work_item* mat_vec_data_h, struct mat_vec_data_info* mat_vec_info, int max_batched_size, int current_work_item_index)
{
        int current_dense_work_size = 0;
        int current_dense_matrix_size = 0;
//	int m2_max = 0;
	int m1, m2;


//        while ((current_dense_matrix_size*m2_max <= max_batched_size)&&(current_work_item_index+current_dense_work_size<mat_vec_info->dense_count))
//        {
//		m1 = mat_vec_data_h[current_work_item_index+current_dense_work_size].set1_u-mat_vec_data_h[current_work_item_index+current_dense_work_size].set1_l+1;
//		m2 = mat_vec_data_h[current_work_item_index+current_dense_work_size].set2_u-mat_vec_data_h[current_work_item_index+current_dense_work_size].set2_l+1;
//		m2_max = max(m2_max,m2);
//
//                current_dense_matrix_size += m1;
//
//                current_dense_work_size++;
//
////		printf("%d %d %d %d %d\n",current_dense_work_size, current_dense_matrix_size, max_batched_size, current_work_item_index, max_batched_size);
//        }

//        if (current_dense_matrix_size*m2_max > max_batched_size)  // attention, m2_max was changing during the adaption process, so this does not necessarily result in the correct output
//                current_dense_work_size--;

//	// this is whi I now start again from the beginning now using a fixed m2_max to make sure that changing m2_max during the adaption does not cause trouble

	current_dense_work_size = 0;
	current_dense_matrix_size = 0;

//	int actually_needed = 0;

        while ((current_dense_matrix_size <= max_batched_size)&&(current_work_item_index+current_dense_work_size<mat_vec_info->dense_count))
        {
                m1 = mat_vec_data_h[current_work_item_index+current_dense_work_size].set1_u-mat_vec_data_h[current_work_item_index+current_dense_work_size].set1_l+1;
                m2 = mat_vec_data_h[current_work_item_index+current_dense_work_size].set2_u-mat_vec_data_h[current_work_item_index+current_dense_work_size].set2_l+1;
                current_dense_matrix_size += m1*m2;

                current_dense_work_size++;

//              printf("%d %d %d %d %d\n",current_dense_work_size, current_dense_matrix_size, max_batched_size, current_work_item_index, max_batched_size);
        }


        if (current_dense_matrix_size > max_batched_size)
                current_dense_work_size--;

//	printf("used: %lu, allocated: %lu\n", current_dense_matrix_size*sizeof(double)/1024/1024, max_batched_size*sizeof(double)/1024/1024);

        return current_dense_work_size;
}


void precompute_aca_for_h_matrix_mvp(struct work_item* mat_vec_data, struct mat_vec_data_info* mat_vec_info, struct point_set* input_set1, struct point_set* input_set2, double eta, double epsilon, int k, double** U, double** V, struct system_assembler* assem)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);

	precompute_batched_aca(&mat_vec_data[mat_vec_info->dense_count], mat_vec_info->aca_count, input_set1, input_set2, stat, handle, eta, epsilon, k, U, V, assem);


    hipblasDestroy(handle);
}


void precompute_aca_for_h_matrix_mvp_custom(struct work_item* mat_vec_data, struct mat_vec_data_info* mat_vec_info, struct point_set* input_set1, struct point_set* input_set2, double eta, double epsilon, int k, double** U, double** V, struct system_assembler* assem, int aca_offset, int aca_count)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);

	precompute_batched_aca(&mat_vec_data[mat_vec_info->dense_count+aca_offset], aca_count, input_set1, input_set2, stat, handle, eta, epsilon, k, U, V, assem);


    hipblasDestroy(handle);
}

int compute_current_dense_work_size(struct work_item* mat_vec_data_h, struct mat_vec_data_info* mat_vec_info, int max_batched_size, double batching_ratio, int current_work_item_index)
{
	int current_dense_work_size = 0;
	int current_full_matrix_size_a = 0;
	int current_full_matrix_size_b = 0;
	int current_actually_used_matrix_size = 0;

	while ((current_full_matrix_size_a*current_full_matrix_size_b <= max_batched_size*max_batched_size)&&(current_work_item_index+current_dense_work_size<mat_vec_info->dense_count))
	{
		int size_a = mat_vec_data_h[current_work_item_index+current_dense_work_size].set1_u-mat_vec_data_h[current_work_item_index+current_dense_work_size].set1_l+1;
		int size_b = mat_vec_data_h[current_work_item_index+current_dense_work_size].set2_u-mat_vec_data_h[current_work_item_index+current_dense_work_size].set2_l+1;

		current_full_matrix_size_a += size_a;
		current_full_matrix_size_b += size_b;

		current_actually_used_matrix_size += size_a*size_b; 

		current_dense_work_size++;

		if ((double)current_actually_used_matrix_size/(double)(current_full_matrix_size_a*current_full_matrix_size_b)<batching_ratio)
			break;
	}


	if (current_full_matrix_size_a*current_full_matrix_size_b > max_batched_size*max_batched_size)
		current_dense_work_size--;

	return current_dense_work_size;
}

int compute_current_aca_work_size(struct work_item* mat_vec_data_h, struct mat_vec_data_info* mat_vec_info, int max_batched_size, int current_work_item_index)
{
        int current_aca_work_size = 0;
        int current_aca_matrix_size = 0;

        while ((current_aca_matrix_size <= max_batched_size)&&(current_work_item_index+current_aca_work_size<mat_vec_info->aca_count+mat_vec_info->dense_count)&&(current_aca_work_size<(65536-2)))
        {
                current_aca_matrix_size += mat_vec_data_h[current_work_item_index+current_aca_work_size].set1_u-mat_vec_data_h[current_work_item_index+current_aca_work_size].set1_l+1;

                current_aca_work_size++;
        }

        if (current_aca_matrix_size > max_batched_size)
                current_aca_work_size--;

        return current_aca_work_size;
}

void predict_precomputing_memory_requirements(struct work_item* mat_vec_data, struct mat_vec_data_info* mat_vec_info, int* predicted_dense_work_size)
{
        struct work_item* mat_vec_data_h = new struct work_item[mat_vec_info->dense_count+mat_vec_info->aca_count];

        hipMemcpy(mat_vec_data_h, mat_vec_data, (mat_vec_info->dense_count+mat_vec_info->aca_count)*sizeof(struct work_item), hipMemcpyDeviceToHost);

	int required_dense_work_size = 0;
	int m1,m2;

	for (int i=0; i<mat_vec_info->dense_count; i++)
	{
		m1 = mat_vec_data_h[i].set1_u-mat_vec_data_h[i].set1_l+1;
		m2 = mat_vec_data_h[i].set2_u-mat_vec_data_h[i].set2_l+1;
		required_dense_work_size += m1*m2;
	}

	printf("Requiring %lf MB of memory and a dense_work_size of %d for dense blocks.\n", (double)required_dense_work_size*(double)sizeof(double)/(1024.0*1024.0), required_dense_work_size);	

	*predicted_dense_work_size = (int)((double)required_dense_work_size*1.05);

	delete [] mat_vec_data_h;

}


void precompute_work_sizes(int** dense_work_size, int** aca_work_size, int* dense_batch_count, int* aca_batch_count, struct work_item* mat_vec_data, struct mat_vec_data_info* mat_vec_info, int max_batched_dense_size, int max_batched_aca_size)
{
        struct work_item* mat_vec_data_h = new struct work_item[mat_vec_info->dense_count+mat_vec_info->aca_count];

        hipMemcpy(mat_vec_data_h, mat_vec_data, (mat_vec_info->dense_count+mat_vec_info->aca_count)*sizeof(struct work_item), hipMemcpyDeviceToHost);

        int current_dense_work_size;
        int current_dense_work_item_index = 0;

	dense_batch_count[0]=0;
	aca_batch_count[0]=0;

	// computing the number of batches that have to be found

        current_dense_work_size = compute_current_dense_work_size_magma(mat_vec_data_h, mat_vec_info, max_batched_dense_size, current_dense_work_item_index);
	dense_batch_count[0]++;	

        while (current_dense_work_size > 0)
        {
                current_dense_work_item_index += current_dense_work_size;

                current_dense_work_size = compute_current_dense_work_size_magma(mat_vec_data_h, mat_vec_info, max_batched_dense_size, current_dense_work_item_index);
		dense_batch_count[0]++;
	}

        int current_aca_work_size;
        int current_aca_work_item_index = mat_vec_info->dense_count;

        current_aca_work_size = compute_current_aca_work_size(mat_vec_data_h, mat_vec_info, max_batched_aca_size, current_aca_work_item_index);
	aca_batch_count[0]++;

        while (current_aca_work_size > 0)
        {
                current_aca_work_item_index += current_aca_work_size;

                current_aca_work_size = compute_current_aca_work_size(mat_vec_data_h, mat_vec_info, max_batched_aca_size, current_aca_work_item_index);

		aca_batch_count[0]++;
        }


	// now, I can allocate the appropriatly sized arrays and store the sizes of the batches
	
	dense_work_size[0] = new int[dense_batch_count[0]];
	aca_work_size[0] = new int[aca_batch_count[0]];

	int index = 0;

	current_dense_work_item_index = 0;


        current_dense_work_size = compute_current_dense_work_size_magma(mat_vec_data_h, mat_vec_info, max_batched_dense_size, current_dense_work_item_index);

	dense_work_size[0][index] = current_dense_work_size;
	index++;

        while (current_dense_work_size > 0)
        {
                current_dense_work_item_index += current_dense_work_size;

                current_dense_work_size = compute_current_dense_work_size_magma(mat_vec_data_h, mat_vec_info, max_batched_dense_size, current_dense_work_item_index);
		dense_work_size[0][index] = current_dense_work_size;
                index++;
        }

	index = 0;

        current_aca_work_item_index = mat_vec_info->dense_count;
	
        current_aca_work_size = compute_current_aca_work_size(mat_vec_data_h, mat_vec_info, max_batched_aca_size, current_aca_work_item_index);
        aca_work_size[0][index] = current_aca_work_size;
	index++;

        while (current_aca_work_size > 0)
        {
                current_aca_work_item_index += current_aca_work_size;
                current_aca_work_size = compute_current_aca_work_size(mat_vec_data_h, mat_vec_info, max_batched_aca_size, current_aca_work_item_index);

                aca_work_size[0][index] = current_aca_work_size;
		index++;
        }



        delete [] mat_vec_data_h;


}

void h_matrix_mvp(double* x, double* y, struct work_item* mat_vec_data, struct mat_vec_data_info* mat_vec_info, struct point_set* input_set1, struct point_set* input_set2, double eta, double epsilon, int k, double* dA, double* U, double* V, struct system_assembler* assem, int max_batched_dense_size, double dense_batching_ratio, int max_batched_aca_size, magma_queue_t magma_queue, int* dense_work_size, int* aca_work_size, int dense_batch_count, int aca_batch_count, bool use_precomputed_aca, bool use_precomputed_dense)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);


		// very dirty way to get point count and dimensionality of points
		int point_count_1,dim;
		int* point_count_1_d; hipMalloc((void**)&point_count_1_d, sizeof(int));
		int* dim_d; hipMalloc((void**)&dim_d, sizeof(int));
		linear_algebra_get_point_count_dim<<<1,1>>>(point_count_1_d, dim_d, input_set1);
		hipMemcpy(&point_count_1, point_count_1_d, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&dim, dim_d, sizeof(int), hipMemcpyDeviceToHost);
		hipFree(point_count_1_d); hipFree(dim_d);

	TIME_ssstart;

	int current_dense_work_item_index = 0;

	if (use_precomputed_dense)
	{
		apply_batched_dense_magma(x, y, &mat_vec_data[current_dense_work_item_index], dense_work_size[0], input_set1, input_set2, stat, handle, assem, &magma_queue, dA, true);

	}
	else
	{
		hipMalloc((void**)&dA, sizeof(double)*max_batched_dense_size);  // Actually this does not allocate dA such that it is accessible
										 // by data->dA. However dA will be freed immediately anyway
	
		for (int b = 0; b<dense_batch_count; b++)
		{
			if (dense_work_size[b]>0)
			{
				apply_batched_dense_magma(x, y, &mat_vec_data[current_dense_work_item_index], dense_work_size[b], input_set1, input_set2, stat, handle, assem, &magma_queue, dA, false);
	
				current_dense_work_item_index += dense_work_size[b];	
			}
		
	        }

		hipFree(dA);
	}


	TIME_ssstop("dense blocks");


	TIME_ssstart;


	if (use_precomputed_aca)
	{
		thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);
		thrust::device_ptr<struct work_item> dense_end_ptr = thrust::partition_point(mat_vec_data_ptr, mat_vec_data_ptr+mat_vec_info->total_count, is_not_WT_ACA());
	
		apply_precomputed_batched_aca_magma(x, y, &mat_vec_data[mat_vec_info->dense_count], mat_vec_info->aca_count, input_set1, input_set2, stat, handle, &magma_queue, eta, epsilon, k, U, V);
	}
	else
	{
		thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);

		int current_aca_work_item_index = mat_vec_info->dense_count;

		for (int b = 0; b<aca_batch_count; b++)
		{
			if (aca_work_size[b] > 0)
			{
				apply_batched_aca(x, y, &mat_vec_data[current_aca_work_item_index], aca_work_size[b], input_set1, input_set2, stat, handle, eta, epsilon, k, assem);
	
				current_aca_work_item_index += aca_work_size[b];	
			}
        	}
	
	}



	TIME_ssstop("ACA blocks");

    
	hipblasDestroy(handle);
}


void h_matrix_mvp_parallel(double* x, double* y, struct work_item* mat_vec_data, struct mat_vec_data_info* mat_vec_info, struct point_set* input_set1, struct point_set* input_set2, double eta, double epsilon, int k, double* dA, double* U, double* V, struct system_assembler* assem, int max_batched_dense_size, double dense_batching_ratio, int max_batched_aca_size, magma_queue_t magma_queue, int* dense_work_size, int* aca_work_size, int dense_batch_count, int aca_batch_count, bool use_precomputed_aca, bool use_precomputed_dense, int dense_offset, int aca_offset, int proc)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);


		// very dirty way to get point count and dimensionality of points
		int point_count_1,dim;
		int* point_count_1_d; hipMalloc((void**)&point_count_1_d, sizeof(int));
		int* dim_d; hipMalloc((void**)&dim_d, sizeof(int));
		linear_algebra_get_point_count_dim<<<1,1>>>(point_count_1_d, dim_d, input_set1);
		hipMemcpy(&point_count_1, point_count_1_d, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&dim, dim_d, sizeof(int), hipMemcpyDeviceToHost);
		hipFree(point_count_1_d); hipFree(dim_d);

	TIME_ssstart;

	if (use_precomputed_dense)
	{
		apply_batched_dense_magma(x, y, &mat_vec_data[dense_offset], dense_work_size[proc], input_set1, input_set2, stat, handle, assem, &magma_queue, dA, true);

	}
	else
	{
		printf("Parallel H matrix execution without precomputing is currently not supported\n");
		exit(1);	
	}

	TIME_ssstop("dense blocks");


	TIME_ssstart;


	if (use_precomputed_aca)
	{
		thrust::device_ptr<struct work_item> mat_vec_data_ptr(mat_vec_data);
		thrust::device_ptr<struct work_item> dense_end_ptr = thrust::partition_point(mat_vec_data_ptr, mat_vec_data_ptr+mat_vec_info->total_count, is_not_WT_ACA());
	
		apply_precomputed_batched_aca_magma(x, y, &mat_vec_data[mat_vec_info->dense_count+aca_offset], aca_work_size[proc], input_set1, input_set2, stat, handle, &magma_queue, eta, epsilon, k, U, V);
	}
	else
	{
		printf("Parallel H matrix execution without precomputing is currently not supported\n");
		exit(1);
	}



	TIME_ssstop("ACA blocks");

    
	hipblasDestroy(handle);
}





void sequential_h_matrix_mvp_without_batching(double* x, double* y, struct work_item* mat_vec_data, struct mat_vec_data_info* mat_vec_info, struct point_set* input_set1, struct point_set* input_set2, double eta, double epsilon, int k, struct system_assembler* assem)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);

	struct work_item current_mat_vec_data;

	// set output vector to zero

		// very dirty way to get point count and dimensionality of points
		int point_count_1,dim;
		int* point_count_1_d; hipMalloc((void**)&point_count_1_d, sizeof(int));
		int* dim_d; hipMalloc((void**)&dim_d, sizeof(int));
		linear_algebra_get_point_count_dim<<<1,1>>>(point_count_1_d, dim_d, input_set1);
		hipMemcpy(&point_count_1, point_count_1_d, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&dim, dim_d, sizeof(int), hipMemcpyDeviceToHost);
		hipFree(point_count_1_d); hipFree(dim_d);

	thrust::device_ptr<double> y_ptr(y);
	thrust::fill(y_ptr, y_ptr+point_count_1, 0.0);


	TIME_ssstart;

	for (int i=0; i<mat_vec_info->total_count; i++)
	{
		// get current work item to handle
		hipMemcpy(&current_mat_vec_data, &mat_vec_data[i], sizeof(struct work_item), hipMemcpyDeviceToHost);

		// handling of dense blocks
		if (current_mat_vec_data.work_type==WT_DENSE)
		{
			apply_dense_matrix_for_current_work_item(x, y, current_mat_vec_data, input_set1, input_set2, stat, handle, assem);
		}  // handling of low rank blocks
	}


	TIME_ssstop("dense blocks without batching");


	TIME_ssstart;

	for (int i=0; i<mat_vec_info->total_count; i++)
	{
		// get current work item to handle
		hipMemcpy(&current_mat_vec_data, &mat_vec_data[i], sizeof(struct work_item), hipMemcpyDeviceToHost);

		// handling of low rank blocks
		if (current_mat_vec_data.work_type==WT_ACA)
		{
			apply_aca_for_current_work_item(x, y, current_mat_vec_data, input_set1, input_set2, stat, handle, eta, epsilon, k, assem);
		}
	}

	TIME_ssstop("ACA blocks without_batching");


    hipblasDestroy(handle);
}

