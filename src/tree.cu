#include "hip/hip_runtime.h"
// Copyright (C) 2016 Peter Zaspel
//
// This file is part of hmglib.
//
// hmglib is free software: you can redistribute it and/or modify it under the
// terms of the GNU Lesser General Public License as published by the Free
// Software Foundation, either version 3 of the License, or (at your option) any
// later version.
//
// hmglib is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
// A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
// details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with hmglib.  If not, see <http://www.gnu.org/licenses/>.

#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/unique.h>
#include <thrust/sort.h>

#include "tree.h"

#ifndef CHECK_CUDA_ERROR
#define CHECK_CUDA_ERROR
void checkCUDAError(const char* msg) {
hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
#endif


//! computation of the split within a morton code based tree
//! this codes is taken from
//! "Thinking Parallel, Part III: Tree Construction on the GPU"
__device__ __forceinline__ int findSplit( struct morton_code *codes, int first, int last)
{
    // Identical Morton codes => split the range in the middle.

    uint64_t firstCode = codes->code[first];
    uint64_t lastCode = codes->code[last];

    if (firstCode == lastCode)
        return (first + last) >> 1;

    // Calculate the number of highest bits that are the same
    // for all objects, using the count-leading-zeros intrinsic.

    int commonPrefix = __clzll(firstCode ^ lastCode);

    // Use binary search to find where the next bit differs.
    // Specifically, we are looking for the highest object that
    // shares more than commonPrefix bits with the first one.

    int split = first; // initial guess
    int step = last - first;

    do
    {
        step = (step + 1) >> 1; // exponential decrease
        int newSplit = split + step; // proposed new position

        if (newSplit < last)
        {
        	uint64_t splitCode = codes->code[newSplit];
            int splitPrefix = __clzll(firstCode ^ splitCode);
            if (splitPrefix > commonPrefix)
                split = newSplit; // accept proposal
        }
    }
    while (step > 1);

    return split;
}

__host__ __device__ __forceinline__ bool is_valid_work_item(struct work_item item)
{
	return (item.set1_l!=-1);
//	return ((item.set1_l!=-1) && (item.set1_u!=-1) && (item.set2_l!=-1) && (item.set2_u!=-1));
//	return ((item.set1_l>0) && (item.set1_l<100));
}


__host__ __device__ __forceinline__ bool at_least_one_block_smaller_than_threshold(struct work_item item, int c_leaf)
{
	return (((item.set1_u-item.set1_l+1)<=c_leaf) || ((item.set2_u-item.set2_l+1)<=c_leaf));
}


__host__ __device__ __forceinline__ double compute_diameter(double* min, double* max, int dim)
{
	double diam = 0.0;

	for (int d=0; d<dim; d++)
	{
		diam += (max[d]-min[d]) * (max[d]-min[d]);
	}
	diam = sqrt(diam);

	return diam;
}

__host__ __device__ __forceinline__ double compute_distance(double* min1, double* max1, double* min2, double* max2, int dim)
{
	double dist = 0.0;

	for (int d=0; d<dim; d++)
	{
		dist += (fmax(0.0, min1[d]-max2[d]) * fmax(0.0, min1[d]-max2[d])) + (fmax(0.0, min2[d]-max1[d]) * fmax(0.0, min2[d]-max1[d]));
	}
	dist = sqrt(dist);

	return dist;
}


__host__ __device__ __forceinline__ bool bounding_box_admissibility(struct work_item item, struct point_set* input_set1, struct point_set* input_set2, double eta)
{
/*
	int l1 = item.set1_l;
	int u1 = item.set1_u;
	int l2 = item.set2_l;
	int u2 = item.set2_u;

	int dim = input_set1->dim;
	double min1[MAX_POINT_DIMENSION];
	double max1[MAX_POINT_DIMENSION];
	double min2[MAX_POINT_DIMENSION];
	double max2[MAX_POINT_DIMENSION];

	for (int d=0; d<dim; d++)
	{
		min1[d] = input_set1->coords[d][l1];
		max1[d] = input_set1->coords[d][l1];
		min2[d] = input_set2->coords[d][l2];
		max2[d] = input_set2->coords[d][l2];
		for (int i=l1+1; i<=u1; i++)
		{
			min1[d] = fmin(min1[d], input_set1->coords[d][i]);
			max1[d] = fmax(max1[d], input_set1->coords[d][i]);
		}
		for (int i=l2+1; i<=u2; i++)
		{
			min2[d] = fmin(min2[d], input_set2->coords[d][i]);
			max2[d] = fmax(max2[d], input_set2->coords[d][i]);
		}
	}
*/
	double diam1 = compute_diameter(item.min1, item.max1, item.dim);
	double diam2 = compute_diameter(item.min2, item.max2, item.dim);

	double dist = compute_distance(item.min1, item.max1, item.min2, item.max2, item.dim);

//	return (fmin(diam1, diam2) <= (eta*dist));
	return (fmax(diam1, diam2) <= (eta*dist));
}


void print_work_items(struct work_item* work_items, int work_item_count)
{
	for (int i=0; i<work_item_count; i++)
	{
		struct work_item item;
		hipMemcpy(&item, &work_items[i], sizeof(struct work_item), hipMemcpyDeviceToHost);
		printf("S1_L: %d,  S1_U: %d,  S2_L: %d,  S2_U: %d,  work_type: %s\n", item.set1_l, item.set1_u, item.set2_l, item.set2_u, (item.work_type==WT_ACA) ? "ACA" : "DENSE");
	}
}

void write_work_items(char* file_name, struct work_item* work_items, int work_item_count)
{
	FILE* f= fopen(file_name, "w");
	for (int i=0; i<work_item_count; i++)
	{
		struct work_item item;
		hipMemcpy(&item, &work_items[i], sizeof(struct work_item), hipMemcpyDeviceToHost);
		fprintf(f, "S1_L: %d,  S1_U: %d,  S2_L: %d,  S2_U: %d,  work_type: %s\n", item.set1_l, item.set1_u, item.set2_l, item.set2_u, (item.work_type==WT_ACA) ? "ACA" : "DENSE");
	}
	fclose(f);
}



__global__ void init_tree_array_root(struct work_item *current_level_data, struct work_item item)
{
	current_level_data[0] = item;
}

__global__ void invalidate_array(struct work_item* items, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx>=size)
		return;

	items[idx].set1_l = -1;
	items[idx].set1_u = -1;
	items[idx].set2_l = -1;
	items[idx].set2_u = -1;
}

__global__ void set_array(int* a, int value, int count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx>=count)
		return;

	a[idx] = value;
}

__global__ void count_for_new_level(struct work_item *current_level_data, struct work_item* next_level_data, int* new_mat_vec_counts, int* new_child_counts, int total_children, struct morton_code* input_set1_codes, struct morton_code* input_set2_codes, struct point_set* input_set1, struct point_set* input_set2, double eta, int current_level, int max_level, int c_leaf)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= total_children)
		return;

	struct work_item work = current_level_data[idx];

	// start by counting children
	if (!is_valid_work_item(work))
	{
		new_child_counts[idx] = 0;
		new_mat_vec_counts[idx] = 0;
	}
	else // we are having a valid work item
	{
		bool is_admissible = bounding_box_admissibility(work, input_set1, input_set2, eta);

//		int level_1 = round(log((double)(work.set1_u-work.set1_l+1))/log(2.0));
//		int level_2 = round(log((double)(work.set2_u-work.set2_l+1))/log(2.0));
		
		if (work.level_1==work.level_2)
		{
			if (is_admissible)
			{
				// ACA
				new_child_counts[idx] = 0;
				new_mat_vec_counts[idx] = 1;
				}
			else if (at_least_one_block_smaller_than_threshold(work, c_leaf) || (current_level+1)>=max_level )
			{
				// dense MVP
				new_child_counts[idx] = 0;
				new_mat_vec_counts[idx] = 1;
			}
			else
			{
		                int split_set1 = findSplit( input_set1_codes, work.set1_l, work.set1_u);
                		int split_set2 = findSplit( input_set2_codes, work.set2_l, work.set2_u);

				int min1 = min(split_set1-work.set1_l,work.set1_u-(split_set1+1));
				int min2 = min(split_set2-work.set2_l,work.set2_u-(split_set2+1));

				if ((min1>=1)&&(min2>=1))
				{
                                        // create children
                                        new_child_counts[idx] = 4;
                                        new_mat_vec_counts[idx] = 0;
				}
				else  
				{
					// dense MVP
					new_child_counts[idx] = 0;
					new_mat_vec_counts[idx] = 1;
				}
			}
		}
		else
		{
			// split only one block
			new_child_counts[idx] = 2;
			new_mat_vec_counts[idx] = 0;
		}
	}

}

__global__ void compute_bounding_boxes(struct work_item *current_level_data, int total_children, struct point_set* input_set1, struct point_set* input_set2)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= total_children)
		return;

	struct work_item* work = &current_level_data[idx];

	int l1 = work->set1_l;
	int u1 = work->set1_u;
	int l2 = work->set2_l;
	int u2 = work->set2_u;

	int dim = input_set1->dim;
	work->dim = dim;

	double min1[MAX_POINT_DIMENSION];
	double max1[MAX_POINT_DIMENSION];
	double min2[MAX_POINT_DIMENSION];
	double max2[MAX_POINT_DIMENSION];

	for (int d=0; d<dim; d++)
	{
		min1[d] = input_set1->coords[d][l1];
		max1[d] = input_set1->coords[d][l1];
		min2[d] = input_set2->coords[d][l2];
		max2[d] = input_set2->coords[d][l2];
		for (int i=l1+1; i<=u1; i++)
		{
			min1[d] = fmin(min1[d], input_set1->coords[d][i]);
			max1[d] = fmax(max1[d], input_set1->coords[d][i]);
		}
		for (int i=l2+1; i<=u2; i++)
		{
			min2[d] = fmin(min2[d], input_set2->coords[d][i]);
			max2[d] = fmax(max2[d], input_set2->coords[d][i]);
		}
	}

	for (int d=0; d<dim; d++)
	{
		work->min1[d] = min1[d];
		work->max1[d] = max1[d];
		work->min2[d] = min2[d];
		work->max2[d] = max2[d];
	}

}



__global__ void generate_new_level(struct work_item *current_level_data, struct work_item* next_level_data, struct work_item* mat_vec_data_at_current_offset, int* child_counts, int total_children, int* new_mat_vec_offsets, int* new_child_offsets, struct morton_code* input_set1_codes, struct morton_code* input_set2_codes, struct point_set* input_set1, struct point_set* input_set2, double eta, int current_level, int max_level, int c_leaf)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= total_children)
		return;
	// compute offset for children and do updating (in parallel)
	// this update process is a warp-wide collective operation and therefore has to include invalid
	// work items (-> warp lanes on invalid work items still have to take part in the collective operation)

	struct work_item work = current_level_data[idx];

	if (child_counts[idx]==4)
	{
		int offset = new_child_offsets[idx];

		// create children
		struct work_item child11, child12, child21, child22;

		int split_set1 = findSplit( input_set1_codes, work.set1_l, work.set1_u);
		int split_set2 = findSplit( input_set2_codes, work.set2_l, work.set2_u);

		child11.set1_l = work.set1_l;
		child11.set1_u = split_set1;
		child11.set2_l = work.set2_l;
		child11.set2_u = split_set2;
		child11.level_1 = work.level_1+1;
		child11.level_2 = work.level_2+1;

		child12.set1_l = work.set1_l;
		child12.set1_u = split_set1;
		child12.set2_l = split_set2+1;
		child12.set2_u = work.set2_u;
		child12.level_1 = work.level_1+1;
		child12.level_2 = work.level_2+1;

		child21.set1_l = split_set1+1;
		child21.set1_u = work.set1_u;
		child21.set2_l = work.set2_l;
		child21.set2_u = split_set2;
		child21.level_1 = work.level_1+1;
		child21.level_2 = work.level_2+1;

		child22.set1_l = split_set1+1;  // correct ?
		child22.set1_u = work.set1_u;
		child22.set2_l = split_set2+1;
		child22.set2_u = work.set2_u;
		child22.level_1 = work.level_1+1;
		child22.level_2 = work.level_2+1;

		child11.dim = work.dim;
		child12.dim = work.dim;
		child21.dim = work.dim;
		child22.dim = work.dim;


		// insert new children in work queue
		next_level_data[offset]=child11;
		next_level_data[offset+1]=child12;
		next_level_data[offset+2]=child21;
		next_level_data[offset+3]=child22;
	}
	else if (child_counts[idx]==2)
	{
//                int level_1 = round(log((double)(work.set1_u-work.set1_l+1))/log(2.0));
//                int level_2 = round(log((double)(work.set2_u-work.set2_l+1))/log(2.0));

                int split_set1 = findSplit( input_set1_codes, work.set1_l, work.set1_u);
                int split_set2 = findSplit( input_set2_codes, work.set2_l, work.set2_u);

                int min1 = min(split_set1-work.set1_l,work.set1_u-(split_set1+1));
                int min2 = min(split_set2-work.set2_l,work.set2_u-(split_set2+1));


		if (work.level_1<work.level_2)
		{
			int offset = new_child_offsets[idx];
			// create children
			struct work_item child11, child21;

			int split_set1 = findSplit( input_set1_codes, work.set1_l, work.set1_u);

	                child11.set1_l = work.set1_l;
	                child11.set1_u = split_set1;
         	       	child11.set2_l = work.set2_l;
                	child11.set2_u = work.set2_u;
			child11.level_1 = work.level_1+1;
			child11.level_2 = work.level_2;

        	        child21.set1_l = split_set1+1;
                	child21.set1_u = work.set1_u;
	                child21.set2_l = work.set2_l;
        	        child21.set2_u = work.set2_u;
			child21.level_1 = work.level_1+1;
			child21.level_2 = work.level_2;

			child11.dim = work.dim;
			child21.dim = work.dim;

			// insert new children in work queue
			next_level_data[offset]=child11;
			next_level_data[offset+1]=child21;
		}
		else
		{
	                int offset = new_child_offsets[idx];

	                // create children
	                struct work_item child11, child12;
	
	                int split_set2 = findSplit( input_set2_codes, work.set2_l, work.set2_u);
	
	                child11.set1_l = work.set1_l;
	                child11.set1_u = work.set1_u;
	                child11.set2_l = work.set2_l;
	                child11.set2_u = split_set2;
			child11.level_1 = work.level_1;
			child11.level_2 = work.level_2+1;
	
		        child12.set1_l = work.set1_l;
	                child12.set1_u = work.set1_u;
        	        child12.set2_l = split_set2+1;
	                child12.set2_u = work.set2_u;
			child12.level_1 = work.level_1;
			child12.level_2 = work.level_2+1;
	
			child11.dim = work.dim;
			child12.dim = work.dim;

	                // insert new children in work queue
        	        next_level_data[offset]=child11;
        	        next_level_data[offset+1]=child12;
		}
	}
	else
	{
		bool is_admissible = bounding_box_admissibility(work, input_set1, input_set2, eta);

		if (is_admissible)
			work.work_type = WT_ACA;
		else
			work.work_type = WT_DENSE;

		mat_vec_data_at_current_offset[new_mat_vec_offsets[idx]] = work;
	}

}

hipEvent_t sstart, sstop;
float mmilliseconds;

#define TIME_sstart {hipEventCreate(&sstart); hipEventCreate(&sstop); hipEventRecord(sstart);}
#define TIME_sstop(a) {hipEventRecord(sstop); hipEventSynchronize(sstop); hipEventElapsedTime(&mmilliseconds, sstart, sstop); printf("%s: Elapsed time: %lf ms\n", a, mmilliseconds); }

void traverse_with_arrays(struct work_item root_h, struct work_item* mat_vec_data, int* mat_vec_data_count, struct morton_code* input_set1_codes, struct morton_code* input_set2_codes, struct point_set* input_set1, struct point_set* input_set2, double eta, int max_level, int c_leaf, int max_elements_in_array)
{
	struct work_item* current_level_data;
	struct work_item* next_level_data;

	hipMalloc((void**)&current_level_data, max_elements_in_array*sizeof(struct work_item));
	hipMalloc((void**)&next_level_data, max_elements_in_array*sizeof(struct work_item));

	// calculate GPU thread configuration
	int block_size = 512;
	int grid_size = (max_elements_in_array + (block_size - 1)) / block_size;

	// invalidate tree level data arrays
	// TODO: The following two kernel calls require a relatively large amount of runtime; what to do?
	invalidate_array<<<grid_size, block_size>>>(current_level_data, max_elements_in_array);
	checkCUDAError("invalidate_array0");
	invalidate_array<<<grid_size, block_size>>>(next_level_data, max_elements_in_array);
	checkCUDAError("invalidate_array1");

	// fill initial node into first tree level
	init_tree_array_root<<<1, 1>>>(current_level_data, root_h);
	checkCUDAError("init_tree_array_root");

	int* new_mat_vec_counts;   // number of MatVecs that will be generated per valid node of the current tree level
	int* new_child_counts;     // number of child nodes that will be generated per valid node of the current tree level
	int* new_child_offsets;    // storage for the offsets for the new child nodes in the next tree level

	// allocation
	hipMalloc((void**)&new_mat_vec_counts, max_elements_in_array*sizeof(int));
	hipMalloc((void**)&new_child_counts, max_elements_in_array*sizeof(int));
	hipMalloc((void**)&new_child_offsets, max_elements_in_array*sizeof(int));
	// pointer fun
	thrust::device_ptr<int> new_mat_vec_counts_ptr(new_mat_vec_counts);
	thrust::device_ptr<int> new_child_counts_ptr(new_child_counts);
	thrust::device_ptr<int> new_child_offsets_ptr(new_child_offsets);

	int total_new_mat_vecs;  // temp field to store total number of new MatVecs of the current level

	int total_children = 1;  // number of nodes on current level
	int total_new_children;  // temp field to store total number of new nodes on next level
	int grid_size_for_children = (total_children + (block_size - 1)) / block_size;  // field to store the grid size for
																					// kernels that follow the node count

	struct work_item* mat_vec_data_at_current_offset = mat_vec_data;	// array to store the current tail of the queue / array that holds the MatVecs

	for (int current_level=0; current_level<max_level; current_level++)  // run over all arrays
	{
		set_array<<<grid_size_for_children, block_size>>>(new_child_counts, 0, total_children);  //  will compute new child counts for total_children nodes
		checkCUDAError("set_array");

		// find number of children & MatVecs per node on current level
		count_for_new_level<<<grid_size_for_children, block_size>>>(current_level_data, next_level_data, new_mat_vec_counts, new_child_counts, total_children, input_set1_codes, input_set2_codes, input_set1, input_set2, eta, current_level, max_level, c_leaf);
		checkCUDAError("count_for_new_level");

		// compute total number of new children & MatVecs
		total_new_mat_vecs = thrust::reduce(new_mat_vec_counts_ptr, new_mat_vec_counts_ptr+total_children);
		total_new_children = thrust::reduce(new_child_counts_ptr, new_child_counts_ptr+total_children);

		*mat_vec_data_count = *mat_vec_data_count + total_new_mat_vecs;

		// compute node offsets in new level & offsets for MatVecs
		thrust::exclusive_scan(new_mat_vec_counts_ptr, new_mat_vec_counts_ptr+total_children, new_mat_vec_counts_ptr);   // here, I reuse the field to store the offsets (for memory efficiency reasons)
		thrust::exclusive_scan(new_child_counts_ptr, new_child_counts_ptr+total_children, new_child_offsets_ptr);  // here, I store the offsets in a dedicated field

		// generate new level with nodes and write new MatVecs into queue
		generate_new_level<<<grid_size_for_children, block_size>>>(current_level_data, next_level_data, mat_vec_data_at_current_offset, new_child_counts, total_children, new_mat_vec_counts, new_child_offsets, input_set1_codes, input_set2_codes, input_set1, input_set2, eta, current_level, max_level, c_leaf);
		checkCUDAError("generate_new_level");

		// move forward tail of MatVecs queue
		mat_vec_data_at_current_offset = &mat_vec_data_at_current_offset[total_new_mat_vecs];

		// data on current level is no longer needed -> cleanup
		invalidate_array<<<grid_size_for_children, block_size>>>(current_level_data, total_children);
		checkCUDAError("invalidate_array2");

		// switch to next level by flipping pointers to array
		struct work_item* tmp_level_data_pointer;
		tmp_level_data_pointer = next_level_data;
		next_level_data = current_level_data;
		current_level_data = tmp_level_data_pointer;
		total_children = total_new_children;

		// compute new compute configuration for children computations
		grid_size_for_children = (total_children + (block_size - 1)) / block_size;

		if (total_children==0) // stopping when no more children are generated
			break;
	}
}

void traverse_with_dynamic_arrays(struct work_item root_h, struct work_item* mat_vec_data, int* mat_vec_data_count, struct morton_code* input_set1_codes, struct morton_code* input_set2_codes, struct point_set* input_set1, struct point_set* input_set2, double eta, int max_level, int c_leaf, int max_elements_in_array)
{
	struct work_item* current_level_data = 0;
	struct work_item* next_level_data = 0;

//	hipMalloc((void**)&current_level_data, max_elements_in_array*sizeof(struct work_item));
//	hipMalloc((void**)&next_level_data, max_elements_in_array*sizeof(struct work_item));

	int* new_mat_vec_counts;   // number of MatVecs that will be generated per valid node of the current tree level
	int* new_child_counts;     // number of child nodes that will be generated per valid node of the current tree level
	int* new_child_offsets;    // storage for the offsets for the new child nodes in the next tree level

	int total_new_mat_vecs;  // temp field to store total number of new MatVecs of the current level

	int total_children = 1;  // number of nodes on current level
	int total_new_children;  // temp field to store total number of new nodes on next level
	int block_size = 512;
	int grid_size_for_children = (total_children + (block_size - 1)) / block_size;  // field to store the grid size for
																					// kernels that follow the node count

	struct work_item* mat_vec_data_at_current_offset = mat_vec_data;	// array to store the current tail of the queue / array that holds the MatVecs

	// allocate array for current level
	hipMalloc((void**)&current_level_data, total_children*sizeof(struct work_item));
	checkCUDAError("cudaMalloc0");
	invalidate_array<<<(total_children + (block_size - 1)) / block_size, block_size>>>(current_level_data, total_children);
	checkCUDAError("invalidate_array0");

	// fill initial node into first tree level
	init_tree_array_root<<<1, 1>>>(current_level_data, root_h);
	checkCUDAError("init_tree_array_root");

	for (int current_level=0; current_level<max_level; current_level++)  // run over all arrays
	{
		// allocation
		hipMalloc((void**)&new_mat_vec_counts, total_children*sizeof(int));
		checkCUDAError("cudaMalloc01");
		hipMalloc((void**)&new_child_counts, total_children*sizeof(int));
		checkCUDAError("cudaMalloc02");
		hipMalloc((void**)&new_child_offsets, total_children*sizeof(int));
		checkCUDAError("cudaMalloc03");
		// pointer fun
		thrust::device_ptr<int> new_mat_vec_counts_ptr(new_mat_vec_counts);
		thrust::device_ptr<int> new_child_counts_ptr(new_child_counts);
		thrust::device_ptr<int> new_child_offsets_ptr(new_child_offsets);

		set_array<<<grid_size_for_children, block_size>>>(new_child_counts, 0, total_children);  //  will compute new child counts for total_children nodes
		checkCUDAError("set_array");

		// find number of children & MatVecs per node on current level
		count_for_new_level<<<grid_size_for_children, block_size>>>(current_level_data, next_level_data, new_mat_vec_counts, new_child_counts, total_children, input_set1_codes, input_set2_codes, input_set1, input_set2, eta, current_level, max_level, c_leaf);
		checkCUDAError("count_for_new_level");

		// compute total number of new children & MatVecs
		total_new_mat_vecs = thrust::reduce(new_mat_vec_counts_ptr, new_mat_vec_counts_ptr+total_children);
		total_new_children = thrust::reduce(new_child_counts_ptr, new_child_counts_ptr+total_children);

		*mat_vec_data_count = *mat_vec_data_count + total_new_mat_vecs;

		// compute node offsets in new level & offsets for MatVecs
		thrust::exclusive_scan(new_mat_vec_counts_ptr, new_mat_vec_counts_ptr+total_children, new_mat_vec_counts_ptr);   // here, I reuse the field to store the offsets (for memory efficiency reasons)
		thrust::exclusive_scan(new_child_counts_ptr, new_child_counts_ptr+total_children, new_child_offsets_ptr);  // here, I store the offsets in a dedicated field

		// allocate array for next level
		hipMalloc((void**)&next_level_data, total_new_children*sizeof(struct work_item));
		checkCUDAError("cudaMalloc1");
		if (total_new_children > 0)  // handle case in which no new level is generated, does not work with 0 grid size
		{
			invalidate_array<<<(total_new_children + (block_size - 1)) / block_size, block_size>>>(next_level_data, total_new_children);
			checkCUDAError("invalidate_array1");
		}

		// generate new level with nodes and write new MatVecs into queue
		generate_new_level<<<grid_size_for_children, block_size>>>(current_level_data, next_level_data, mat_vec_data_at_current_offset, new_child_counts, total_children, new_mat_vec_counts, new_child_offsets, input_set1_codes, input_set2_codes, input_set1, input_set2, eta, current_level, max_level, c_leaf);
		checkCUDAError("generate_new_level");

		// move forward tail of MatVecs queue
		mat_vec_data_at_current_offset = &mat_vec_data_at_current_offset[total_new_mat_vecs];

		// data on current level is no longer needed -> cleanup
		hipFree(current_level_data);
		hipFree(new_mat_vec_counts);
		hipFree(new_child_counts);
		hipFree(new_child_offsets);

		// switch to next level
		current_level_data = next_level_data;
		next_level_data = 0;
		total_children = total_new_children;

		// compute new compute configuration for children computations
		grid_size_for_children = (total_children + (block_size - 1)) / block_size;

		if (total_children==0) // stopping when no more children are generated
		{
			hipFree(current_level_data);
			break;
		}
	}
}

struct min_or_m2_in_second_argument : public thrust::binary_function<int, int, int>
{
	__host__ __device__ int operator()(int a, int b)
	{
		return (b!=-2) ? max(a,b) : -2;
	}
};

struct equals_zero : public thrust::unary_function<int, int>
{
	__host__ __device__ int operator()(int x) { return x==0; }
};

void compute_bounding_boxes_fun_old(struct work_item* current_level_data, int total_children, struct point_set* input_set1, struct point_set* input_set2)
{
	int block_size = 512;
	compute_bounding_boxes<<<(total_children + (block_size - 1)) / block_size, block_size>>>(current_level_data, total_children, input_set1, input_set2);
	checkCUDAError("compute_bounding_boxes");
}

__global__ void set_bounds_for_keys(int* keys, struct work_item* current_level_data, int total_children, int set_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= total_children)
		return;

	struct work_item* work = &current_level_data[idx];

	int l, u;

	if (set_num==1)
	{
		l = work->set1_l;
		u = work->set1_u;
	}
	else
	{
		l = work->set2_l;
		u = work->set2_u;
	}

	if (l!=u)
	{
		keys[l] = idx;
		keys[u] = -2;
	}
}

__global__ void set_bounds_for_keys_using_limits(int* keys, int* l, int* u, int total_children, int set_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= total_children)
		return;

	if (l[idx]!=u[idx])
	{
		keys[l[idx]] = (idx+1);
		keys[u[idx]] = -(idx+1);
	}
}

__global__ void correct_upper_bound(int* keys, struct work_item* current_level_data, int total_children, int set_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= total_children)
		return;

	struct work_item* work = &current_level_data[idx];

	int l,u;

	if (set_num==1)
	{
		l = work->set1_l;
		u = work->set1_u;
	}
	else
	{
		l = work->set2_l;
		u = work->set2_u;
	}


//	if (l!=u)
		keys[u] = idx;

}

__global__ void correct_upper_bound_using_limits(int* keys, int* l, int* u, int total_children, int set_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= total_children)
		return;

//	if (l[idx]!=u[idx])
		keys[u[idx]] = (idx+1);

}

__global__ void set_bounding_box_minmax(double* maxs, int* output_keys, int d, int dim, int maxs_size, struct work_item* current_level_data, int type){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= maxs_size)
		return;

	int output_pos = output_keys[idx];

	if (output_pos==-1)
	{
		return;
	}

	struct work_item* work = &current_level_data[output_pos];

	switch (type)
	{
		case 1: work->max1[d] = maxs[idx]; break;
		case 2: work->max2[d] = maxs[idx]; break;
		case 3: work->min1[d] = maxs[idx]; break;
		case 4: work->min2[d] = maxs[idx]; break;
	}
	work->dim = dim;
}

__global__ void get_point_count_dim(int* point_count, int* dim, struct point_set* input_set1)
{
	*point_count = input_set1->size;
	*dim = input_set1->dim;
}

__global__ void get_coords_pointer(double** coords, int dim, struct point_set* input_set)
{
	coords[0] = input_set->coords[dim];
}

void print_int(int* array, int n)
{
	int* array_h = new int[n];
	hipMemcpy(array_h, array, n*sizeof(int), hipMemcpyDeviceToHost);
	for (int i=0; i<n; i++)
		printf("% d ", array_h[i]);
	printf("\n");
	delete[] array_h;
}

void print_double(double* array, int n)
{
	double* array_h = new double[n];
	hipMemcpy(array_h, array, n*sizeof(double), hipMemcpyDeviceToHost);
	for (int i=0; i<n; i++)
		printf("% .5ef ", array_h[i]);
	printf("\n");
	delete[] array_h;
}

void print_bool(bool* array, int n)
{
	bool* array_h = new bool[n];
	hipMemcpy(array_h, array, n*sizeof(bool), hipMemcpyDeviceToHost);
	for (int i=0; i<n; i++)
		printf("%d ", array_h[i]);
	printf("\n");
	delete[] array_h;
}



__global__ void get_work_item_point_set_limits(int* l, int* u, struct work_item* current_level_data, int total_children, int point_set_nr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= total_children)
		return;

	if (point_set_nr==1)
	{
		l[idx] = current_level_data[idx].set1_l;
		u[idx] = current_level_data[idx].set1_u;
	}
	else
	{
		l[idx] = current_level_data[idx].set2_l;
		u[idx] = current_level_data[idx].set2_u;
	}

}

__global__ void initialize_lookup_map(int* map, int* l, int total_children)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= total_children)
		return;

	if (idx==0)
	{
		map[idx] = 0;
	}
	else if (l[idx]!=l[idx-1])
	{
		map[idx] = 1;
	}
	else
		map[idx] = 0;
}

__global__ void apply_permutation_to_map(int* map, int* tmp_map, int* permutation, int total_children)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= total_children)
		return;

	map[permutation[idx]] = tmp_map[idx];
}

// this method computes a map from the work item index to the bounding box computation lookup table entry
void compute_map_for_lookup_table(int* map, struct work_item* current_level_data, int total_children, struct point_set* input_set1, struct point_set* input_set2, int set_nr)
{
	int block_size = 512;

	thrust::device_ptr<int> map_ptr(map);

	// this field will store the permutation applied by the sort command on the "l" field		
	int* permutation;
	hipMalloc((void**)&permutation, total_children*sizeof(int));
	thrust::device_ptr<int> permutation_ptr(permutation);
	thrust::sequence(permutation_ptr, permutation_ptr+total_children);

	// l, u will be vectors storing the lower and upper bounds of the indices mapping this node to actual points in
        // the Z-order curve -- ordered point list 
	// this will just be a linearization of the data contained in the work_item structs
	int* l;
	int* u;
	hipMalloc((void**)&l, total_children*sizeof(int));
	hipMalloc((void**)&u, total_children*sizeof(int));
	thrust::device_ptr<int> l_ptr(l);
	thrust::device_ptr<int> u_ptr(u);

	// kernel to transfer the members set<set_nr>_l and set<set_nr>_u to the arrays l and u for set <set_nr>
	get_work_item_point_set_limits<<<(total_children + (block_size - 1)) / block_size, block_size>>>(l, u, current_level_data, total_children, set_nr);
	hipDeviceSynchronize();
	checkCUDAError("get_work_item_point_set_limits");

	hipFree(u);  // TODO u is not really used; remove?

	// %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
	// The lookup table is only constructed base on the lower bounds of the indices,
        // i.e. "l". This is possible due to the construction of the block cluster tree:
        // We here only look at one of the two point sets. And each point set has the
        // same level of refinement. Therefore, there is a unique disjoint decomposition
        // of the set on a given level. This leads to the fact that identical entries in
        // l will have identical entries in u (which is why it is enough to consider l".
        // %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%  


	// sort the "l" field while keeping the permuation in "permutation"
	thrust::stable_sort_by_key(l_ptr, l_ptr+total_children, permutation_ptr);

	// temporary map that will be later on re-permuted against the sorting process
	int* tmp_map;
	hipMalloc((void**)&tmp_map, total_children*sizeof(int));
	thrust::device_ptr<int> tmp_map_ptr(tmp_map);

	// l:       |  5 |  7 |  7 |  8 | 20 | 20 | 35 | 35 | 40 | 40 | 40 | 40 |
	// tmp_map: |  0 |  1 |  0 |  1 |  1 |  0 |  1 |  0 |  1 |  0 |  0 |  0 |
	initialize_lookup_map<<<(total_children + (block_size - 1)) / block_size, block_size>>>(tmp_map, l, total_children);
	hipDeviceSynchronize();
	checkCUDAError("initialize_lookup_map");

	hipFree(l);

	// tmp_map: |  0 |  1 |  1 |  2 |  3 |  3 |  4 |  4 |  5 |  5 |  5 |  5 |
	thrust::inclusive_scan(tmp_map_ptr, tmp_map_ptr+total_children, tmp_map_ptr);
	
	// pemute "tmp_map" following "permuation" and store it in "map"
	apply_permutation_to_map<<<(total_children + (block_size - 1)) / block_size, block_size>>>(map, tmp_map, permutation, total_children);
	hipDeviceSynchronize();
	checkCUDAError("apply_permutation_to_map");

	// cleanup
	hipFree(tmp_map);
	hipFree(permutation);
}

// this function computes the lookup table for the bounding box computation results
void compute_lookup_table(double*** lookup_table_min, double*** lookup_table_max, int* lookup_table_size, struct work_item* current_level_data, int total_children, struct point_set* input_set1, struct point_set* input_set2, int set_nr)
{
	int block_size = 512;

	// l, u will be vectors storing the lower and upper bounds of the indices mapping this node to actual points in
        // the Z-order curve -- ordered point list 
	// this will just be a linearization of the data contained in the work_item structs
	int* l;
	int* u;
	hipMalloc((void**)&l, total_children*sizeof(int));
	hipMalloc((void**)&u, total_children*sizeof(int));
	thrust::device_ptr<int> l_ptr(l);
	thrust::device_ptr<int> u_ptr(u);

	// kernel to transfer the members set<set_nr>_l and set<set_nr>_u to the arrays l and u for set <set_nr>
	get_work_item_point_set_limits<<<(total_children + (block_size - 1)) / block_size, block_size>>>(l, u, current_level_data, total_children, set_nr);
	hipDeviceSynchronize();
	checkCUDAError("get_work_item_point_set_limits");

	// sort l and u
	thrust::stable_sort(l_ptr, l_ptr+total_children);
	thrust::stable_sort(u_ptr, u_ptr+total_children);

	// remove all double occurences of entries in l
	thrust::device_ptr<int> new_end_unique = thrust::unique(l_ptr, l_ptr+total_children);

	// the lookup table has the size of the unique lower bounds
	*lookup_table_size = new_end_unique - l_ptr;

	// since consecutive entries in u are identical when consecutive entries in l are identical (see big comment in the above routine)
	// we can apply the unique operation to u without taking care about the output size, etc.
        thrust::device_ptr<int> new_end_unique2 = thrust::unique(u_ptr, u_ptr+total_children);
	
	// very dirty way to get point count and dimensionality of points
	int point_count,dim;
	int* point_count_d; hipMalloc((void**)&point_count_d, sizeof(int));
	int* dim_d; hipMalloc((void**)&dim_d, sizeof(int));
	if (set_nr==1)
		get_point_count_dim<<<1,1>>>(point_count_d, dim_d, input_set1);
	else
		get_point_count_dim<<<1,1>>>(point_count_d, dim_d, input_set2);
	hipMemcpy(&point_count, point_count_d, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&dim, dim_d, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(point_count_d); hipFree(dim_d);

	// create lookup_table of size lookup_table_size (as double arrays with either outer pointers on CPU or GPU)
	double** lookup_table_min_h = new double*[dim];
	double** lookup_table_max_h = new double*[dim];
	hipMalloc((void**)lookup_table_min, dim*sizeof(double*));
	hipMalloc((void**)lookup_table_max, dim*sizeof(double));
	for (int d=0; d<dim; d++)
		hipMalloc((void**)&(lookup_table_min_h[d]), lookup_table_size[0]*sizeof(double));
	for (int d=0; d<dim; d++)
		hipMalloc((void**)&(lookup_table_max_h[d]), lookup_table_size[0]*sizeof(double));
	hipMemcpy(*lookup_table_min, lookup_table_min_h, dim*sizeof(double*), hipMemcpyHostToDevice);
	hipMemcpy(*lookup_table_max, lookup_table_max_h, dim*sizeof(double*), hipMemcpyHostToDevice);

	// pointers in which the dimension-wise pointer to the point coordinate array per input set is stored
	double** coords_pointer;
	double* coords_pointer_h;
	hipMalloc((void**)&coords_pointer, sizeof(double*));

	// array of the same size as the point array, storing a mapping of points to the actual lookup table entry to which they belong
	int* keys;
	hipMalloc((void**)&keys,point_count*sizeof(int));
	thrust::device_ptr<int> keys_ptr(keys);

	// this is the output array for the "keys" for the batched min/max reduction
	int* output_keys;
	hipMalloc((void**)&output_keys,point_count*sizeof(int)); // point_count is a too large upper bound
	thrust::device_ptr<int> output_keys_ptr(output_keys);

	thrust::pair<thrust::device_ptr<int>, thrust::device_ptr<double> > new_end;

	// -------------------------------------------------------------------
	// setting up a map of each point to the lookup table entries (unique)
	// -------------------------------------------------------------------

	// keys: | 0 | 0 | 0 | 0 | 0 | 0 | 0 | 0 |
	set_array<<<(point_count + (block_size - 1)) / block_size, block_size>>>(keys, 0, point_count);
	hipDeviceSynchronize();
	checkCUDAError("set_array");

	// setting upper an lower bounds of ranges
	// idx:    0  1  2
	// idx+1:  1  2  3   <- Note: I am storing idx+1 as key, to be able to characterize "empty" entries as "0"
	// l[idx]: 2  0  5      |
	// u[idx]: 4  1  7     \|/ 
	// idx        | 0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 |
	// keys[idx]: | 2 |-2 | 1 | 0 |-1 | 3 | 0 |-3 |
	set_bounds_for_keys_using_limits<<<(*lookup_table_size + (block_size - 1)) / block_size, block_size>>>(keys, l, u, *lookup_table_size, set_nr);
	hipDeviceSynchronize();
	checkCUDAError("set_bounds_for_keys");

	// filling ranges
	// keys[idx]: | 2 | 0 | 1 | 1 | 0 | 3 | 3 | 0 |
	thrust::inclusive_scan(keys_ptr, keys_ptr+point_count, keys_ptr);

	// correcting upper bounds
	// keys[idx]: | 2 | 2 | 1 | 1 | 1 | 3 | 3 | 3 |
	correct_upper_bound_using_limits<<<(*lookup_table_size + (block_size - 1)) / block_size, block_size>>>(keys, l, u, *lookup_table_size, set_nr);
	hipDeviceSynchronize();
	checkCUDAError("correct_upper_bound");

	// this field will be used to store the output of the min/max reductions,
	// i.e. this is the lookup table result for the currently computed dimension
	double* tmp_lookup_table;
	hipMalloc((void**)&tmp_lookup_table, point_count*sizeof(double));
	thrust::device_ptr<double> tmp_lookup_table_ptr(tmp_lookup_table);

	// compute coordinate maxima per dimension
	for (int d=0; d<dim; d++)
	{
		// get pointer to point coordinates for input set "set_nr" and dimension "d"
		if (set_nr==1)
			get_coords_pointer<<<1,1>>>(coords_pointer, d, input_set1);
		else
			get_coords_pointer<<<1,1>>>(coords_pointer, d, input_set2);

		// copy pointer to host memory for a direct access
		hipMemcpy(&coords_pointer_h, coords_pointer, sizeof(double*), hipMemcpyDeviceToHost);
		checkCUDAError("cudaMemcpy11");
		thrust::device_ptr<double> coords_current_dim_ptr(coords_pointer_h);

//		// create thrust pointer for lookup_table for current dimension
//		thrust::device_ptr<double> lookup_table_max_current_dim_ptr(lookup_table_max_h[d]);

		// apply maximum reduction per coordinate subset (wrt. lookup table index)
		// note: keys are already in contigous (non-interrupted) blocks due to the Z order curve ordering
		new_end = thrust::reduce_by_key(keys_ptr, keys_ptr+point_count, coords_current_dim_ptr, output_keys_ptr, tmp_lookup_table_ptr, thrust::equal_to<int>(), thrust::maximum<double>());
		int output_size = new_end.first - output_keys_ptr;

		// remove empty entries in lookup table, which are identified by a "0" in the key
		thrust::device_ptr<double> new_end_without_empty_entries;
		new_end_without_empty_entries = thrust::remove_if(tmp_lookup_table_ptr, tmp_lookup_table_ptr+output_size, output_keys_ptr, equals_zero());
		checkCUDAError("remove_if");
		output_size = new_end_without_empty_entries - tmp_lookup_table_ptr;

		// computed maximum is copied into the lookup table
		// Note: ordering of lookup table entries is implicitly created; mapping via output_keys is not necessary
		
		hipMemcpy(lookup_table_max_h[d], tmp_lookup_table, output_size*sizeof(double), hipMemcpyDeviceToDevice);
		checkCUDAError("cudaMemcpy12ja");

//		set_bounding_box_minmax<<<(output_size + (block_size - 1)) / block_size, block_size>>>(minmaxs, output_keys, d, dim, output_size, current_level_data, 1);
//		hipDeviceSynchronize();
//		checkCUDAError("set_bounding_box_minmax");
	}

	// compute coordinate minima per dimension
	for (int d=0; d<dim; d++)
	{
		// get pointer to point coordinates for input set "set_nr" and dimension "d"
		if (set_nr==1)
			get_coords_pointer<<<1,1>>>(coords_pointer, d, input_set1);
		else
			get_coords_pointer<<<1,1>>>(coords_pointer, d, input_set2);

		// copy pointer to host memory for a direct access
		hipMemcpy(&coords_pointer_h, coords_pointer, sizeof(double*), hipMemcpyDeviceToHost);
		checkCUDAError("cudaMemcpy21");
		thrust::device_ptr<double> coords_current_dim_ptr(coords_pointer_h);

//		// create thrust pointer for lookup_table for current dimension
//		thrust::device_ptr<double> lookup_table_min_current_dim_ptr(lookup_table_min_h[d]);

		// apply minimum reduction per coordinate subset (wrt. lookup table index)
		// note: keys are already in contigous (non-interrupted) blocks due to the Z order curve ordering
		new_end = thrust::reduce_by_key(keys_ptr, keys_ptr+point_count, coords_current_dim_ptr, output_keys_ptr, tmp_lookup_table_ptr, thrust::equal_to<int>(), thrust::minimum<double>());
		int output_size = new_end.first - output_keys_ptr;

		// remove empty entries in lookup table
		thrust::device_ptr<double> new_end_without_empty_entries;
		new_end_without_empty_entries = thrust::remove_if(tmp_lookup_table_ptr, tmp_lookup_table_ptr+output_size, output_keys_ptr, equals_zero());
		output_size = new_end_without_empty_entries - tmp_lookup_table_ptr;

		// computed minimum is copied into the lookup table
		// Note: ordering of lookup table entries is implicitly created; mapping via output_keys is not necessary
		hipMemcpy(lookup_table_min_h[d], tmp_lookup_table, output_size*sizeof(double), hipMemcpyDeviceToDevice);
		checkCUDAError("cudaMemcpy22");
	}

	hipFree(tmp_lookup_table);

	hipFree(l);
	hipFree(u);
	hipFree(coords_pointer);
	hipFree(keys);
	hipFree(output_keys);
}

// for each child node this kernel looks up the bounding box (min & max / dimension) in the lookup table (via map "map")
__global__ void set_bounding_box_minmax_using_lookup_table(double** lookup_table_min, double** lookup_table_max, int* map, int dim, struct work_item* current_level_data, int total_children, int type)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= total_children)
		return;

	// get mapping from child node index to lookup table index
	int map_index = map[idx];  // DEBUG: is this correct for unbalanced trees ? should I have 2 maps?

	// take the node
	struct work_item* work = &current_level_data[idx];

	// set the lookup table value for either the first point set in the node or for the second one
	if (type==1)
	{
		for (int d=0; d<dim; d++)
		{
			work->max1[d] = lookup_table_max[d][map_index];
			work->min1[d] = lookup_table_min[d][map_index];
		}
	}
	else
	{
		for (int d=0; d<dim; d++)
		{
			work->max2[d] = lookup_table_max[d][map_index];
			work->min2[d] = lookup_table_min[d][map_index];
		}
	}

	// set dimension
	work->dim = dim;
}


// this method computes the bounding boxes for each work_item in current_level_data (i.e. for each node on the current
// tree level and sets the boxes as parameters in each work_item / node
void compute_bounding_boxes_fun(struct work_item* current_level_data, int total_children, struct point_set* input_set1, struct point_set* input_set2)
{
	// very dirty way to get point count and dimensionality of points
	int point_count,dim;
	int* point_count_d; hipMalloc((void**)&point_count_d, sizeof(int));
	int* dim_d; hipMalloc((void**)&dim_d, sizeof(int));
	get_point_count_dim<<<1,1>>>(point_count_d, dim_d, input_set1);
	hipMemcpy(&point_count, point_count_d, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&dim, dim_d, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(point_count_d); hipFree(dim_d);

	// this field will hold the mapping of the node / work_item indices to the lookup table
	int* map;
	hipMalloc((void**)&map, total_children*sizeof(double));

	// these fields will hold the lookup tables for the bounding boxes (min & max)
	double** lookup_table_min;
	double** lookup_table_max;
	int lookup_table_size;


	int block_size = 512;


	// ---------------------------------------------------------------------------
	// compute the bounding box for the first set in each node
	// ---------------------------------------------------------------------------



	// given the current_level_data, i.e. the nodes on this level, compute the mapping from the node indices to the lookup table entries
	compute_map_for_lookup_table(map, current_level_data, total_children, input_set1, input_set2, 1);

	// now compute the lookup table
	compute_lookup_table(&lookup_table_min, &lookup_table_max, &lookup_table_size, current_level_data, total_children, input_set1, input_set2, 1);

	// finally use the lookup table to assign the computed bounding boxes to the nodes	
	set_bounding_box_minmax_using_lookup_table<<<(total_children + (block_size - 1)) / block_size, block_size>>>(lookup_table_min, lookup_table_max, map, dim, current_level_data, total_children, 1);
	hipDeviceSynchronize();
	checkCUDAError("set_bounding_box_minmax_using_lookup_table");

	// cleaning up the lookup tables which were generated in "compute_lookup_table"
	double** tmp_array = new double*[dim];
	hipMemcpy(tmp_array, lookup_table_min, dim*sizeof(double*), hipMemcpyDeviceToHost);
	checkCUDAError("cudaMemcpy1");
	for (int d=0; d<dim; d++)
	{
		hipFree(tmp_array[d]);
		checkCUDAError("cudaFree1");
	}
	hipMemcpy(tmp_array, lookup_table_max, dim*sizeof(double*), hipMemcpyDeviceToHost);
	checkCUDAError("cudaMemcpy2");
	for (int d=0; d<dim; d++)
	{
		hipFree(tmp_array[d]);
		checkCUDAError("cudaFree2");
	}

	// ---------------------------------------------------------------------------
	// compute the bounding box for the second set in each node
	// ---------------------------------------------------------------------------

	// given the current_level_data, i.e. the nodes on this level, compute the mapping from the node indices to the lookup table entries
	compute_map_for_lookup_table(map, current_level_data, total_children, input_set1, input_set2, 2);

	// now compute the lookup table
	compute_lookup_table(&lookup_table_min, &lookup_table_max, &lookup_table_size, current_level_data, total_children, input_set1, input_set2, 2);

	// finally use the lookup table to assign the computed bounding boxes to the nodes	
	set_bounding_box_minmax_using_lookup_table<<<(total_children + (block_size - 1)) / block_size, block_size>>>(lookup_table_min, lookup_table_max, map, dim, current_level_data, total_children, 2);
	hipDeviceSynchronize();
	checkCUDAError("set_bounding_box_minmax_using_lookup_table");

	// cleaning up the lookup tables which were generated in "compute_lookup_table"
	hipMemcpy(tmp_array, lookup_table_min, dim*sizeof(double*), hipMemcpyDeviceToHost);
	checkCUDAError("cudaMemcpy1");
	for (int d=0; d<dim; d++)
	{
		hipFree(tmp_array[d]);
		checkCUDAError("cudaFree1");
	}
	hipMemcpy(tmp_array, lookup_table_max, dim*sizeof(double*), hipMemcpyDeviceToHost);
	checkCUDAError("cudaMemcpy2");
	for (int d=0; d<dim; d++)
	{
		hipFree(tmp_array[d]);
		checkCUDAError("cudaFree2");
	}

	
	delete[] tmp_array;

	hipFree(map);
}

/*

void compute_bounding_boxes_fun(struct work_item* current_level_data, int total_children, struct point_set* input_set1, struct point_set* input_set2)
{
	int block_size = 512;

	// very dirty way to get point count and dimensionality of points
	int point_count,dim;
	int* point_count_d; hipMalloc((void**)&point_count_d, sizeof(int));
	int* dim_d; hipMalloc((void**)&dim_d, sizeof(int));
	get_point_count_dim<<<1,1>>>(point_count_d, dim_d, input_set1);
	hipMemcpy(&point_count, point_count_d, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&dim, dim_d, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(point_count_d); hipFree(dim_d);

	double** coords_pointer;
	double* coords_pointer_h;
	hipMalloc((void**)&coords_pointer, sizeof(double*));

	int* keys;
	hipMalloc((void**)&keys,point_count*sizeof(int));
	thrust::device_ptr<int> keys_ptr(keys);

	double* minmaxs;
	int* output_keys;
	hipMalloc((void**)&minmaxs,point_count*sizeof(double)); // point_count is a bad upper bound
	hipMalloc((void**)&output_keys,point_count*sizeof(int)); // point_count is a bad upper bound

	thrust::device_ptr<double> minmaxs_ptr(minmaxs);
	thrust::device_ptr<int> output_keys_ptr(output_keys);

	struct min_or_m2_in_second_argument op;

	thrust::pair<thrust::device_ptr<int>, thrust::device_ptr<double> > new_end;

	print_work_items(current_level_data, total_children);

	set_array<<<(point_count + (block_size - 1)) / block_size, block_size>>>(keys, -1, point_count);
	hipDeviceSynchronize();
	checkCUDAError("set_array");

	printf("keys after set array:\n");
	print_int(keys,point_count);

	set_bounds_for_keys<<<(total_children + (block_size - 1)) / block_size, block_size>>>(keys, current_level_data, total_children, 1);
	hipDeviceSynchronize();
	checkCUDAError("set_bounds_for_keys");

	printf("keys after set bounds for keys:\n");
	print_int(keys,point_count);

	thrust::inclusive_scan(keys_ptr, keys_ptr+point_count, keys_ptr, op);

	printf("keys after inclusive scan:\n");
	print_int(keys,point_count);

	correct_upper_bound<<<(total_children + (block_size - 1)) / block_size, block_size>>>(keys, current_level_data, total_children, 1);
	hipDeviceSynchronize();
	checkCUDAError("correct_upper_bound");

	printf("keys after correct upper bound:\n");
	print_int(keys,point_count);

	for (int d=0; d<dim; d++)
	{
		get_coords_pointer<<<1,1>>>(coords_pointer, d, input_set1);

		hipMemcpy(&coords_pointer_h, coords_pointer, sizeof(double*), hipMemcpyDeviceToHost);
		thrust::device_ptr<double> coords_current_dim_ptr(coords_pointer_h);

		printf("keys, coords_current_dim\n");
		print_int(keys, point_count);
		print_double(coords_pointer_h, point_count);

		new_end = thrust::reduce_by_key(keys_ptr, keys_ptr+point_count, coords_current_dim_ptr, output_keys_ptr, minmaxs_ptr, thrust::equal_to<int>(), thrust::maximum<double>());

		int output_size = new_end.first - output_keys_ptr;

		printf("max for d=%d\n", d);
		print_double(minmaxs, output_size);


		set_bounding_box_minmax<<<(output_size + (block_size - 1)) / block_size, block_size>>>(minmaxs, output_keys, d, dim, output_size, current_level_data, 1);
		hipDeviceSynchronize();
		checkCUDAError("set_bounding_box_minmax");
	}

	for (int d=0; d<dim; d++)
	{
		get_coords_pointer<<<1,1>>>(coords_pointer, d, input_set1);

		hipMemcpy(&coords_pointer_h, coords_pointer, sizeof(double*), hipMemcpyDeviceToHost);
		thrust::device_ptr<double> coords_current_dim_ptr(coords_pointer_h);

//		printf("keys, coords_current_dim\n");
//		print_int(keys, point_count);
//		print_double(coords_pointer_h, point_count);

		new_end = thrust::reduce_by_key(keys_ptr, keys_ptr+point_count, coords_current_dim_ptr, output_keys_ptr, minmaxs_ptr, thrust::equal_to<int>(), thrust::minimum<double>());

		int output_size = new_end.first - output_keys_ptr;

//		printf("min for d=%d\n", d);
//		print_double(minmaxs, output_size);

		set_bounding_box_minmax<<<(output_size + (block_size - 1)) / block_size, block_size>>>(minmaxs, output_keys, d, dim, output_size, current_level_data, 3);
		hipDeviceSynchronize();
		checkCUDAError("set_bounding_box_minmax");

	}

	set_array<<<(point_count + (block_size - 1)) / block_size, block_size>>>(keys, -1, point_count);
	hipDeviceSynchronize();
	checkCUDAError("set_array");

	set_bounds_for_keys<<<(total_children + (block_size - 1)) / block_size, block_size>>>(keys, current_level_data, total_children, 2);
	hipDeviceSynchronize();
	checkCUDAError("set_bounds_for_keys");

	thrust::inclusive_scan(keys_ptr, keys_ptr+point_count, keys_ptr, op);

	correct_upper_bound<<<(total_children + (block_size - 1)) / block_size, block_size>>>(keys, current_level_data, total_children, 2);
	hipDeviceSynchronize();
	checkCUDAError("correct_upper_bound");

	for (int d=0; d<dim; d++)
	{
		get_coords_pointer<<<1,1>>>(coords_pointer, d, input_set2);
		hipMemcpy(&coords_pointer_h, coords_pointer, sizeof(double*), hipMemcpyDeviceToHost);
		thrust::device_ptr<double> coords_current_dim_ptr(coords_pointer_h);

//		printf("keys, coords_current_dim\n");
//		print_int(keys, point_count);
//		print_double(coords_pointer_h, point_count);

		new_end = thrust::reduce_by_key(keys_ptr, keys_ptr+point_count, coords_current_dim_ptr, output_keys_ptr, minmaxs_ptr, thrust::equal_to<int>(), thrust::maximum<double>());

		int output_size = new_end.first - output_keys_ptr;

//		printf("max for d=%d\n", d);
//		print_double(minmaxs, output_size);

		set_bounding_box_minmax<<<(output_size + (block_size - 1)) / block_size, block_size>>>(minmaxs, output_keys, d, dim, output_size, current_level_data, 2);
		hipDeviceSynchronize();
		checkCUDAError("set_bounding_box_minmax");
	}

	for (int d=0; d<dim; d++)
	{
		get_coords_pointer<<<1,1>>>(coords_pointer, d, input_set2);
		hipMemcpy(&coords_pointer_h, coords_pointer, sizeof(double*), hipMemcpyDeviceToHost);
		thrust::device_ptr<double> coords_current_dim_ptr(coords_pointer_h);

//		printf("keys, coords_current_dim\n");
//		print_int(keys, point_count);
//		print_double(coords_pointer_h, point_count);

		new_end = thrust::reduce_by_key(keys_ptr, keys_ptr+point_count, coords_current_dim_ptr, output_keys_ptr, minmaxs_ptr, thrust::equal_to<int>(), thrust::minimum<double>());

		int output_size = new_end.first - output_keys_ptr;

//		printf("min for d=%d\n", d);
//		print_double(minmaxs, output_size);

		set_bounding_box_minmax<<<(output_size + (block_size - 1)) / block_size, block_size>>>(minmaxs, output_keys, d, dim, output_size, current_level_data, 4);
		hipDeviceSynchronize();
		checkCUDAError("set_bounding_box_minmax");
	}
	hipDeviceSynchronize();

	hipFree(keys);
	checkCUDAError("hipFree");
	hipFree(minmaxs);
	checkCUDAError("hipFree");
	hipFree(output_keys);
	checkCUDAError("hipFree");
	hipFree(coords_pointer);
	checkCUDAError("hipFree");
}

*/

void traverse_with_dynamic_arrays_dynamic_output(struct work_item root_h, struct work_item** mat_vec_data, int* mat_vec_data_count, int* mat_vec_data_array_size, struct morton_code* input_set1_codes, struct morton_code* input_set2_codes, struct point_set* input_set1, struct point_set* input_set2, double eta, int max_level, int c_leaf, int max_elements_in_array)
{
	struct work_item* current_level_data = 0;
	struct work_item* next_level_data = 0;

//	hipMalloc((void**)&current_level_data, max_elements_in_array*sizeof(struct work_item));
//	hipMalloc((void**)&next_level_data, max_elements_in_array*sizeof(struct work_item));

	int* new_mat_vec_counts;   // number of MatVecs that will be generated per valid node of the current tree level
	int* new_child_counts;     // number of child nodes that will be generated per valid node of the current tree level
	int* new_child_offsets;    // storage for the offsets for the new child nodes in the next tree level

	int total_new_mat_vecs;  // temp field to store total number of new MatVecs of the current level

	int total_children = 1;  // number of nodes on current level
	int total_new_children;  // temp field to store total number of new nodes on next level
	int block_size = 512;
	int grid_size_for_children = (total_children + (block_size - 1)) / block_size;  // field to store the grid size for
																					// kernels that follow the node count

	struct work_item* mat_vec_data_at_current_offset = *mat_vec_data;	// array to store the current tail of the queue / array that holds the MatVecs

	// allocate array for current level
	hipMalloc((void**)&current_level_data, total_children*sizeof(struct work_item));
	checkCUDAError("cudaMalloc0");
	invalidate_array<<<(total_children + (block_size - 1)) / block_size, block_size>>>(current_level_data, total_children);
	hipDeviceSynchronize();
	checkCUDAError("invalidate_array0");

	// fill initial node into first tree level
	init_tree_array_root<<<1, 1>>>(current_level_data, root_h);
	hipDeviceSynchronize();
	checkCUDAError("init_tree_array_root");

	for (int current_level=0; current_level<max_level; current_level++)  // run over all arrays
	{
//		TIME_sstart;
//		compute_bounding_boxes_fun_old(current_level_data, total_children, input_set1, input_set2);
		compute_bounding_boxes_fun(current_level_data, total_children, input_set1, input_set2);
//		TIME_sstop("compute_bounding_boxes");

		hipDeviceSynchronize();
		checkCUDAError("hipDeviceSynchronize");

		// allocation
		hipMalloc((void**)&new_mat_vec_counts, total_children*sizeof(int));
		checkCUDAError("cudaMalloc01");
		hipMalloc((void**)&new_child_counts, total_children*sizeof(int));
		checkCUDAError("cudaMalloc02");
		hipMalloc((void**)&new_child_offsets, total_children*sizeof(int));
		checkCUDAError("cudaMalloc03");
		// pointer fun
		thrust::device_ptr<int> new_mat_vec_counts_ptr(new_mat_vec_counts);
		thrust::device_ptr<int> new_child_counts_ptr(new_child_counts);
		thrust::device_ptr<int> new_child_offsets_ptr(new_child_offsets);

		set_array<<<grid_size_for_children, block_size>>>(new_child_counts, 0, total_children);  //  will compute new child counts for total_children nodes
		hipDeviceSynchronize();
		checkCUDAError("set_array");

		// find number of children & MatVecs per node on current level
		count_for_new_level<<<grid_size_for_children, block_size>>>(current_level_data, next_level_data, new_mat_vec_counts, new_child_counts, total_children, input_set1_codes, input_set2_codes, input_set1, input_set2, eta, current_level, max_level, c_leaf);
		hipDeviceSynchronize();
		checkCUDAError("count_for_new_level");


		size_t free_mem;
		size_t total_mem;

		// compute total number of new children & MatVecs
		total_new_mat_vecs = thrust::reduce(new_mat_vec_counts_ptr, new_mat_vec_counts_ptr+total_children);
		total_new_children = thrust::reduce(new_child_counts_ptr, new_child_counts_ptr+total_children);

		// compute node offsets in new level & offsets for MatVecs
		thrust::exclusive_scan(new_mat_vec_counts_ptr, new_mat_vec_counts_ptr+total_children, new_mat_vec_counts_ptr);   // here, I reuse the field to store the offsets (for memory efficiency reasons)
		thrust::exclusive_scan(new_child_counts_ptr, new_child_counts_ptr+total_children, new_child_offsets_ptr);  // here, I store the offsets in a dedicated field

		// dynamically increase the size of the MatVec data array, if necessary
		// WARNING: This is still not the best possible implementation since (starting from a specific size) it always requires to reallocate memory
		if ((*mat_vec_data_count + total_new_mat_vecs)> *mat_vec_data_array_size)
		{
			struct work_item* new_array;  // pointer for new array
			struct work_item* old_array = *mat_vec_data;  // save pointer to old array
//	        hipMemGetInfo(&free_mem, &total_mem);
//		printf("MatVecDataCount: %d\n", *mat_vec_data_count);
//		printf("Memory free before malloc: %d / %d MB\n", (int)(free_mem/1024/1024), (int)(total_mem/1024/1024));
//		printf("realloc malloc %p %d\n",new_array, (*mat_vec_data_count + total_new_mat_vecs)*sizeof(struct work_item) );
			hipMalloc((void**) &new_array, (*mat_vec_data_count + total_new_mat_vecs)*sizeof(struct work_item));  // allocate new, larger array
			checkCUDAError("nach realloc");
			hipMemcpy(new_array, old_array, (*mat_vec_data_count)*sizeof(struct work_item), hipMemcpyDeviceToDevice);  // transfer data to new array
			checkCUDAError("nach memcopy");
			*mat_vec_data = new_array;  // set new field as standard mat_vec_data array
			*mat_vec_data_array_size = *mat_vec_data_count + total_new_mat_vecs;  // store size of new array
			hipFree(old_array);  // delete old array
			checkCUDAError("nach free");
			mat_vec_data_at_current_offset = &new_array[*mat_vec_data_count];  // re-set tail of output queue
		}

//	        hipMemGetInfo(&free_mem, &total_mem);
//		printf("Memory free before malloc: %d / %d MB\n", (int)(free_mem/1024/1024), (int)(total_mem/1024/1024));


//		printf("malloc %p %d\n",next_level_data, total_new_children*sizeof(struct work_item));

		// allocate array for next level
		hipMalloc((void**)&next_level_data, total_new_children*sizeof(struct work_item));
		checkCUDAError("cudaMalloc1");
//	        hipMemGetInfo(&free_mem, &total_mem);
//		printf("Memory free after malloc: %d / %d MB\n", (int)(free_mem/1024/1024), (int)(total_mem/1024/1024));

		if (total_new_children > 0)  // handle case in which no new level is generated, does not work with 0 grid size
		{
			invalidate_array<<<(total_new_children + (block_size - 1)) / block_size, block_size>>>(next_level_data, total_new_children);
			hipDeviceSynchronize();
			checkCUDAError("invalidate_array1");
		}


		// generate new level with nodes and write new MatVecs into queue
		generate_new_level<<<grid_size_for_children, block_size>>>(current_level_data, next_level_data, mat_vec_data_at_current_offset, new_child_counts, total_children, new_mat_vec_counts, new_child_offsets, input_set1_codes, input_set2_codes, input_set1, input_set2, eta, current_level, max_level, c_leaf);
		hipDeviceSynchronize();
		checkCUDAError("generate_new_level");


		// move forward tail of MatVecs queue and update size
		mat_vec_data_at_current_offset = &mat_vec_data_at_current_offset[total_new_mat_vecs];
		*mat_vec_data_count = *mat_vec_data_count + total_new_mat_vecs;

		// data on current level is no longer needed -> cleanup
		hipFree(current_level_data);
		hipFree(new_mat_vec_counts);
		hipFree(new_child_counts);
		hipFree(new_child_offsets);

		// switch to next level
		current_level_data = next_level_data;
		next_level_data = 0;
		total_children = total_new_children;

		// compute new compute configuration for children computations
		grid_size_for_children = (total_children + (block_size - 1)) / block_size;

		if (total_children==0) // stopping when no more children are generated
		{
			hipFree(current_level_data);
			break;
		}
	}
}



